#include "hip/hip_runtime.h"
#include <cmath>
#include <omp.h>
#include <numeric>
#include <algorithm>
#include <set>
#include <queue>

#include "ReBELL.hpp"
#include "parallelAlgorithm.cuh"
#include "CudaTimeCalculator.cuh"

#define COL_BLOCK_SIZE 32

void noReorderRow(const sparseMatrix::CSR<float> &matrix, std::vector<UIN> &reorderedRows, float &time) {
    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();
    reorderedRows.resize(matrix.row());
    iota(reorderedRows.begin(), reorderedRows.end(), 0);

    std::vector<UIN> numColIndices(matrix.row());
#pragma omp parallel for
    for (int row = 0; row < matrix.row(); ++row) {
        numColIndices[row] = matrix.rowOffsets()[row + 1] - matrix.rowOffsets()[row];
    }

    host::sort_by_key(numColIndices.data(), numColIndices.data() + numColIndices.size(),
                      reorderedRows.data());

    // Remove zero rows
    {
        UIN startIndexOfNonZeroRow = 0;
        while (startIndexOfNonZeroRow < reorderedRows.size()
            && matrix.rowOffsets()[reorderedRows[startIndexOfNonZeroRow] + 1]
                - matrix.rowOffsets()[reorderedRows[startIndexOfNonZeroRow]] == 0) {
            ++startIndexOfNonZeroRow;
        }
        reorderedRows.erase(reorderedRows.begin(), reorderedRows.begin() + startIndexOfNonZeroRow);
    }
    timeCalculator.endClock();
    time = timeCalculator.getTime();
}

void encoding(const sparseMatrix::CSR<float> &matrix, std::vector<std::vector<UIN>> &encodings) {
    const int colBlock = std::ceil(static_cast<float>(matrix.col()) / COL_BLOCK_SIZE);
    encodings.resize(matrix.row());
#pragma omp parallel for schedule(dynamic)
    for (int row = 0; row < matrix.row(); ++row) {
        encodings[row].resize(colBlock);
        for (int idx = matrix.rowOffsets()[row]; idx < matrix.rowOffsets()[row + 1]; ++idx) {
            const int col = matrix.colIndices()[idx];
            ++encodings[row][col / COL_BLOCK_SIZE];
        }
    }
}

void calculateDispersion(const UIN col,
                         const std::vector<std::vector<UIN>> &encodings,
                         std::vector<UIN> &dispersions) {
#pragma omp parallel for schedule(dynamic)
    for (int row = 0; row < encodings.size(); ++row) {
        UIN numOfNonZeroColBlocks = 0;
        UIN zeroFillings = 0;
        for (int colBlockIdx = 0; colBlockIdx < encodings[row].size(); ++colBlockIdx) {
            const UIN numOfNonZeroCols = encodings[row][colBlockIdx];
            if (numOfNonZeroCols != 0) {
                ++numOfNonZeroColBlocks;
                zeroFillings += BLOCK_COL_SIZE - numOfNonZeroCols;
            }
        }
        dispersions[row] = COL_BLOCK_SIZE * numOfNonZeroColBlocks + zeroFillings;
    }
}

// return similarity between two encodings
float clusterComparison(const std::vector<UIN> &encoding_rep, const std::vector<UIN> &encoding_cmp) {
    UIN sum_of_squares_rep = 0;
    UIN sum_of_squares_cmp = 0;
    for (int idx = 0; idx < encoding_rep.size(); ++idx) {
        sum_of_squares_rep += encoding_rep[idx] * encoding_rep[idx];
        sum_of_squares_cmp += encoding_cmp[idx] * encoding_cmp[idx];
    }
    if (sum_of_squares_rep == 0 && sum_of_squares_cmp == 0) {
        return 1.0f;
    } else if ((sum_of_squares_rep == 0 || sum_of_squares_cmp == 0)) {
        return 0.0f;
    }
    float norm_rep = sqrt((float) sum_of_squares_rep);
    float norm_cmp = sqrt((float) sum_of_squares_cmp);
    float min_sum = 0.0f;
    float max_sum = 0.0f;
    for (int idx = 0; idx < encoding_rep.size(); ++idx) {
        float sim_rep = (float) encoding_rep[idx] / norm_rep;
        float sim_cmp = (float) encoding_cmp[idx] / norm_cmp;
        min_sum += fminf(sim_rep, sim_cmp);
        max_sum += fmaxf(sim_rep, sim_cmp);
    }
    return min_sum / max_sum;
}

void clustering(const std::vector<std::vector<UIN>> &encodings,
                const std::vector<UIN> &rows, const UIN startIndexOfNonZeroRow, std::vector<int> &clusterIds) {

//    UIN num = 0;
    for (int idx = startIndexOfNonZeroRow; idx < encodings.size() - 1; ++idx) {
        if (idx > startIndexOfNonZeroRow && clusterIds[rows[idx]] != -1) {
            continue;
        }
        clusterIds[rows[idx]] = idx;
#pragma omp parallel for schedule(dynamic)
        for (int cmpIdx = idx + 1; cmpIdx < encodings.size(); ++cmpIdx) {
            if (clusterIds[rows[cmpIdx]] != -1) {
                continue;
            }
            const float similarity =
                clusterComparison(encodings[rows[startIndexOfNonZeroRow]], encodings[rows[cmpIdx]]);
            if (similarity > row_similarity_threshold_alpha) {
                clusterIds[rows[cmpIdx]] = clusterIds[rows[idx]];
//                ++num;
            }
        }
    }
//    printf("!!! num = %d\n", num);
}

void rowReordering_cpu(const sparseMatrix::CSR<float> &matrix, std::vector<UIN> &rows, float &time) {

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();

    std::vector<std::vector<UIN>> encodings;
    encoding(matrix, encodings);

    std::vector<UIN> dispersions(matrix.row());
    calculateDispersion(matrix.col(), encodings, dispersions);

    std::vector<UIN> ascendingRow(matrix.row()); // Store the original row id
    std::iota(ascendingRow.begin(), ascendingRow.end(), 0); // ascending = {0, 1, 2, 3, ... rows-1}
    std::stable_sort(ascendingRow.begin(),
                     ascendingRow.end(),
                     [&dispersions](size_t i, size_t j) { return dispersions[i] < dispersions[j]; });

    std::vector<int> clusterIds(matrix.row(), -1);
    UIN startIndexOfNonZeroRow = 0;
    while (startIndexOfNonZeroRow < matrix.row() && dispersions[ascendingRow[startIndexOfNonZeroRow]] == 0) {
        clusterIds[ascendingRow[startIndexOfNonZeroRow]] = 0;
        ++startIndexOfNonZeroRow;
    }

    clustering(encodings, ascendingRow, startIndexOfNonZeroRow, clusterIds);

    rows.resize(matrix.row());
    std::iota(rows.begin(),
              rows.end(),
              0); // rowIndices = {0, 1, 2, 3, ... rows-1}
    std::stable_sort(rows.begin(),
                     rows.end(),
                     [&clusterIds](int i, int j) { return clusterIds[i] < clusterIds[j]; });

    // Remove zero rows
    {
        startIndexOfNonZeroRow = 0;
        while (startIndexOfNonZeroRow < matrix.row()
            && matrix.rowOffsets()[rows[startIndexOfNonZeroRow] + 1]
                - matrix.rowOffsets()[rows[startIndexOfNonZeroRow]] == 0) {
            ++startIndexOfNonZeroRow;
        }
        rows.erase(rows.begin(), rows.begin() + startIndexOfNonZeroRow);
    }

    timeCalculator.endClock();
    time = timeCalculator.getTime();
}

float normalized_weighted_jaccard_sim(std::vector<int> A_pattern,
                                      std::vector<int> B_pattern,
                                      int current_group_size,
                                      int block_size) {
    float score_A = 0.0;
    float score_B = 0.0;

    float sim_A = 0.0;
    float sim_B = 0.0;

    float min_sum = 0.0;
    float max_sum = 0.0;

    for (int i = 0; i < B_pattern.size(); i++) {
        score_A += A_pattern[i] * A_pattern[i];
        score_B += B_pattern[i] * B_pattern[i];
    }

    score_A = sqrt(score_A);
    score_B = sqrt(score_B);

    if (score_A == 0 && score_B == 0)
        return 1.0;
    if (score_A == 0 || score_B == 0)
        return 0.0;

    for (int i = 0; i < B_pattern.size(); i++) {
        if (A_pattern[i] == 0 && B_pattern[i] == 0) {
            continue;
        }

        sim_A = A_pattern[i] / score_A;
        sim_B = B_pattern[i] / score_B;

        min_sum += std::min(sim_A, sim_B);
        max_sum += std::max(sim_A, sim_B);
    }

    return (min_sum / max_sum);
}

std::vector<int> merge_rows(std::vector<int> A, std::vector<int> B) {
    std::vector<int> result(A.size());

    for (int i = 0; i < A.size(); i++) {
        result[i] = A[i] + B[i];
    }
    return result;
}

std::vector<int> bsa_rowReordering_cpu(const sparseMatrix::CSR<float> &matrix,
                                       const float similarity_threshold_alpha,
                                       const int block_size,
                                       float &reordering_time) {
    int rows = matrix.row();
    std::vector<int> row_permutation;
    std::priority_queue<std::pair<float, int>> row_queue;
    std::priority_queue<std::pair<float, int>> inner_queue;
    std::vector<std::vector<int>> patterns(rows, std::vector<int>((rows + block_size - 1) / block_size));

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();
    for (int r = 0; r < rows; r++) {
        std::set<int> dense_partition;
        int score = 0;
        int start_pos = matrix.rowOffsets()[r];
        int end_pos = matrix.rowOffsets()[r + 1];
        int nnz = end_pos - start_pos;
        if (nnz == 0) {
            row_permutation.push_back(r);
            continue;
        }

        for (int nz = start_pos; nz < end_pos; nz++) {
            int col = matrix.colIndices()[nz];
            patterns[r][col / block_size]++;
            dense_partition.insert(col / block_size);
        }

        for (int t = 0; t < patterns[r].size(); t++) {
            if (patterns[r][t]) {
                score += block_size - patterns[r][t];
            }
        }

        row_queue.push(std::make_pair(-1 * (score + (float) dense_partition.size() * nnz), -1 * r));
    }

    // usleep(100000);
    int cluster_cnt = 0;
    while (!row_queue.empty()) {
        int current_group_size = 1;
        int i = -1 * row_queue.top().second;
        row_queue.pop();
        cluster_cnt++;

        row_permutation.push_back(i);

        std::vector<int> pattern = patterns[i];
        int j;
        while (!row_queue.empty()) {
            auto j_pair = row_queue.top();
            j = -1 * j_pair.second;

            row_queue.pop();

            std::vector<int> B_pattern = patterns[j];

            float sim = normalized_weighted_jaccard_sim(pattern, B_pattern, current_group_size, block_size);

            if (sim <= similarity_threshold_alpha) {
                inner_queue.push(j_pair);
            } else {
                row_permutation.push_back(j);
                pattern = merge_rows(pattern, B_pattern);
                current_group_size++;
            }
        }

        inner_queue.swap(row_queue);
    }

    // Remove zero rows
    {
        UIN startIndexOfNonZeroRow = 0;
        while (startIndexOfNonZeroRow < row_permutation.size()
            && matrix.rowOffsets()[row_permutation[startIndexOfNonZeroRow] + 1]
                - matrix.rowOffsets()[row_permutation[startIndexOfNonZeroRow]] == 0) {
            ++startIndexOfNonZeroRow;
        }
        row_permutation.erase(row_permutation.begin(), row_permutation.begin() + startIndexOfNonZeroRow);
    }

    timeCalculator.endClock();
    reordering_time = timeCalculator.getTime();
    std::cout << reordering_time << "ms" << std::endl;

    return row_permutation;
}

namespace kernel {

template<typename T>
static __inline__ __device__ T warp_reduce_sum(T value) {
    /* aggregate all value that each thread within a warp holding.*/
    T ret = value;

    for (int w = 1; w < warpSize; w = w << 1) {
        T tmp = __shfl_xor_sync(0xffffffff, ret, w);
        ret += tmp;
    }
    return ret;
}

template<typename T>
static __inline__ __device__ T reduce_sum(T value, T *shm) {
    unsigned int stride;
    unsigned int tid = threadIdx.x;
    T tmp = warp_reduce_sum(value); // perform warp shuffle first for less utilized shared memory

    unsigned int block_warp_id = tid / warpSize;
    unsigned int lane = tid % warpSize;
    if (lane == 0)
        shm[block_warp_id] = tmp;
    __syncthreads();
    for (stride = blockDim.x / (2 * warpSize); stride >= 1; stride = stride >> 1) {
        if (block_warp_id < stride && lane == 0) {
            shm[block_warp_id] += shm[block_warp_id + stride];
        }

        __syncthreads();
    }
    return shm[0];
}

__global__ void calculateDispersion(const UIN *colidx, const UIN *rowptr,
                                    int *weighted_partitions, int *dispersion_score,
                                    int num_blocks_per_row, int col_block_size) {
    extern __shared__ int shm[];
    __shared__ int *encoding;
    __shared__ int *local_result;
    encoding = (int *) &shm[0];
    local_result = (int *) &shm[num_blocks_per_row];
    int row_in_charge = blockIdx.x;
    int row_start = rowptr[row_in_charge];
    int row_nz_count = rowptr[row_in_charge + 1] - row_start;
    // if (row_nz_count == 0)
    //     return;

    for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
        encoding[i] = 0;
    }
    __syncthreads();

    for (int i = threadIdx.x; i < row_nz_count; i += blockDim.x) {
        int col_idx = colidx[row_start + i];
        atomicAdd(&encoding[col_idx / col_block_size], 1);
    }
    __syncthreads();

    int store_offset = row_in_charge * num_blocks_per_row;
    int result_tmp = 0;
    int dense_partition_size = 0;
    for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
        int value = encoding[i];
        weighted_partitions[store_offset + i] = value;
        int is_dense_partition = (value != 0);
        dense_partition_size += is_dense_partition;
        result_tmp += is_dense_partition * (col_block_size - value);
    }
    int result = reduce_sum(result_tmp + row_nz_count * dense_partition_size, local_result);

    if (threadIdx.x == 0) {
        dispersion_score[row_in_charge] = result;
    } else
        return;
}

static __device__ void mutex_lock(unsigned int *mutex) {

    if (threadIdx.x == 0) {
        unsigned int ns = 8;
        while (atomicCAS(mutex, 0, 1) == 1) {
            __nanosleep(ns);
            if (ns < 256) {
                ns *= 2;
            }
        }
    }
    __syncthreads();
}

static __device__ void mutex_unlock(unsigned int *mutex) {
    if (threadIdx.x == 0) {
        atomicExch(mutex, 0);
    }
    __syncthreads();
}

static __device__ float calculate_similarity_norm_weighted_jaccard(const int *encoding_rep,
                                                                   const int *encoding_cmp,
                                                                   int num_blocks_per_row,
                                                                   int *scratch,
                                                                   float *float_scratch) {

    float similarity;
    int sum_of_squares_rep = 0;
    int sum_of_squares_cmp = 0;

    for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
        int e_rep_i = encoding_rep[i];
        int e_cmp_i = encoding_cmp[i];

        sum_of_squares_rep += e_rep_i * e_rep_i;
        sum_of_squares_cmp += e_cmp_i * e_cmp_i;
    }
    sum_of_squares_rep = reduce_sum(sum_of_squares_rep, scratch);
    sum_of_squares_cmp = reduce_sum(sum_of_squares_cmp, scratch);

    if (threadIdx.x == 0) {
        scratch[0] = sum_of_squares_rep;
        scratch[1] = sum_of_squares_cmp;
    }
    __syncthreads();
    sum_of_squares_rep = scratch[0];
    sum_of_squares_cmp = scratch[1];

    if (sum_of_squares_rep == 0 && sum_of_squares_cmp == 0) {
        return 1.0f;
    } else if ((sum_of_squares_rep == 0 || sum_of_squares_cmp == 0)) {
        return 0.0f;
    }
    __syncthreads();

    float norm_rep = sqrt((float) sum_of_squares_rep);
    float norm_cmp = sqrt((float) sum_of_squares_cmp);
    float min_sum = 0.0f;
    float max_sum = 0.0f;

    for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
        float sim_rep = ((float) encoding_rep[i]) / norm_rep;
        float sim_cmp = ((float) encoding_cmp[i]) / norm_cmp;
        min_sum += fminf(sim_rep, sim_cmp);
        max_sum += fmaxf(sim_rep, sim_cmp);
    }
    min_sum = reduce_sum(min_sum, float_scratch);
    max_sum = reduce_sum(max_sum, float_scratch);
    __syncthreads();

    if (threadIdx.x == 0) // only the first warp holds valid values, and use only one thread for simple write
    {
        float sim = min_sum / max_sum;
        float_scratch[0] = sim;
    }
    __syncthreads();
    similarity = float_scratch[0];
    return similarity;
}

static __global__ void bsa_clustering(const int *weighted_partitions,
                                      const int cluster_id,
                                      int *ascending_idx,
                                      volatile int *cluster_ids,
                                      int start_idx,
                                      int num_rows,
                                      int num_blocks_per_row,
                                      float alpha,
                                      size_t shm_size,
                                      unsigned int *mutexes,
                                      int *cluster_id_to_launch,
                                      int *start_idx_to_launch) {
    extern __shared__ int shm[];
    __shared__ int *encoding_rep;
    __shared__ int *scratch;
    __shared__ float *float_scratch;
    encoding_rep = shm;
    scratch = &encoding_rep[num_blocks_per_row];
    float_scratch = (float *) &scratch[blockDim.x / warpSize];

    bool next_cluster_created = false;

    mutex_lock(&mutexes[start_idx]);
    cluster_ids[start_idx] = cluster_id;
    for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
        encoding_rep[i] = weighted_partitions[ascending_idx[start_idx] * num_blocks_per_row + i];
    }
    __syncthreads();

    mutex_unlock(&mutexes[start_idx]);
    mutex_lock(&mutexes[start_idx + 1]);
    cluster_id_to_launch[0] = -1;
    start_idx_to_launch[0] = -1;

    for (int idx = start_idx + 1; idx < num_rows; idx++) {
        volatile int cluster_tmp = cluster_ids[idx];
        if (cluster_tmp != -1) {
            if (idx < num_rows - 1) {
                mutex_lock(&mutexes[idx + 1]);
            }
            mutex_unlock(&mutexes[idx]);
            continue;
        }

        int row = ascending_idx[idx]; // ascending_idx[idx];
        const int *encoding_cmp = &weighted_partitions[row * num_blocks_per_row];
        float similarity;

        similarity = calculate_similarity_norm_weighted_jaccard(encoding_rep,
                                                                encoding_cmp,
                                                                num_blocks_per_row,
                                                                scratch,
                                                                float_scratch);

        if (threadIdx.x == 0) {
            float_scratch[0] = similarity;
        }

        __syncthreads();
        similarity = float_scratch[0];

        if (similarity > alpha) {

            if (threadIdx.x == 0) {
                cluster_ids[idx] = cluster_id;
            }

            for (int i = threadIdx.x; i < num_blocks_per_row; i += blockDim.x) {
                encoding_rep[i] += encoding_cmp[i];
            }

            __syncthreads();
        } else {
            if (!next_cluster_created) {
                if (threadIdx.x == 0) {

                    bsa_clustering<<<1, blockDim.x, shm_size, cudaStreamFireAndForget>>>(weighted_partitions,
                                                                                         cluster_id + 1,
                                                                                         ascending_idx,
                                                                                         cluster_ids,
                                                                                         idx,
                                                                                         num_rows,
                                                                                         num_blocks_per_row,
                                                                                         alpha,
                                                                                         shm_size,
                                                                                         mutexes,
                                                                                         cluster_id_to_launch,
                                                                                         start_idx_to_launch);

                    hipError_t err = hipGetLastError();
                    scratch[0] = (int) hipGetLastError();
                    if (err == cudaErrorLaunchPendingCountExceeded) {
                        cluster_id_to_launch[0] = cluster_id + 1;
                        start_idx_to_launch[0] = idx;
                    }
                }
            }

            next_cluster_created = true;
        }

        if (idx < num_rows - 1) {
            mutex_lock(&mutexes[idx + 1]);
        }
        mutex_unlock(&mutexes[idx]);
    }
}

} // namespace kernel

void calculateDispersion(const sparseMatrix::CSR<float> &matrix,
                         dev::vector<int> &Encodings_gpu,
                         std::vector<int> &Dispersions,
                         dev::vector<int> Dispersions_gpu,
                         const dev::vector<UIN> &rowptr_gpu,
                         const dev::vector<UIN> &colidx_gpu,
                         int num_blocks_per_row,
                         UIN block_size) {
    int blockdim = WARP_SIZE * 4;
    int grid = matrix.row();

    size_t shm_size = num_blocks_per_row * sizeof(UIN) + (blockdim * sizeof(UIN) / WARP_SIZE);
    kernel::calculateDispersion<<<grid, blockdim, shm_size>>>(colidx_gpu.data(), rowptr_gpu.data(),
        Encodings_gpu.data(),
        Dispersions_gpu.data(),
        num_blocks_per_row, block_size);
    hipDeviceSynchronize();

    Dispersions = d2h(Dispersions_gpu);
}

std::vector<UIN> get_permutation_gpu(const sparseMatrix::CSR<float> &mat,
                                     std::vector<int> ascending_idx,
                                     const dev::vector<int> &Encodings,
                                     const std::vector<int> &Dispersions,
                                     int num_blocks_per_row,
                                     float alpha,
                                     int &cluster_cnt) {

    std::vector<int> cluster_ids(mat.row(), -1);
    dev::vector<unsigned int> mutexes(mat.row(), 0);
    int *cluster_id_to_launch, *start_idx_to_launch;

    hipHostMalloc((void **) &cluster_id_to_launch, sizeof(int), hipHostMallocMapped);
    hipHostMalloc((void **) &start_idx_to_launch, sizeof(int), hipHostMallocMapped);

    hipDeviceSynchronize();

    dev::vector<int> ascending_idx_gpu(ascending_idx);

    int blockdim;
    if (num_blocks_per_row < 32) {
        blockdim = 32;
    } else {
        int num_scan_iterate = 4;
        int blockdim_candidate = WARP_SIZE * ceil((float) (num_blocks_per_row / num_scan_iterate) / (float) WARP_SIZE);
        blockdim_candidate = blockdim_candidate > 32 ? blockdim_candidate : 32;
        blockdim = 1024 < blockdim_candidate ? 1024 : blockdim_candidate;
    }
    // blockdim = 1024;

    int grid = 1;

    size_t
        shm_size = (blockdim * sizeof(int) + blockdim * sizeof(float)) / WARP_SIZE + sizeof(int) * num_blocks_per_row;

    hipStream_t initial_stream;
    hipStreamCreateWithFlags(&initial_stream, hipStreamNonBlocking);

    int zero_row_idx = 0;
    int *ascending_idx_head = &ascending_idx[0];

    while (true) {
        if (zero_row_idx == mat.row())
            break;
        if (Dispersions[ascending_idx_head[zero_row_idx]] == 0) {
            // printf("%d is zero row next row = %d\n", ascending_idx[zero_row_idx], ascending_idx[zero_row_idx + 1]);
            cluster_ids[zero_row_idx] = 0;
            zero_row_idx++;
        } else
            break;
    }

    dev::vector<int> cluster_ids_gpu(cluster_ids);

    hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 32768);

    size_t limit;
    int exponent = 2;
    hipDeviceGetLimit(&limit, cudaLimitDevRuntimePendingLaunchCount);

    cluster_id_to_launch[0] = 1;
    start_idx_to_launch[0] = zero_row_idx;

    do {
        kernel::bsa_clustering<<<grid, blockdim, shm_size, initial_stream>>>(Encodings.data(),
            cluster_id_to_launch[0],
            ascending_idx_gpu.data(),
            cluster_ids_gpu.data(),
            start_idx_to_launch[0],
            mat.row(),
            num_blocks_per_row,
            alpha,
            shm_size,
            mutexes.data(),
            cluster_id_to_launch,
            start_idx_to_launch);

        hipDeviceSynchronize();
        limit = limit * exponent;
        if (hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, limit) != hipSuccess) {
            limit = limit / 2;
            exponent = 1;
            hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, limit);
        }

    } while (cluster_id_to_launch[0] != -1);

    cluster_ids = d2h(cluster_ids_gpu);

    auto compare_by_cluster_id = [&cluster_ids](int i, int j) {
      return cluster_ids[i] < cluster_ids[j];
    };
    std::vector<int> indices(mat.row());
    std::iota(indices.begin(), indices.end(), 0);
    std::stable_sort(indices.begin(), indices.end(), compare_by_cluster_id);
    std::vector<UIN> permutation(mat.row());
    for (int i = 0; i < mat.row(); i++) {
        permutation[i] = ascending_idx_head[indices[i]];
    }
    cluster_cnt = cluster_ids[indices[mat.row() - 1]] + (int) (zero_row_idx != 0);
    // cluster_cnt = cluster_ids[mat.row() - 1];

    hipStreamDestroy(initial_stream);

    hipHostFree(cluster_id_to_launch);
    hipHostFree(start_idx_to_launch);

    hipDeviceSetLimit(cudaLimitDevRuntimePendingLaunchCount, 2048);

    return permutation;
}

std::vector<UIN> bsa_rowReordering_gpu(const sparseMatrix::CSR<float> &matrix,
                                       float alpha,
                                       UIN block_size,
                                       float &reordering_time) {

    std::vector<UIN> row_permutation;
    // int num_blocks_per_row = (lhs.cols + block_size - 1) / block_size;
    int num_blocks_per_row = ceil((float) matrix.col() / (float) block_size);

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();

    /*prepare resources -start*/
    std::vector<int> Dispersions;
    dev::vector<int> Encodings_gpu(num_blocks_per_row * matrix.row(), 0);
    dev::vector<int> Dispersions_gpu(matrix.row(), 0);
    dev::vector<UIN> rowptr_gpu(matrix.rowOffsets());
    dev::vector<UIN> colidx_gpu(matrix.colIndices());
    /*prepare resources -done*/

    /*Preprocessing: calculate Encodings and dispersions -start*/
    calculateDispersion(matrix,
                        Encodings_gpu,
                        Dispersions,
                        Dispersions_gpu,
                        rowptr_gpu,
                        colidx_gpu,
                        num_blocks_per_row,
                        block_size);
    /*Preprocessing: calculate Encodings and dispersions -done*/

    /*Prepare Clustering -start*/
    std::vector<int> ascending(matrix.row());
    iota(ascending.begin(), ascending.end(), 0); // ascending = {0, 1, 2, 3, ... lhs.rows-1}
    stable_sort(ascending.begin(),
                ascending.end(),
                [Dispersions](size_t i, size_t j) { return Dispersions[i] < Dispersions[j]; });
    /*Prepare Clustering -done*/

    /*Perform BSA-reordering via gpu -start*/
    int cluster_cnt = 0;
    row_permutation = get_permutation_gpu(matrix,
                                          ascending,
                                          Encodings_gpu,
                                          Dispersions,
                                          num_blocks_per_row,
                                          alpha,
                                          cluster_cnt);
    /*Perform BSA-reordering via gpu -done*/

    // Remove zero rows
    {
        UIN startIndexOfNonZeroRow = 0;
        while (startIndexOfNonZeroRow < row_permutation.size()
            && matrix.rowOffsets()[row_permutation[startIndexOfNonZeroRow] + 1]
                - matrix.rowOffsets()[row_permutation[startIndexOfNonZeroRow]] == 0) {
            ++startIndexOfNonZeroRow;
        }
        row_permutation.erase(row_permutation.begin(), row_permutation.begin() + startIndexOfNonZeroRow);
    }

    timeCalculator.endClock();
    reordering_time = timeCalculator.getTime();
    // cout << reordering_time << "ms" << endl;

    return row_permutation;
}