#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "sddmmKernel.cuh"
#include "TensorCoreConfig.cuh"
#include "ReBELL.hpp"
#include "CudaTimeCalculator.cuh"

namespace kernel {

using namespace nvcuda;

__global__ void checkFragmentData() {
    constexpr UIN wmmaM = 16;
    constexpr UIN wmmaN = 16;
    constexpr UIN wmmaK = 8;
    using matrixAType = float;
    using matrixBType = float;
    using matrixATypeFragment = wmma::precision::tf32;
    using matrixBTypeFragment = wmma::precision::tf32;

    constexpr UIN aTileSize = wmmaM * wmmaK;
    constexpr UIN bTileSize = wmmaK * wmmaN;

    constexpr UIN bRow = wmmaN;
    constexpr UIN bCol = wmmaK;

    constexpr UIN ldATile = wmmaK;
    constexpr UIN ldBTile = wmmaK;

    __shared__ matrixAType aTileSMEM[aTileSize];
    __shared__ matrixBType bTileSMEM[bTileSize];

    const UIN warpId = threadIdx.x / WARP_SIZE;
    const UIN laneId = threadIdx.x % WARP_SIZE;

    if (warpId == 0 && laneId == 0) {
        for (int i = 0; i < aTileSize; ++i) {
            aTileSMEM[i] = static_cast<matrixAType>(i);

        }

//        int row = 0;
//        int col = 0;
//        for (int i = 0; i < bTileSize; ++i) {
//            row %= wmmaK;
//            bTileSMEM[i] = static_cast<matrixBType>(row * wmmaK + col);
//            ++row;
//            if (i % ldBTile == 0) {
//                ++col;
//            }
//        }
        if (bRow == wmmaK) {
            for (int i = 0; i < bTileSize; ++i) {
                bTileSMEM[i] = static_cast<matrixBType>(i);
            }
        } else {
            for (int row = 0; row < wmmaK; ++row) {
                for (int col = 0; col < wmmaN; ++col) {
                    bTileSMEM[row + col * ldBTile] = static_cast<matrixBType>(row * wmmaN + col);
                }
            }
        }
    }

    if (warpId == 0 && laneId == 0) {
        printf("\nmatrix A data : \n");
        printf("| |");
        for (int col = 0; col < wmmaK; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < wmmaK + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < wmmaM; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < wmmaK; ++col) {
                printf("%.0f|", static_cast<float>(aTileSMEM[row * wmmaK + col]));
            }
            printf("\n");
        }

        printf("\nmatrix B data : ");
        if (ldBTile == wmmaN) { printf("(rwo major)\n"); } else { printf("(column major)\n"); }
        printf("| |");
        for (int col = 0; col < bCol; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < bCol + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < bRow; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < bCol; ++col) {
                printf("%.0f|", static_cast<float>(bTileSMEM[row * ldBTile + col]));
            }
            printf("\n");
        }
        printf("\n");

        printf("\nmatrix C data : \n");
        printf("| |");
        for (int col = 0; col < wmmaN; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < wmmaN + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < wmmaM; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < wmmaN; ++col) {
                float c = 0.0f;
                for (int k = 0; k < wmmaK; ++k) {
                    const float a = aTileSMEM[row * ldATile + k];
                    const float b = bTileSMEM[k + col * ldBTile];
                    c += a * b;
                }
                printf("%.0f|", static_cast<float>(c));
            }
            printf("\n");
        }
        printf("\n");
    }

    if (warpId == 0) {
        wmma::fragment<wmma::matrix_a, wmmaM, wmmaN, wmmaK, matrixATypeFragment, wmma::row_major> aFrag;
        wmma::fragment<wmma::matrix_b, wmmaM, wmmaN, wmmaK, matrixBTypeFragment, wmma::col_major> bFrag;

        wmma::fragment<wmma::accumulator, wmmaM, wmmaN, wmmaK, float> cFrag;

        fill_fragment(cFrag, 0.0f);

        wmma::load_matrix_sync(aFrag, aTileSMEM, ldATile);
        wmma::load_matrix_sync(bFrag, bTileSMEM, ldBTile);

        wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);

        if (laneId == 0) {
            printf("\nFragment A tiled data : \n");
        }
        for (int laneIdx = 0; laneIdx < WARP_SIZE; ++laneIdx) {
            if (warpId == 0 && laneId == laneIdx) {
                printf("|T%d|", laneId);
                for (int idxOfFragment = 0; idxOfFragment < aFrag.num_elements; ++idxOfFragment) {
                    printf("%.0f|", static_cast<float>(aFrag.x[idxOfFragment]));
                }
                printf("\n");
            }
        }

        if (laneId == 0) {
            printf("\nFragment B tiled data : \n");
        }
        for (int laneIdx = 0; laneIdx < WARP_SIZE; ++laneIdx) {
            if (warpId == 0 && laneId == laneIdx) {
                printf("|T%d|", laneId);
                for (int idxOfFragment = 0; idxOfFragment < bFrag.num_elements; ++idxOfFragment) {
                    printf("%.0f|", static_cast<float>(bFrag.x[idxOfFragment]));
                }
                printf("\n");
            }
        }

        if (laneId == 0) {
            printf("\nFragment C tiled data : \n");
        }
        for (int laneIdx = 0; laneIdx < WARP_SIZE; ++laneIdx) {
            if (warpId == 0 && laneId == laneIdx) {
                printf("|T%d|", laneId);
                for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                    printf("%.0f|", static_cast<float>(cFrag.x[idxOfFragment]));
                }
                printf("\n");
            }
        }
    }
}

template<typename T>
__global__ void convertDataType(const UIN n, const float *in, T *out) {
    const UIN idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<T>(in[idx]);
//        printf("in[%d] = %f, static_cast<float>out[%d] = %f\n", idx, in[idx], idx, static_cast<float>(out[idx]));
    }
}

template __global__ void convertDataType<int>(const UIN n, const float *in, int *out);
template __global__ void convertDataType<float>(const UIN n, const float *in, float *out);
template __global__ void convertDataType<double>(const UIN n, const float *in, double *out);
template __global__ void convertDataType<half>(const UIN n, const float *in, half *out);

// m16n16k16
// blockDim: [64, 1, 1]
__global__ void sddmm_gpu_dense_block_m16n16k16_block64_rowPanel_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                               const UIN N,
                                                                                               const UIN K,
                                                                                               const MATRIX_A_TYPE *matrixA,
                                                                                               const MATRIX_B_TYPE *matrixB,
                                                                                               const UIN numNonZeroRow,
                                                                                               const UIN *reorderedRows,
                                                                                               const UIN *reorderedCols,
                                                                                               const UIN *reorderedColOffset,
                                                                                               const UIN *blockRowOffsets,
                                                                                               const UIN *blockValues,
                                                                                               MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = N;

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K) {
            // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 4; ++iter) {
                const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = kIter + laneId % 16;

                aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;
#pragma unroll
            for (int iter = 0; iter < 8; ++iter) {
                const UIN bRowId = kIter + warpId * 8 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_N);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel
__global__ void sddmm_gpu_dense_block_m16n16k16_block64_rowPanel_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                               const UIN N,
                                                                                               const UIN K,
                                                                                               const MATRIX_A_TYPE *matrixA,
                                                                                               const MATRIX_B_TYPE *matrixB,
                                                                                               const UIN numNonZeroRow,
                                                                                               const UIN *reorderedRows,
                                                                                               const UIN *reorderedCols,
                                                                                               const UIN *reorderedColOffset,
                                                                                               const UIN *blockRowOffsets,
                                                                                               const UIN *blockValues,
                                                                                               MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K) {
            // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 4; ++iter) {
                const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = kIter + laneId % 16;

                aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 8; ++iter) {
                const UIN bRowId = kIter + warpId * 8 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel中的2个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block64_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                      const UIN N,
                                                                                      const UIN K,
                                                                                      const MATRIX_A_TYPE *matrixA,
                                                                                      const MATRIX_B_TYPE *matrixB,
                                                                                      const UIN numNonZeroRow,
                                                                                      const UIN *reorderedRows,
                                                                                      const UIN *reorderedCols,
                                                                                      const UIN *reorderedColOffset,
                                                                                      const UIN *blockRowOffsets,
                                                                                      const UIN *blockValues,
                                                                                      MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentIter = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

    const UIN lda = K;
    const UIN ldb = N;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId % 16;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel中的2个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block64_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                      const UIN N,
                                                                                      const UIN K,
                                                                                      const MATRIX_A_TYPE *matrixA,
                                                                                      const MATRIX_B_TYPE *matrixB,
                                                                                      const UIN numNonZeroRow,
                                                                                      const UIN *reorderedRows,
                                                                                      const UIN *reorderedCols,
                                                                                      const UIN *reorderedColOffset,
                                                                                      const UIN *blockRowOffsets,
                                                                                      const UIN *blockValues,
                                                                                      MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 2;

    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentIter = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId % 16;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [128, 1, 1]
// 一个thread block负责一个row panel中的4个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block128_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                       const UIN N,
                                                                                       const UIN K,
                                                                                       const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                       const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                       const float alpha,
                                                                                       const float beta,
                                                                                       const UIN numNonZeroRow,
                                                                                       const UIN *__restrict__ reorderedRows,
                                                                                       const UIN *__restrict__ reorderedCols,
                                                                                       const UIN *__restrict__ reorderedColOffset,
                                                                                       const UIN *__restrict__ blockRowOffsets,
                                                                                       const UIN *__restrict__ blockValues,
                                                                                       MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 4;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_N) * 2;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks) * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K * 2) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 4) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }

        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 2; ++iter) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter * WMMA_K, WMMA_K * 2);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter * WMMA_K, WMMA_K * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [256, 1, 1]
// 一个thread block负责一个row panel中的8个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block256_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                       const UIN N,
                                                                                       const UIN K,
                                                                                       const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                       const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                       const float alpha,
                                                                                       const float beta,
                                                                                       const UIN numNonZeroRow,
                                                                                       const UIN *__restrict__ reorderedRows,
                                                                                       const UIN *__restrict__ reorderedCols,
                                                                                       const UIN *__restrict__ reorderedColOffset,
                                                                                       const UIN *__restrict__ blockRowOffsets,
                                                                                       const UIN *__restrict__ blockValues,
                                                                                       MATRIX_C_TYPE *matrixP) {
    constexpr int number_of_tiles_loaded_in_one_cycle = 32 / WMMA_K;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_K) * number_of_tiles_loaded_in_one_cycle;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * each_thread_block_counts_the_number_Of_col_blocks)
        * number_of_tiles_loaded_in_one_cycle;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * each_thread_block_counts_the_number_Of_col_blocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = N;

    // Loop over K, one iteration 32
    for (int kIter = 0; kIter < K; kIter += 32) {
        // Load matrix A into shared memory, each thread loads 2 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 2; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 2) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 64 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
        }

        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 32; iter += WMMA_K) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter, 32);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter, 32);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }

        }
    }
}

// m16n16k16
// blockDim: [256, 1, 1]
// 一个thread block负责一个row panel中的8个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block256_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                       const UIN N,
                                                                                       const UIN K,
                                                                                       const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                       const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                       const float alpha,
                                                                                       const float beta,
                                                                                       const UIN numNonZeroRow,
                                                                                       const UIN *__restrict__ reorderedRows,
                                                                                       const UIN *__restrict__ reorderedCols,
                                                                                       const UIN *__restrict__ reorderedColOffset,
                                                                                       const UIN *__restrict__ blockRowOffsets,
                                                                                       const UIN *__restrict__ blockValues,
                                                                                       MATRIX_C_TYPE *matrixP) {
    constexpr int number_of_tiles_loaded_in_one_cycle = 32 / WMMA_K;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_K) * number_of_tiles_loaded_in_one_cycle;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * each_thread_block_counts_the_number_Of_col_blocks)
        * number_of_tiles_loaded_in_one_cycle;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * each_thread_block_counts_the_number_Of_col_blocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration 32
    for (int kIter = 0; kIter < K; kIter += 32) {
        // Load matrix A into shared memory, each thread loads 2 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 2; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 2) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 64 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }

        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 32; iter += WMMA_K) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter, 32);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter, 32);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [256, 1, 1]
// 一个thread block负责一个row panel中的8个col block
__global__ void sddmm_gpu_dense_block_m16n16k8_block256_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                      const UIN N,
                                                                                      const UIN K,
                                                                                      const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                      const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                      const float alpha,
                                                                                      const float beta,
                                                                                      const UIN numNonZeroRow,
                                                                                      const UIN *__restrict__ reorderedRows,
                                                                                      const UIN *__restrict__ reorderedCols,
                                                                                      const UIN *__restrict__ reorderedColOffset,
                                                                                      const UIN *__restrict__ blockRowOffsets,
                                                                                      const UIN *__restrict__ blockValues,
                                                                                      MATRIX_C_TYPE *matrixP) {
    constexpr int number_of_tiles_loaded_in_one_cycle = 32 / WMMA_K;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_K) * number_of_tiles_loaded_in_one_cycle;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * each_thread_block_counts_the_number_Of_col_blocks)
        * number_of_tiles_loaded_in_one_cycle;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * each_thread_block_counts_the_number_Of_col_blocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration 32
#pragma unroll 2
    for (int kIter = 0; kIter < K; kIter += 32) {
        // Load matrix A into shared memory, each thread loads 2 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 2; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 2) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 64 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? (matrixA[aRowId * lda + aColId]) : static_cast<MATRIX_A_TYPE>(0.0f);
        }

        __syncthreads();

        // Load matrix B into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll 4
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0.0f);
        }

        __syncwarp();

        // Compute the matrix multiplication
#pragma unroll
        for (int iter = 0; iter < 32; iter += WMMA_K) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter, 32);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter, 32);

                // Convert to TF32
#pragma unroll
                for (int i = 0; i < aFrag.num_elements; ++i) aFrag.x[i] = wmma::__float_to_tf32(aFrag.x[i]);
#pragma unroll
                for (int i = 0; i < bFrag.num_elements; ++i) bFrag.x[i] = wmma::__float_to_tf32(bFrag.x[i]);

                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
//            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
//                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
                matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [256, 1, 1]
// 一个thread block负责一个row panel中的8个col block
__global__ void sddmm_gpu_dense_block_m16n16k8_block256_noSMEM_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                             const UIN N,
                                                                                             const UIN K,
                                                                                             const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                             const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                             const float alpha,
                                                                                             const float beta,
                                                                                             const UIN numNonZeroRow,
                                                                                             const UIN *__restrict__ reorderedRows,
                                                                                             const UIN *__restrict__ reorderedCols,
                                                                                             const UIN *__restrict__ reorderedColOffset,
                                                                                             const UIN *__restrict__ blockRowOffsets,
                                                                                             const UIN *__restrict__ blockValues,
                                                                                             MATRIX_C_TYPE *matrixP) {

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockId = blockIdx.y * each_thread_block_counts_the_number_Of_col_blocks + warpId;
    if (colBlockId >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration 8
    for (int kIter = 0; kIter < K; kIter += 8) {

        // Load matrix A
#pragma unroll
        for (int indexOfFragment = 0; indexOfFragment < aFrag.num_elements; ++indexOfFragment) {
            UIN localRow, localCol;
            calculateMatrixAFragmentCoordinates(laneId, indexOfFragment, localRow, localCol);

            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + localRow;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + localCol;

            aFrag.x[indexOfFragment] = (aRowId < M && aColId < K) ?
                (matrixA[aRowId * lda + aColId]) : static_cast<MATRIX_A_TYPE>(0.0f);

            if (rowPanelId == 0 && colBlockId == 0) {
                printf(
                    "colBlockId = %d, warpId = %d, laneId = %d, index = %d, localRow = %d, localCol = %d, aRowId = %d, aColId = %d, aFrag.x = %f\n",
                    colBlockId,
                    warpId,
                    laneId,
                    indexOfFragment,
                    localRow,
                    localCol,
                    aRowId,
                    aColId,
                    aFrag.x[indexOfFragment]);
            }
        }

        // Load matrix B
#pragma unroll
        for (int indexOfFragment = 0; indexOfFragment < bFrag.num_elements; ++indexOfFragment) {
            UIN localRow, localCol;
            calculateMatrixBFragmentCoordinates(laneId, indexOfFragment, localRow, localCol);

            const UIN bRowId = kIter + localRow;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + localCol;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bFrag.x[indexOfFragment] = (bRowId < K && bColId < N) ?
                matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0.0f);

            if (rowPanelId == 0 && colBlockId == 0) {
                printf(
                    "colBlockId = %d, warpId = %d, laneId = %d, index = %d, localRow = %d, localCol = %d, bRowId = %d, bColId = %d, bFrag.x = %f\n",
                    colBlockId,
                    warpId,
                    laneId,
                    indexOfFragment,
                    localRow,
                    localCol,
                    bRowId,
                    bColId,
                    bFrag.x[indexOfFragment]);
            }
        }

        // Convert to TF32
#pragma unroll
        for (int i = 0; i < aFrag.num_elements; ++i)aFrag.x[i] = wmma::__float_to_tf32(aFrag.x[i]);
#pragma unroll
        for (int i = 0; i < bFrag.num_elements; ++i)bFrag.x[i] = wmma::__float_to_tf32(bFrag.x[i]);

        __syncthreads();

        // Compute the matrix multiplication
        wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);

        __syncthreads();
    }

    // Store the result
#pragma unroll
    for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
        const float c = alpha * cFrag.x[idxOfFragment];

//        if(warpId ==0 && rowPanelId == 0){
//            printf("laneId = %d, idxOfFragment = %d, c = %f\n", laneId, idxOfFragment, c);
//        }

        UIN localRow, localCol;
        calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

        const UIN idxOfMatrixP =
            blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

        // Saved when the value is not 0
        if (idxOfMatrixP != NULL_VALUE) {
            matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
        }

        if (idxOfMatrixP == 0) {
            printf("idxOfMatrixP = %d, c = %f, blockIndex = %d \n",
                   idxOfMatrixP,
                   c,
                   startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol);
        }
    }
}

// m16n16k16
// blockDim: [512, 1, 1]
// 一个thread block负责一个row panel中的16个col block
__global__ void sddmm_gpu_dense_block_m16n16k16_block512_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                       const UIN N,
                                                                                       const UIN K,
                                                                                       const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                       const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                       const float alpha,
                                                                                       const float beta,
                                                                                       const UIN numNonZeroRow,
                                                                                       const UIN *__restrict__ reorderedRows,
                                                                                       const UIN *__restrict__ reorderedCols,
                                                                                       const UIN *__restrict__ reorderedColOffset,
                                                                                       const UIN *__restrict__ blockRowOffsets,
                                                                                       const UIN *__restrict__ blockValues,
                                                                                       MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 16;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_N) * 2;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks) * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K * 2) {
        // Load matrix A into shared memory, each thread loads 1 element, conflict-free access
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + warpId;
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId;

        aTileSMEM[warpId * 32 + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);


        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 2; ++iter) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter * WMMA_K, WMMA_K * 2);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter * WMMA_K, WMMA_K * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateMatrixCFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// blockDim: [256,1,1]
__global__ void sddmm_gpu_sparse_residue_block256_rowPanel_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                         const UIN N,
                                                                                         const UIN K,
                                                                                         const float *__restrict__ matrixA,
                                                                                         const float *__restrict__ matrixB,
                                                                                         const float alpha,
                                                                                         const float beta,
                                                                                         const UIN numNonZeroRow,
                                                                                         const UIN *__restrict__ reorderedRows,
                                                                                         const UIN *__restrict__ sparsePartDataOffsets,
                                                                                         const UIN *__restrict__ sparsePartData,
                                                                                         const UIN *__restrict__ relativeRows,
                                                                                         const UIN *__restrict__ sparsePartColIndices,
                                                                                         float *matrixP) {
    // 线程块中线程数量
    constexpr int numWarpsPerBlock = 8;

    constexpr int kStep = 32;

    constexpr int aTileSMEMSize = WMMA_M * kStep; // 512

    constexpr int eachThreadLoadsTheNumberOfMatrixADatas = aTileSMEMSize / (WARP_SIZE * numWarpsPerBlock); // 2
    constexpr int eachWarpLoadsTheNumberOfMatrixADatas = WARP_SIZE * eachThreadLoadsTheNumberOfMatrixADatas; // 64
    constexpr int eachWarpLoadsTheNumberOfMatrixARows = WMMA_M / numWarpsPerBlock; // 2

    __shared__ float aTileSMEM[aTileSMEMSize];

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN tId = threadIdx.x;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration 32 elements
    for (int kIter = 0; kIter < K; kIter += kStep) {
        // Load matrix A into shared memory, conflict-free access
#pragma unroll
        for (int rowIter = 0; rowIter < eachWarpLoadsTheNumberOfMatrixARows; ++rowIter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) +
                (warpId * eachWarpLoadsTheNumberOfMatrixARows) + rowIter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
#pragma unroll
            for (int colIter = 0; colIter < kStep; colIter += WARP_SIZE) {
                const UIN aColId = kIter + colIter + laneId;

                aTileSMEM[warpId * eachWarpLoadsTheNumberOfMatrixADatas + rowIter * kStep + colIter + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<float>(0);
            }
        }

        __syncthreads();

        // Load matrix B and compute the matrix multiplication
        for (int iter = sparsePartDataOffsets[rowPanelId] + tId;
             iter < sparsePartDataOffsets[rowPanelId + 1];
             iter += blockDim.x) { // Iterate over all the sparse data in the current row panel
            const UIN relativeRow = relativeRows[iter];
            const UIN col = sparsePartColIndices[iter];
            const UIN indexOfMatrixP = sparsePartData[iter];

            float c = 0.0f;
            for (int localKIter = 0; localKIter < kStep; localKIter += 4) {
                const float4 aData = *((float4 *) &aTileSMEM[relativeRow * kStep + localKIter]);
                const float4 bData = *((float4 *) &matrixB[col * ldb + kIter + localKIter]);
                c += aData.x * bData.x + aData.y * bData.y + aData.z * bData.z + aData.w * bData.w;
            }

            matrixP[indexOfMatrixP] += c;
        }

        __syncthreads();
    }
}

// blockDim: [256,1,1]
__global__ void sddmm_gpu_sparse_residue_block256_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                const UIN N,
                                                                                const UIN K,
                                                                                const float *__restrict__ matrixA,
                                                                                const float *__restrict__ matrixB,
                                                                                const float alpha,
                                                                                const float beta,
                                                                                const UIN numNonZeroRow,
                                                                                const UIN *__restrict__ reorderedRows,
                                                                                const UIN *__restrict__ sparsePartDataOffsets,
                                                                                const UIN *__restrict__ sparsePartData,
                                                                                const UIN *__restrict__ relativeRows,
                                                                                const UIN *__restrict__ sparsePartColIndices,
                                                                                float *matrixP) {
    // 线程块中线程数量
    constexpr int numWarpsPerBlock = 8;
    constexpr int numThreadsPerBlock = numWarpsPerBlock * WARP_SIZE; // 256

    constexpr int kStep = 32;

    constexpr int aTileSMEMSize = WMMA_M * kStep; // 512
    constexpr int cSMEMSize = numThreadsPerBlock; // 256

    constexpr int eachThreadLoadsTheNumberOfMatrixADatas = aTileSMEMSize / (WARP_SIZE * numWarpsPerBlock); // 2
    constexpr int eachWarpLoadsTheNumberOfMatrixADatas = WARP_SIZE * eachThreadLoadsTheNumberOfMatrixADatas; // 64
    constexpr int eachWarpLoadsTheNumberOfMatrixARows = WMMA_M / numWarpsPerBlock; // 2

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;

    const UIN startIndexOfSparseDataCurrentBlock = sparsePartDataOffsets[rowPanelId] + blockIdx.y * cSMEMSize;
    const UIN indexBoundaryCurrentRowPanel = sparsePartDataOffsets[rowPanelId + 1];

    // If the current block is out of the boundary, return
    if (startIndexOfSparseDataCurrentBlock >= indexBoundaryCurrentRowPanel) {
        return;
    }

    const UIN index = startIndexOfSparseDataCurrentBlock + threadIdx.x;

    const UIN relativeRow = relativeRows[index];
    const UIN col = sparsePartColIndices[index];

    __shared__ float aTileSMEM[aTileSMEMSize];
    __shared__ float pSMEM[cSMEMSize];

    pSMEM[threadIdx.x] = 0.0f;

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration 32 elements
    for (int kIter = 0; kIter < K; kIter += kStep) {
        // Load matrix A into shared memory, conflict-free access
#pragma unroll 2
        for (int rowIter = 0; rowIter < eachWarpLoadsTheNumberOfMatrixARows; ++rowIter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) +
                (warpId * eachWarpLoadsTheNumberOfMatrixARows) + rowIter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;
            aTileSMEM[warpId * eachWarpLoadsTheNumberOfMatrixADatas + rowIter * kStep + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<float>(0);
        }

        __syncthreads();

        // Load matrix B and compute the matrix multiplication
        if (index < indexBoundaryCurrentRowPanel) {
#pragma unroll 4
            for (int localKIter = 0; localKIter < kStep; localKIter += 4) {
                const float4 aData = *((float4 *) &aTileSMEM[relativeRow * kStep + localKIter]);
                const float4 bData = *((float4 *) &matrixB[col * ldb + kIter + localKIter]);
                pSMEM[threadIdx.x] += aData.x * bData.x + aData.y * bData.y + aData.z * bData.z + aData.w * bData.w;
            }
        }

        __syncthreads();
    }

    if (index < indexBoundaryCurrentRowPanel) {
        matrixP[sparsePartData[index]] = pSMEM[threadIdx.x];
    }
}

// blockDim: [512,1,1]
__global__ void sddmm_gpu_sparse_residue_block512_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                const UIN N,
                                                                                const UIN K,
                                                                                const float *__restrict__ matrixA,
                                                                                const float *__restrict__ matrixB,
                                                                                const float alpha,
                                                                                const float beta,
                                                                                const UIN numNonZeroRow,
                                                                                const UIN *__restrict__ reorderedRows,
                                                                                const UIN *__restrict__ sparsePartDataOffsets,
                                                                                const UIN *__restrict__ sparsePartData,
                                                                                const UIN *__restrict__ relativeRows,
                                                                                const UIN *__restrict__ sparsePartColIndices,
                                                                                float *matrixP) {
    // 线程块中线程数量
    constexpr int numWarpsPerBlock = 16;
    constexpr int numThreadsPerBlock = numWarpsPerBlock * WARP_SIZE; // 512

    constexpr int kStep = 32;

    constexpr int aTileSMEMSize = WMMA_M * kStep; // 512
    constexpr int cSMEMSize = numThreadsPerBlock; // 512

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;

    const UIN startIndexOfSparseDataCurrentBlock = sparsePartDataOffsets[rowPanelId] + blockIdx.y * cSMEMSize;
    const UIN indexBoundaryCurrentRowPanel = sparsePartDataOffsets[rowPanelId + 1];

    // If the current block is out of the boundary, return
    if (startIndexOfSparseDataCurrentBlock >= indexBoundaryCurrentRowPanel) {
        return;
    }

    const UIN index = startIndexOfSparseDataCurrentBlock + threadIdx.x;

    const UIN relativeRow = relativeRows[index];
    const UIN col = sparsePartColIndices[index];

    __shared__ float aTileSMEM[aTileSMEMSize];
    __shared__ float pSMEM[cSMEMSize];

    pSMEM[threadIdx.x] = 0.0f;

    // Loop over K, one iteration 32 elements
    for (int kIter = 0; kIter < K; kIter += kStep) {
        // Load matrix A into shared memory, conflict-free access
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + warpId;
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId;
        aTileSMEM[warpId * kStep + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * K + aColId] : static_cast<float>(0);

        __syncthreads();

        // Load matrix B and compute the matrix multiplication
        if (index < indexBoundaryCurrentRowPanel) {
#pragma unroll 4
            for (int localKIter = 0; localKIter < kStep; localKIter += 4) {
                const float4 aData = *((float4 *) &aTileSMEM[relativeRow * kStep + localKIter]);
                const float4 bData = *((float4 *) &matrixB[col * K + kIter + localKIter]);
                pSMEM[threadIdx.x] += aData.x * bData.x + aData.y * bData.y + aData.z * bData.z + aData.w * bData.w;
            }
        }

        __syncthreads();
    }

    if (index < indexBoundaryCurrentRowPanel) {
        matrixP[sparsePartData[index]] = pSMEM[threadIdx.x];
    }
}

// blockDim: [512,1,1]
__global__ void sddmm_gpu_sparse_residue_block512_shuffle_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                        const UIN N,
                                                                                        const UIN K,
                                                                                        const float *__restrict__ matrixA,
                                                                                        const float *__restrict__ matrixB,
                                                                                        const float alpha,
                                                                                        const float beta,
                                                                                        const UIN numNonZeroRow,
                                                                                        const UIN *__restrict__ reorderedRows,
                                                                                        const UIN *__restrict__ sparsePartDataOffsets,
                                                                                        const UIN *__restrict__ sparsePartData,
                                                                                        const UIN *__restrict__ relativeRows,
                                                                                        const UIN *__restrict__ sparsePartColIndices,
                                                                                        float *matrixP) {
    // 线程块中线程数量
    constexpr int numWarpsPerBlock = 16;
    constexpr int numThreadsPerBlock = numWarpsPerBlock * WARP_SIZE; // 512

    constexpr int kStep = 32;
    constexpr int kStepPerThread = kStep / 2;

    constexpr int aTileSMEMSize = WMMA_M * kStep; // 512
    constexpr int pSMEMSize = numThreadsPerBlock / 2; // 256

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN tId = threadIdx.x;

    const UIN rowPanelId = blockIdx.x;

    const UIN startIndexOfSparseDataCurrentBlock = sparsePartDataOffsets[rowPanelId] + blockIdx.y * pSMEMSize;
    const UIN indexBoundaryCurrentRowPanel = sparsePartDataOffsets[rowPanelId + 1];

    // If the current block is out of the boundary, return
    if (startIndexOfSparseDataCurrentBlock >= indexBoundaryCurrentRowPanel) {
        return;
    }

    const UIN index = startIndexOfSparseDataCurrentBlock + (tId >> 1);

    const UIN relativeRow = relativeRows[index];
    const UIN col = sparsePartColIndices[index];

    __shared__ float aTileSMEM[aTileSMEMSize];
    __shared__ float pSMEM[pSMEMSize];

    if (tId < pSMEMSize) { pSMEM[threadIdx.x] = 0.0f; }

    // 如果tid是偶数则是0; 如果tid是奇数则是1. 确保不同线程并行处理不同的数据段, 避免了线程之间的数据竞争
    const UIN oddOrEven = laneId & 1;

    // Loop over K, one iteration 128 elements
    for (int kIter = 0; kIter < K; kIter += kStep) {

        // Load matrix A into shared memory, conflict-free access
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + warpId;
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId;
        aTileSMEM[warpId * kStep + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * K + aColId] : static_cast<float>(0);
        __syncthreads();

        // Load matrix B and compute the matrix multiplication, 2 thread calculate one element
        float sm1 = 0, sm2 = 0;
        if (index < indexBoundaryCurrentRowPanel) {
#pragma unroll 4
            for (int localKIter = oddOrEven * kStepPerThread; localKIter < (oddOrEven + 1) * kStepPerThread;
                 localKIter += 8) {
                const float4 rtmp1 = *((float4 *) &aTileSMEM[relativeRow * kStep + localKIter]);
                const float4 ctmp1 = *((float4 *) &matrixB[col * K + kIter + localKIter]);
                sm1 += rtmp1.x * ctmp1.x + rtmp1.y * ctmp1.y + rtmp1.z * ctmp1.z + rtmp1.w * ctmp1.w;

                const float4 rtmp2 = *((float4 *) &aTileSMEM[relativeRow * kStep + localKIter + 4]);
                const float4 ctmp2 = *((float4 *) &matrixB[col * K + kIter + localKIter + 4]);
                sm2 += rtmp2.x * ctmp2.x + rtmp2.y * ctmp2.y + rtmp2.z * ctmp2.z + rtmp2.w * ctmp2.w;
            }
        }

        const unsigned mask = (1 << tId) | (1 << (tId ^ 1)); // 只同步相邻线程
        sm1 += __shfl_xor_sync(mask, sm1, 1); // 使用shuffle指令. 使线程0的sm1加到线程1的sm1上, 线程1的sm1加到线程0的sm1上
        sm2 += __shfl_xor_sync(mask, sm2, 1);

        pSMEM[threadIdx.x >> 1] += (sm1 + sm2); // 将分来计算的两个元素加在一起储存到结果矩阵}

        __syncthreads();
    }
    if (index < indexBoundaryCurrentRowPanel) {
        matrixP[sparsePartData[index]] = pSMEM[threadIdx.x >> 1];
    }
}

} // namespace kernel

void sddmm_gpu_rebell(const Matrix<float> &matrixA,
                      const Matrix<float> &matrixB,
                      const float alpha, const float beta,
                      const sparseMatrix::CSR<float> &matrixS,
                      const ReBELL &rebell,
                      sparseMatrix::CSR<float> &matrixP,
                      Logger &logger) {

    // Convert the data type of matrix A and matrix B for use tensor core
    dev::vector<MATRIX_A_TYPE> matrixA_values_convertedType_dev(matrixA.size());
    dev::vector<MATRIX_B_TYPE> matrixB_values_convertedType_dev(matrixB.size());
    {
        dev::vector<float> matrixA_values_dev(matrixA.values());
        dev::vector<float> matrixB_values_dev(matrixB.values());

        const int numThreadPerBlock = 1024;
        kernel::convertDataType<<< (matrixA.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixA.size(), matrixA_values_dev.data(), matrixA_values_convertedType_dev.data());
        kernel::convertDataType<<< (matrixB.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixB.size(), matrixB_values_dev.data(), matrixB_values_convertedType_dev.data());
    }

    // Copy the data from the host to the device
    dev::vector<UIN> reorderedRowIndices_dev(rebell.reorderedRows());
    dev::vector<UIN> reorderedColIndices_dev(rebell.reorderedCols());
    dev::vector<UIN> reorderedColIndicesOffset_dev(rebell.reorderedColOffsets());
    dev::vector<UIN> blockRowOffsets_dev(rebell.blockRowOffsets());
    dev::vector<UIN> blockValues_dev(rebell.blockValues());
    dev::vector<UIN> sparsePartDataOffsets_dev(rebell.sparsePartDataOffsets());
    dev::vector<UIN> sparsePartData_dev(rebell.sparsePartData());
    dev::vector<UIN> relativeRows_dev(rebell.sparsePartRelativeRows());
    dev::vector<UIN> sparsePartColIndices_dev(rebell.sparsePartColIndices());
    dev::vector<float> matrixP_dev(matrixS.values().size(), 0);

    dim3 grid_dense, block_dense;

    const UIN eachThreadBlockCountsTheNumberOfColBlocks = 8;
    block_dense.x = WARP_SIZE * eachThreadBlockCountsTheNumberOfColBlocks;

    // Assign row panel to x-axis of grid, and assign col block to y-axis of grid
    grid_dense.x = rebell.numRowPanels();
    grid_dense.y =
        std::ceil(static_cast<float>(rebell.maxNumDenseColBlocks()) / eachThreadBlockCountsTheNumberOfColBlocks);

    logger.gridDim_dense_ = grid_dense;
    logger.blockDim_dense_ = block_dense;

//    hipStream_t denseBlockStream, sparseRemainderStream;
//    hipStreamCreate(&denseBlockStream);
//    hipStreamCreate(&sparseRemainderStream);

    CudaTimeCalculator timeCalculator_denseBlock, timeCalculator_sparseRemainder;

    timeCalculator_denseBlock.startClock();

#ifdef WMMA_16_16_16
    kernel::sddmm_gpu_dense_block_m16n16k16_block256_matrixA_rowMaj_matrixB_colMaj<<<grid_rebell, block_rebell>>>(matrixS.row(), matrixS.col(), matrixA.col(),
        matrixA_values_convertedType_dev.data(),
        matrixB_values_convertedType_dev.data(),
        alpha, beta,
        rebell.reorderedRows().size(),
        reorderedRowIndices_dev.data(),
        reorderedColIndices_dev.data(),
        reorderedColIndicesOffset_dev.data(),
        blockRowOffsets_dev.data(),
        blockValues_dev.data(),
        matrixP_dev.data());
#endif // WMMA_16_16_16

#ifdef WMMA_16_16_8
    kernel::sddmm_gpu_dense_block_m16n16k8_block256_matrixA_rowMaj_matrixB_colMaj<<<grid_dense, block_dense>>>(matrixS.row(), matrixS.col(), matrixA.col(),
        matrixA_values_convertedType_dev.data(),
        matrixB_values_convertedType_dev.data(),
        alpha, beta,
        rebell.reorderedRows().size(),
        reorderedRowIndices_dev.data(),
        reorderedColIndices_dev.data(),
        reorderedColIndicesOffset_dev.data(),
        blockRowOffsets_dev.data(),
        blockValues_dev.data(),
        matrixP_dev.data());
#endif // WMMA_16_16_8

    timeCalculator_denseBlock.endClock();

    dim3 grid_sparse, block_sparse;
    block_sparse.x = sddmm_sparse_remainder_number_of_thread_per_thread_block;
    grid_sparse.x = rebell.numRowPanels();
    grid_sparse.y = rebell.maxNumSparseColBlocks();

    logger.gridDim_sparse_ = grid_sparse;
    logger.blockDim_sparse_ = block_sparse;

    timeCalculator_sparseRemainder.startClock();

    kernel::sddmm_gpu_sparse_residue_block512_shuffle_matrixA_rowMaj_matrixB_colMaj<<<grid_sparse, block_sparse>>>(matrixS.row(), matrixS.col(), matrixA.col(),
        matrixA_values_convertedType_dev.data(),
        matrixB_values_convertedType_dev.data(),
        alpha, beta,
        rebell.reorderedRows().size(),
        reorderedRowIndices_dev.data(),
        sparsePartDataOffsets_dev.data(),
        sparsePartData_dev.data(),
        relativeRows_dev.data(),
        sparsePartColIndices_dev.data(),
        matrixP_dev.data());

    timeCalculator_sparseRemainder.endClock();

    const float densePartTime = timeCalculator_denseBlock.getTime();
    const float sparsePartTime = timeCalculator_sparseRemainder.getTime();

    printf("denseBlockTime: %f ms, sparseRemainderTime: %f ms\n", densePartTime, sparsePartTime);

    logger.zcx_sddmm_time_ = densePartTime + sparsePartTime;

//    hipStreamDestroy(denseBlockStream);
//    hipStreamDestroy(sparseRemainderStream);

    // Copy the results from the device to the host
    matrixP.setValues() = d2h(matrixP_dev);
}