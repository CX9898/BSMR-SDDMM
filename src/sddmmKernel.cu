#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "sddmmKernel.cuh"
#include "TensorCoreConfig.cuh"
#include "ReBELL.hpp"
#include "CudaTimeCalculator.cuh"

namespace kernel {

using namespace nvcuda;

__global__ void checkFragmentData() {
    constexpr UIN aTileSize = WMMA_M * WMMA_K;
    constexpr UIN bTileSize = WMMA_K * WMMA_N;
    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSize];

    const UIN warpId = threadIdx.x / WARP_SIZE;
    const UIN laneId = threadIdx.x % WARP_SIZE;

    if (warpId == 0 && laneId == 0) {
        for (int i = 0; i < aTileSize; ++i) {
            aTileSMEM[i] = static_cast<half>(i);

        }
        for (int i = 0; i < bTileSize; ++i) {
            bTileSMEM[i] = static_cast<half>(i);
        }
    }

    if (warpId == 0 && laneId == 0) {
        printf("\nmatrix A data : \n\n");
        printf("| |");
        for (int col = 0; col < WMMA_K; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < WMMA_K + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < WMMA_M; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < WMMA_K; ++col) {
                printf("%.0f|", static_cast<float>(aTileSMEM[row * WMMA_K + col]));
            }
            printf("\n");
        }

        printf("\nmatrix B data : \n\n");
        printf("| |");
        for (int col = 0; col < WMMA_N; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < WMMA_N + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < WMMA_K; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < WMMA_N; ++col) {
                printf("%.0f|", static_cast<float>(aTileSMEM[row * WMMA_N + col]));
            }
            printf("\n");
        }
        printf("\n");

        printf("matrix C data : \n\n");
        printf("| |");
        for (int col = 0; col < WMMA_N; ++col) {
            printf("%d|", col);
        }
        printf("\n");

        printf("|");
        for (int i = 0; i < WMMA_N + 1; ++i) {
            printf("-|");
        }
        printf("\n");

        for (int row = 0; row < WMMA_M; ++row) {
            printf("|%d|", row);
            for (int col = 0; col < WMMA_N; ++col) {
                MATRIX_C_TYPE c = 0.0f;
                for (int k = 0; k < WMMA_K; ++k) {
                    const MATRIX_A_TYPE a = aTileSMEM[row * WMMA_K + k];
                    const MATRIX_A_TYPE b = bTileSMEM[k * WMMA_N + col];
                    c += a * b;
                }
                printf("%.0f|", static_cast<float>(c));
            }
            printf("\n");
        }
        printf("\n");
    }

    if (warpId == 0) {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

        wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

        fill_fragment(cFrag, 0.0f);

        wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
        wmma::load_matrix_sync(bFrag, bTileSMEM, WMMA_N);

        wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);

        if (laneId == 0) {
            printf("Fragment data : \n\n");
        }
        for (int laneIdx = 0; laneIdx < WARP_SIZE; ++laneIdx) {
            if (warpId == 0 && laneId == laneIdx) {
                printf("|T%d|", laneId);
                for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                    printf("%.0f|", static_cast<float>(cFrag.x[idxOfFragment]));
                }
                printf("\n");
            }
        }
    }
}

template<typename T>
__global__ void convertDataType(const UIN n, const float *in, T *out) {
    const UIN idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<T>(in[idx]);
//        printf("in[%d] = %f, static_cast<float>out[%d] = %f\n", idx, in[idx], idx, static_cast<float>(out[idx]));
    }
}

template __global__ void convertDataType<int>(const UIN n, const float *in, int *out);
template __global__ void convertDataType<float>(const UIN n, const float *in, float *out);
template __global__ void convertDataType<double>(const UIN n, const float *in, double *out);
template __global__ void convertDataType<half>(const UIN n, const float *in, half *out);

// m16n16k16
// blockDim: [64, 1, 1]
__global__ void sddmm_gpu_rebell_m16n16k16_block64_rowPanel_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                          const UIN N,
                                                                                          const UIN K,
                                                                                          const MATRIX_A_TYPE *matrixA,
                                                                                          const MATRIX_B_TYPE *matrixB,
                                                                                          const UIN numNonZeroRow,
                                                                                          const UIN *reorderedRows,
                                                                                          const UIN *reorderedCols,
                                                                                          const UIN *reorderedColOffset,
                                                                                          const UIN *blockRowOffsets,
                                                                                          const UIN *blockValues,
                                                                                          MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = N;

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K) {
            // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 4; ++iter) {
                const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = kIter + laneId % 16;

                aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;
#pragma unroll
            for (int iter = 0; iter < 8; ++iter) {
                const UIN bRowId = kIter + warpId * 8 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_N);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel
__global__ void sddmm_gpu_rebell_m16n16k16_block64_rowPanel_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                          const UIN N,
                                                                                          const UIN K,
                                                                                          const MATRIX_A_TYPE *matrixA,
                                                                                          const MATRIX_B_TYPE *matrixB,
                                                                                          const UIN numNonZeroRow,
                                                                                          const UIN *reorderedRows,
                                                                                          const UIN *reorderedCols,
                                                                                          const UIN *reorderedColOffset,
                                                                                          const UIN *blockRowOffsets,
                                                                                          const UIN *blockValues,
                                                                                          MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K) {
            // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 4; ++iter) {
                const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = kIter + laneId % 16;

                aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 8; ++iter) {
                const UIN bRowId = kIter + warpId * 8 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel中的2个col block
__global__ void sddmm_gpu_rebell_m16n16k16_block64_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                 const UIN N,
                                                                                 const UIN K,
                                                                                 const MATRIX_A_TYPE *matrixA,
                                                                                 const MATRIX_B_TYPE *matrixB,
                                                                                 const UIN numNonZeroRow,
                                                                                 const UIN *reorderedRows,
                                                                                 const UIN *reorderedCols,
                                                                                 const UIN *reorderedColOffset,
                                                                                 const UIN *blockRowOffsets,
                                                                                 const UIN *blockValues,
                                                                                 MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentIter = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

    const UIN lda = K;
    const UIN ldb = N;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId % 16;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            UIN localRow, localCol;
            calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一个thread block负责一个row panel中的2个col block
__global__ void sddmm_gpu_rebell_m16n16k16_block64_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                 const UIN N,
                                                                                 const UIN K,
                                                                                 const MATRIX_A_TYPE *matrixA,
                                                                                 const MATRIX_B_TYPE *matrixB,
                                                                                 const UIN numNonZeroRow,
                                                                                 const UIN *reorderedRows,
                                                                                 const UIN *reorderedCols,
                                                                                 const UIN *reorderedColOffset,
                                                                                 const UIN *blockRowOffsets,
                                                                                 const UIN *blockValues,
                                                                                 MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 2;

    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentIter = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId % 16;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_K);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            UIN localRow, localCol;
            calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [128, 1, 1]
// 一个thread block负责一个row panel中的4个col block
__global__ void sddmm_gpu_rebell_m16n16k16_block128_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                  const UIN N,
                                                                                  const UIN K,
                                                                                  const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                  const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                  const float alpha, const float beta,
                                                                                  const UIN numNonZeroRow,
                                                                                  const UIN *__restrict__ reorderedRows,
                                                                                  const UIN *__restrict__ reorderedCols,
                                                                                  const UIN *__restrict__ reorderedColOffset,
                                                                                  const UIN *__restrict__ blockRowOffsets,
                                                                                  const UIN *__restrict__ blockValues,
                                                                                  MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 4;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_N) * 2;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks) * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K * 2) {
        // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 4; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 4) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 2; ++iter) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter * WMMA_K, WMMA_K * 2);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter * WMMA_K, WMMA_K * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [256, 1, 1]
// 一个thread block负责一个row panel中的8个col block
__global__ void sddmm_gpu_rebell_m16n16k16_block256_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                  const UIN N,
                                                                                  const UIN K,
                                                                                  const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                  const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                  const float alpha, const float beta,
                                                                                  const UIN numNonZeroRow,
                                                                                  const UIN *__restrict__ reorderedRows,
                                                                                  const UIN *__restrict__ reorderedCols,
                                                                                  const UIN *__restrict__ reorderedColOffset,
                                                                                  const UIN *__restrict__ blockRowOffsets,
                                                                                  const UIN *__restrict__ blockValues,
                                                                                  MATRIX_C_TYPE *matrixP) {
    constexpr int number_of_tiles_loaded_in_one_cycle = 32 / WMMA_K;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_K) * number_of_tiles_loaded_in_one_cycle;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * each_thread_block_counts_the_number_Of_col_blocks)
        * number_of_tiles_loaded_in_one_cycle;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * each_thread_block_counts_the_number_Of_col_blocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K, one iteration WMMA_K * 2
    for (int kIter = 0; kIter < K; kIter += WMMA_K * number_of_tiles_loaded_in_one_cycle) {
        // Load matrix A into shared memory, each thread loads 2 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 2; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 2) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * 64 + iter * WARP_SIZE + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
        }

        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }

        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < number_of_tiles_loaded_in_one_cycle; ++iter) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter * WMMA_K, 32);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter * WMMA_K, 32);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [512, 1, 1]
// 一个thread block负责一个row panel中的16个col block
__global__ void sddmm_gpu_rebell_m16n16k16_block512_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                  const UIN N,
                                                                                  const UIN K,
                                                                                  const MATRIX_A_TYPE *__restrict__ matrixA,
                                                                                  const MATRIX_B_TYPE *__restrict__ matrixB,
                                                                                  const float alpha, const float beta,
                                                                                  const UIN numNonZeroRow,
                                                                                  const UIN *__restrict__ reorderedRows,
                                                                                  const UIN *__restrict__ reorderedCols,
                                                                                  const UIN *__restrict__ reorderedColOffset,
                                                                                  const UIN *__restrict__ blockRowOffsets,
                                                                                  const UIN *__restrict__ blockValues,
                                                                                  MATRIX_C_TYPE *matrixP) {
    constexpr int eachThreadBlockCountsTheNumberOfColBlocks = 16;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_N) * 2;
    constexpr int bTileSMEMSize = (WMMA_K * WMMA_N * eachThreadBlockCountsTheNumberOfColBlocks) * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];

    const UIN colBlockIter = blockIdx.y * eachThreadBlockCountsTheNumberOfColBlocks;
    if (colBlockIter >= numColBlocksCurrentRowPanel) {
        return;
    }

    const UIN colBlockId = colBlockIter + warpId;
    const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

    const UIN startIndexOfReorderedColsCurrentColBlock = reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockId;
    const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K * 2) {
        // Load matrix A into shared memory, each thread loads 1 element, conflict-free access
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + warpId;
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId;

        aTileSMEM[warpId * 32 + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);


        // Load matrix B data into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < 16; ++iter) {
            const UIN bRowId = kIter + laneId;
            const UIN reorderedColIndex = startIndexOfReorderedColsCurrentColBlock + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 512 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        for (int iter = 0; iter < 2; ++iter) {
            if (colBlockId < numColBlocksCurrentRowPanel) {
                wmma::load_matrix_sync(aFrag, aTileSMEM + iter * WMMA_K, WMMA_K * 2);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * 512 + iter * WMMA_K, WMMA_K * 2);
                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }

        __syncthreads();
    }

    // Store the result
    if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
        for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
            const float c = alpha * cFrag.x[idxOfFragment];

            UIN localRow, localCol;
            calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

            const UIN idxOfMatrixP =
                blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

            // Saved when the value is not 0
            if (idxOfMatrixP != NULL_VALUE) {
                matrixP[idxOfMatrixP] = c + beta * matrixP[idxOfFragment];
            }
        }
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 在外部进行K迭代
__global__ void sddmm_gpu_rebell_m16n16k16_outkIter_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                  const UIN N,
                                                                                  const UIN K,
                                                                                  const UIN kIter,
                                                                                  const MATRIX_A_TYPE *matrixA,
                                                                                  const MATRIX_B_TYPE *matrixB,
                                                                                  const UIN numNonZeroRow,
                                                                                  const UIN *reorderedRows,
                                                                                  const UIN *reorderedCols,
                                                                                  const UIN *reorderedColOffset,
                                                                                  const UIN *blockRowOffsets,
                                                                                  const UIN *blockValues,
                                                                                  MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = N;

    // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
    for (int iter = 0; iter < 4; ++iter) {
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId % 16;

        aTileSMEM[warpId * 128 + iter * 32 + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
    }

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_N);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();


        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] += cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 在外部进行K迭代
__global__ void sddmm_gpu_rebell_m16n16k16_outkIter_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                  const UIN N,
                                                                                  const UIN K,
                                                                                  const UIN kIter,
                                                                                  const MATRIX_A_TYPE *matrixA,
                                                                                  const MATRIX_B_TYPE *matrixB,
                                                                                  const UIN numNonZeroRow,
                                                                                  const UIN *reorderedRows,
                                                                                  const UIN *reorderedCols,
                                                                                  const UIN *reorderedColOffset,
                                                                                  const UIN *blockRowOffsets,
                                                                                  const UIN *blockValues,
                                                                                  MATRIX_C_TYPE *matrixP) {
    constexpr int aTileSMEMSize = WMMA_M * WMMA_N;
    constexpr int bTileSMEMSize = WMMA_K * WMMA_N * 2;

    __shared__ MATRIX_A_TYPE aTileSMEM[aTileSMEMSize];
    __shared__ MATRIX_B_TYPE bTileSMEM[bTileSMEMSize];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
    for (int iter = 0; iter < 4; ++iter) {
        const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 8) + (laneId / 16) + (iter * 2);
        const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
        const UIN aColId = kIter + laneId % 16;

        aTileSMEM[warpId * 128 + iter * 32 + laneId] =
            (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
    }

    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {
            const UIN bRowId = kIter + warpId * 8 + iter;
            const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                reorderedCols[reorderedColIndex] : N;

            bTileSMEM[warpId * 256 + iter * 32 + laneId] =
                (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
        }
        __syncthreads();

        // Compute the matrix multiplication
        if (colBlockId < numColBlocksCurrentRowPanel) {
            wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_N);
            wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);
            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }

        __syncthreads();


        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] += cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一次加载4*WMMA_K个元素
__global__ void sddmm_gpu_rebell_4WMMA_K_m16n16k16_matrixA_rowMaj_matrixB_rowMaj(const UIN M,
                                                                                 const UIN N,
                                                                                 const UIN K,
                                                                                 const MATRIX_A_TYPE *matrixA,
                                                                                 const MATRIX_B_TYPE *matrixB,
                                                                                 const UIN numNonZeroRow,
                                                                                 const UIN *reorderedRows,
                                                                                 const UIN *reorderedCols,
                                                                                 const UIN *reorderedColOffset,
                                                                                 const UIN *blockRowOffsets,
                                                                                 const UIN *blockValues,
                                                                                 MATRIX_C_TYPE *matrixP) {
    __shared__ MATRIX_A_TYPE aTileSMEM[(16 * 16) * 4];
    __shared__ MATRIX_B_TYPE bTileSMEM[(16 * 32) * 4];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = N;

    const UIN startIndexOfRowsCurrentRowPanel = rowPanelId * ROW_PANEL_SIZE;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K * 4) {
            // Load matrix A into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 16; ++iter) {
                const UIN reorderedRowIndex = startIndexOfRowsCurrentRowPanel + iter;
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = warpId * WARP_SIZE + laneId;

                aTileSMEM[warpId * 32 + iter * 64 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 32 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 32; ++iter) {
                const UIN bRowId = kIter + warpId * 32 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 1024 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                for (int iter = 0; iter < 4; ++iter) {
                    wmma::load_matrix_sync(aFrag, aTileSMEM + iter * 16, WMMA_K * 4);
                    wmma::load_matrix_sync(bFrag, (bTileSMEM + warpId * WMMA_N) + iter * 512, WMMA_N * 2);
                    wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
                }
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

// m16n16k16
// blockDim: [64, 1, 1]
// 一次加载4*WMMA_K个元素
__global__ void sddmm_gpu_rebell_4WMMA_K_m16n16k16_matrixA_rowMaj_matrixB_colMaj(const UIN M,
                                                                                 const UIN N,
                                                                                 const UIN K,
                                                                                 const MATRIX_A_TYPE *matrixA,
                                                                                 const MATRIX_B_TYPE *matrixB,
                                                                                 const UIN numNonZeroRow,
                                                                                 const UIN *reorderedRows,
                                                                                 const UIN *reorderedCols,
                                                                                 const UIN *reorderedColOffset,
                                                                                 const UIN *blockRowOffsets,
                                                                                 const UIN *blockValues,
                                                                                 MATRIX_C_TYPE *matrixP) {
    __shared__ MATRIX_A_TYPE aTileSMEM[(16 * 16) * 4];
    __shared__ MATRIX_B_TYPE bTileSMEM[(16 * 32) * 4];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE_FRAGMENT, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE_FRAGMENT, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x / WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    const UIN startIndexOfRowsCurrentRowPanel = rowPanelId * ROW_PANEL_SIZE;
    const UIN numColBlocksCurrentRowPanel = blockRowOffsets[rowPanelId + 1] - blockRowOffsets[rowPanelId];
    for (int colBlockIter = 0; colBlockIter < numColBlocksCurrentRowPanel; colBlockIter += 2) {

        // Data needs to be reset to zero before calculating the next column block
        fill_fragment(cFrag, 0.0f);

        const UIN colBlockId = colBlockIter + warpId;
        const UIN startIndexOfBlockValuesCurrentBlock = (blockRowOffsets[rowPanelId] + colBlockId) * BLOCK_SIZE;

        const UIN startIndexOfReorderedColsCurrentIter =
            reorderedColOffset[rowPanelId] + BLOCK_COL_SIZE * colBlockIter;
        const UIN endIndexOfReorderedColsCurrentPanel = reorderedColOffset[rowPanelId + 1];

        const UIN reorderedColIndex = startIndexOfReorderedColsCurrentIter + laneId;

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K * 4) {
            // Load matrix A into shared memory, each thread loads 16 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 16; ++iter) {
                const UIN reorderedRowIndex = startIndexOfRowsCurrentRowPanel + iter;
                const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
                const UIN aColId = warpId * WARP_SIZE + laneId;

                aTileSMEM[warpId * 32 + iter * 64 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<MATRIX_A_TYPE>(0);
            }

            // Load matrix B data into shared memory, each thread loads 32 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 32; ++iter) {
                const UIN bRowId = kIter + warpId * 32 + iter;
                const UIN bColId = reorderedColIndex < endIndexOfReorderedColsCurrentPanel ?
                    reorderedCols[reorderedColIndex] : N;

                bTileSMEM[warpId * 1024 + iter * 32 + laneId] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId + bColId * ldb] : static_cast<MATRIX_B_TYPE>(0);
            }
            __syncthreads();

            // Compute the matrix multiplication
            if (colBlockId < numColBlocksCurrentRowPanel) {
                for (int iter = 0; iter < 4; ++iter) {
                    wmma::load_matrix_sync(aFrag, aTileSMEM + iter * 16, WMMA_K * 4);
                    wmma::load_matrix_sync(bFrag, (bTileSMEM + warpId * WMMA_N) + iter * 512, WMMA_N * 2);
                    wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
                }
            }

            __syncthreads();
        }

        // Store the result
        if (colBlockId < numColBlocksCurrentRowPanel) {
#pragma unroll
            for (int idxOfFragment = 0; idxOfFragment < cFrag.num_elements; ++idxOfFragment) {
                UIN localRow, localCol;
                calculateFragmentCoordinates(laneId, idxOfFragment, localRow, localCol);

                const UIN idxOfMatrixP =
                    blockValues[startIndexOfBlockValuesCurrentBlock + localRow * BLOCK_COL_SIZE + localCol];

                // Saved when the value is not 0
                if (idxOfMatrixP != NULL_VALUE) {
                    matrixP[idxOfMatrixP] = cFrag.x[idxOfFragment];
                }
            }
        }
        __syncthreads();
    }
}

__global__ void sddmm_gpu_sparse_residue(const UIN M, const UIN N, const UIN K,
                                         const float *__restrict__ matrixA,
                                         const float *__restrict__ matrixB,
                                         const float alpha, const float beta,
                                         const UIN numNonZeroRow,
                                         const UIN *__restrict__ reorderedRows,
                                         const UIN *__restrict__ sparseCols,
                                         const UIN *__restrict__ sparseColOffset,
                                         float *matrixP) {
    // 线程块中线程数量
    constexpr int eachThreadLoadsTheNumberOfMatrixADatas =
        (WMMA_M * WMMA_K) / (WARP_SIZE * sddmm_rebell_number_of_warps_per_thread_block);
    constexpr int eachWarpLoadsTheNumberOfMatrixADatas = WARP_SIZE * eachThreadLoadsTheNumberOfMatrixADatas;

    constexpr int aTileSMEMSize = (WMMA_M * WMMA_N) * 2;

    __shared__ float aTileSMEM[aTileSMEMSize];

    const UIN laneId = threadIdx.x & 31;
    const UIN warpId = threadIdx.x >> 5;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = K;

    // Loop over K
    for (int kIter = 0; kIter < K; kIter += WMMA_K * 2) {
        // Load matrix A into shared memory, each thread loads 2 elements, conflict-free access
#pragma unroll
        for (int iter = 0; iter < eachThreadLoadsTheNumberOfMatrixADatas; ++iter) {
            const UIN reorderedRowIndex = (rowPanelId * ROW_PANEL_SIZE) + (warpId * 2) + iter;
            const UIN aRowId = reorderedRowIndex < numNonZeroRow ? reorderedRows[reorderedRowIndex] : M;
            const UIN aColId = kIter + laneId;

            aTileSMEM[warpId * eachWarpLoadsTheNumberOfMatrixADatas + iter * WARP_SIZE + laneId] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<float>(0);
        }

        __syncthreads();

        // Load matrix B data


        // Compute the matrix multiplication

        __syncthreads();
    }
}

} // namespace kernel

void sddmm_gpu_rebell(const Matrix<float> &matrixA,
                      const Matrix<float> &matrixB,
                      const float alpha, const float beta,
                      const sparseMatrix::CSR<float> &matrixS,
                      const ReBELL &rebell,
                      sparseMatrix::CSR<float> &matrixP,
                      Logger &logger) {

    // Convert the data type of matrix A and matrix B for use tensor core
    dev::vector<MATRIX_A_TYPE> matrixA_values_convertedType_dev(matrixA.size());
    dev::vector<MATRIX_B_TYPE> matrixB_values_convertedType_dev(matrixB.size());
    {
        dev::vector<float> matrixA_values_dev(matrixA.values());
        dev::vector<float> matrixB_values_dev(matrixB.values());

        const int numThreadPerBlock = 1024;
        kernel::convertDataType<<< (matrixA.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixA.size(), matrixA_values_dev.data(), matrixA_values_convertedType_dev.data());
        kernel::convertDataType<<< (matrixB.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixB.size(), matrixB_values_dev.data(), matrixB_values_convertedType_dev.data());
    }

    // Copy the data from the host to the device
    dev::vector<UIN> reorderedRowIndices_dev(rebell.reorderedRows());
    dev::vector<UIN> reorderedColIndices_dev(rebell.reorderedCols());
    dev::vector<UIN> reorderedColIndicesOffset_dev(rebell.reorderedColOffsets());
    dev::vector<UIN> blockRowOffsets_dev(rebell.blockRowOffsets());
    dev::vector<UIN> blockValues_dev(rebell.blockValues());

    dev::vector<float> matrixP_dev(matrixS.values());

    dim3 grid, block;

    const UIN eachThreadBlockCountsTheNumberOfColBlocks = 8;
    block.x = WARP_SIZE * eachThreadBlockCountsTheNumberOfColBlocks;

    // Assign row panel to x-axis of grid, and assign col block to y-axis of grid
    grid.x = rebell.numRowPanels();
    grid.y = std::ceil(static_cast<float>(rebell.maxNumColBlocks()) / eachThreadBlockCountsTheNumberOfColBlocks);

    logger.gridDim_ = grid;
    logger.blockDim_ = block;

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();

    if (matrixA.storageOrder() == MatrixStorageOrder::row_major
        && matrixB.storageOrder() == MatrixStorageOrder::row_major) {
//        kernel::sddmm_gpu_rebell_m16n16k16_block128_matrixA_rowMaj_matrixB_rowMaj<<<grid, block>>>(matrixS.row(), matrixS.col(), matrixA.col(),
//            matrixA_values_convertedType_dev.data(),
//            matrixB_values_convertedType_dev.data(),
//            rebell.reorderedRows().size(),
//            reorderedRowIndices_dev.data(),
//            reorderedColIndices_dev.data(),
//            reorderedColIndicesOffset_dev.data(),
//            blockRowOffsets_dev.data(),
//            blockValues_dev.data(),
//            matrixP_dev.data());
    } else if (matrixA.storageOrder() == MatrixStorageOrder::row_major
        && matrixB.storageOrder() == MatrixStorageOrder::col_major) {
        kernel::sddmm_gpu_rebell_m16n16k16_block256_matrixA_rowMaj_matrixB_colMaj<<<grid, block>>>(matrixS.row(), matrixS.col(), matrixA.col(),
            matrixA_values_convertedType_dev.data(),
            matrixB_values_convertedType_dev.data(),
            alpha, beta,
            rebell.reorderedRows().size(),
            reorderedRowIndices_dev.data(),
            reorderedColIndices_dev.data(),
            reorderedColIndicesOffset_dev.data(),
            blockRowOffsets_dev.data(),
            blockValues_dev.data(),
            matrixP_dev.data());
    } else {
        fprintf(stderr, "sddmm_gpu_rebell not support this matrix storage order\n");
    }

    timeCalculator.endClock();

    logger.zcx_sddmm_time_ = timeCalculator.getTime();

    // Copy the results from the device to the host
    matrixP.setValues() = d2h(matrixP_dev);
}

// 在外部进行K迭代
void sddmm_gpu_rebell_out_kIter(const Matrix<float> &matrixA,
                                const Matrix<float> &matrixB,
                                const float alpha, const float beta,
                                const sparseMatrix::CSR<float> &matrixS,
                                const ReBELL &rebell,
                                sparseMatrix::CSR<float> &matrixP,
                                float &time) {

    dev::vector<MATRIX_A_TYPE> matrixA_values_convertedType_dev(matrixA.size());
    dev::vector<MATRIX_B_TYPE> matrixB_values_convertedType_dev(matrixB.size());
    {
        dev::vector<float> matrixA_values_dev(matrixA.values());
        dev::vector<float> matrixB_values_dev(matrixB.values());

        const int numThreadPerBlock = 1024;
        kernel::convertDataType<<< (matrixA.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixA.size(), matrixA_values_dev.data(), matrixA_values_convertedType_dev.data());
        kernel::convertDataType<<< (matrixB.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
            matrixB.size(), matrixB_values_dev.data(), matrixB_values_convertedType_dev.data());
    }

    dev::vector<UIN> reorderedRowIndices_dev(rebell.reorderedRows());
    dev::vector<UIN> reorderedColIndices_dev(rebell.reorderedCols());
    dev::vector<UIN> reorderedColIndicesOffset_dev(rebell.reorderedColOffsets());
    dev::vector<UIN> blockRowOffsets_dev(rebell.blockRowOffsets());
    dev::vector<UIN> blockValues_dev(rebell.blockValues());
    dev::vector<float> matrixP_dev(matrixS.nnz());

    dim3 grid, block;
    block.x = 64;
    grid.x = rebell.numRowPanels();

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();
    // Loop over K
    for (int kIter = 0; kIter < matrixA.col(); kIter += WMMA_K) {
        kernel::sddmm_gpu_rebell_m16n16k16_outkIter_matrixA_rowMaj_matrixB_rowMaj<<<grid, block>>>(matrixS.row(), matrixS.col(), matrixA.col(), kIter,
            matrixA_values_convertedType_dev.data(),
            matrixB_values_convertedType_dev.data(),
            rebell.reorderedRows().size(),
            reorderedRowIndices_dev.data(),
            reorderedColIndices_dev.data(),
            reorderedColIndicesOffset_dev.data(),
            blockRowOffsets_dev.data(),
            blockValues_dev.data(),
            matrixP_dev.data());
    }
    timeCalculator.endClock();

    time = timeCalculator.getTime();

    matrixP.setValues() = d2h(matrixP_dev);
}