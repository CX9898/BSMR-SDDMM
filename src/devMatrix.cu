#include <hip/hip_runtime.h>

#include <fstream>

#include "devMatrix.cuh"
#include "devMatrixKernel.cuh"
#include "parallelAlgorithm.cuh"
//#include "util.hpp"
#include "devVector.cuh"
#include "CudaTimeCalculator.cuh"
#include "checkData.hpp"

//#include <set>

template<typename T>
UIN dev::Matrix<T>::rowOfValueIndex(UIN idx) const {
    if (idx == 0) {
        return 0;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        return idx / leadingDimension_;
    } else {
        return idx % leadingDimension_;
    }
}

template<typename T>
UIN dev::Matrix<T>::colOfValueIndex(UIN idx) const {
    if (idx == 0) {
        return 0;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        return idx % leadingDimension_;
    } else {
        return idx / leadingDimension_;
    }
}

template<typename T>
void dev::Matrix<T>::openTensorCoreMode(MatrixMultiplicationOrder multiplicationOrder) {
    if (tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = true;
    rowBeforeChange_ = row_;
    colBeforeChange_ = col_;

    size_t rowComplement = 0;
    size_t colComplement = 0;
    if (multiplicationOrder == MatrixMultiplicationOrder::left_multiplication) {
        rowComplement = rowBeforeChange_ % WMMA_M == 0 ? 0 : WMMA_M - rowBeforeChange_ % WMMA_M;
        colComplement = colBeforeChange_ % WMMA_K == 0 ? 0 : WMMA_K - colBeforeChange_ % WMMA_K;
    } else {
        rowComplement = rowBeforeChange_ % WMMA_K == 0 ? 0 : WMMA_K - rowBeforeChange_ % WMMA_K;
        colComplement = colBeforeChange_ % WMMA_N == 0 ? 0 : WMMA_N - colBeforeChange_ % WMMA_N;
    }

    row_ = rowBeforeChange_ + rowComplement;
    col_ = colBeforeChange_ + colComplement;
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        leadingDimension_ = col_;
    } else {
        leadingDimension_ = row_;
    }

    if (storageOrder_ == MatrixStorageOrder::row_major) {
        for (size_t rowIter = 0; rowIter < rowBeforeChange_; ++rowIter) {
//            values_.insert(values_.begin() + rowIter * leadingDimension_ + colBeforeChange_, colComplement, 0);
        }
//        values_.insert(values_.end(), rowComplement * col_, 0);
    } else {
        for (size_t colIter = 0; colIter < colBeforeChange_; ++colIter) {
//            values_.insert(values_.begin() + colIter * leadingDimension_ + rowBeforeChange_, rowComplement, 0);
        }
//        values_.insert(values_.end(), colComplement * row_, 0);
    }
}

template<typename T>
void dev::Matrix<T>::closeTensorCoreMode() {
    if (!tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = false;

    const size_t rowComplement = row_ < rowBeforeChange_ ? rowBeforeChange_ - row_ : row_ - rowBeforeChange_;
    const size_t colComplement = col_ < colBeforeChange_ ? colBeforeChange_ - col_ : col_ - colBeforeChange_;

    row_ = rowBeforeChange_;
    col_ = colBeforeChange_;
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        leadingDimension_ = col_;
    } else {
        leadingDimension_ = row_;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        for (size_t rowIter = 0; rowIter < rowBeforeChange_; ++rowIter) {
            const auto curRowBeginIter = values_.begin() + rowIter * leadingDimension_ + colBeforeChange_;
//            values_.erase(curRowBeginIter, curRowBeginIter + colComplement);
        }
    } else {
        for (size_t colIter = 0; colIter < colBeforeChange_; ++colIter) {
            const auto curColBeginIter = values_.begin() + colIter * leadingDimension_ + rowBeforeChange_;
//            values_.erase(curColBeginIter, curColBeginIter + rowComplement);
        }
    }
    values_.resize(row_ * col_);
}

template<typename T>
void dev::SparseMatrix<T>::setValuesFromDenseData(UIN row, UIN col, UIN ld, const dev::vector<T> &denseData) {
    if (row < row_ || col < col_) {
        std::cerr << "row < row_ || col < col_" << std::endl;
    }
    values_.resize(nnz_);

    const int numThreads = 1024;
    const int numBlocks = (nnz_ + numThreads - 1) / numThreads;
    getValuesFromDenseData<<<numBlocks, numThreads>>>(row,
                                                      col,
                                                      nnz_,
                                                      ld,
                                                      rowIndex_.data(),
                                                      colIndex_.data(),
                                                      denseData.data(),
                                                      values_.data());
    hipDeviceSynchronize();
}

//template<typename T>
//void dev::SparseMatrix<T>::initializeFromMatrixMarketFile(const std::string &filePath) {
//    std::ifstream inFile;
//    inFile.open(filePath, std::ios::in); // open file
//    if (!inFile.is_open()) {
//        std::cerr << "Error, MatrixMarket file cannot be opened : " << filePath << std::endl;
//        return;
//    }
//
//    std::cout << "SparseMatrix initialize From MatrixMarket file : " << filePath << std::endl;
//
//    std::string line; // Store the data for each line
//    getline(inFile, line); // First line does not operate
//
//    getline(inFile, line);
//    int wordIter = 0;
//    row_ = std::stoi(::util::iterateOneWordFromLine(line, wordIter));
//    col_ = std::stoi(::util::iterateOneWordFromLine(line, wordIter));
//    nnz_ = std::stoi(::util::iterateOneWordFromLine(line, wordIter));
//
//    if (wordIter < line.size()) {
//        std::cerr << "Error, Matrix Market file " << line << " line format is incorrect!" << std::endl;
//    }
//
//    std::vector<UIN> rowIndex(nnz_);
//    std::vector<UIN> colIndex(nnz_);
//    std::vector<T> values(nnz_);
//
//    UIN idx = 0;
//    while (getline(inFile, line)) {
//        wordIter = 0;
//        const UIN row = std::stoi(util::iterateOneWordFromLine(line, wordIter)) - 1;
//        const UIN col = std::stoi(util::iterateOneWordFromLine(line, wordIter)) - 1;
//        const T val = static_cast<T>(std::stod(util::iterateOneWordFromLine(line, wordIter)));
//
//        if (wordIter < line.size()) {
//            std::cerr << "Error, Matrix Market file " << line << " line format is incorrect!" << std::endl;
//        }
//
//        rowIndex[idx] = row;
//        colIndex[idx] = col;
//        values[idx] = val;
//
//        ++idx;
//    }
//
//    inFile.close();
//
//    h2d(rowIndex_,rowIndex);
//    h2d(colIndex_,colIndex);
//    h2d(values_,values);
//
//    rowBeforeChange_ = row_;
//    colBeforeChange_ = col_;
//}

template<typename T>
void dev::SparseMatrix<T>::openTensorCoreModeForSampled(TensorCoreConfig tensorCoreConfig) {
    if (tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = true;
    rowBeforeChange_ = row_;
    colBeforeChange_ = col_;

    const int rowComplement = rowBeforeChange_ % WMMA_M == 0 ? 0 : WMMA_M - rowBeforeChange_ % WMMA_M;
    const int colComplement = colBeforeChange_ % WMMA_N == 0 ? 0 : WMMA_N - colBeforeChange_ % WMMA_N;
    row_ = rowBeforeChange_ + rowComplement;
    col_ = colBeforeChange_ + colComplement;

    const UIN numTileM = row_ / WMMA_M;
    const UIN numTileN = col_ / WMMA_N;

    const UIN numWarpX = tensorCoreConfig.numWarpX();
    const UIN numWarpY = tensorCoreConfig.numWarpY();
    const UIN numWarps = numWarpX * numWarpY;

    const UIN numThreadsPerBlock = NUMBER_OF_THREADS_PER_BLOCK;
    const UIN numBlocks = (numWarps + numThreadsPerBlock - 1) / numThreadsPerBlock;
    CudaTimeCalculator timeCalculator;


    //////////////////////////////// 1
    dev::vector<UIN> numOfIndexPerWarp(numWarps);
    dev::fill_n(numOfIndexPerWarp.data(), numOfIndexPerWarp.size(), 0);

    timeCalculator.startClock();
    getIndexPerWarp_1<<<numBlocks, NUMBER_OF_THREADS_PER_BLOCK>>>(numWarps,
                                                                  numWarpX,
                                                                  numTileM,
                                                                  numTileN,
                                                                  nnz_,
                                                                  rowIndex_.data(),
                                                                  colIndex_.data(),
                                                                  updateNumOfIndexOperator_1(numOfIndexPerWarp.data()));
    timeCalculator.endClock();
    float getNumIndexPerWarp_1_time = timeCalculator.getTime();
    std::cout << "  getNumIndexPerWarp_1_time : " << getNumIndexPerWarp_1_time << " ms" << std::endl;
    std::vector<UIN> rightNum;
    d2h(rightNum, numOfIndexPerWarp);

    timeCalculator.startClock();

    matrixTileMappedToWarpIndex_.resize(numWarps + 1);
    dev::fill_n(matrixTileMappedToWarpIndex_.data(), 1, 0);
    dev::inclusive_scan(numOfIndexPerWarp.data(),
                        numOfIndexPerWarp.data() + numOfIndexPerWarp.size(),
                        matrixTileMappedToWarpIndex_.data() + 1);
    const UIN numIndexData = matrixTileMappedToWarpIndex_.back_data();
    timeCalculator.endClock();
    float inclusive_scan_time = timeCalculator.getTime();
    std::cout << "  inclusive_scan_time : " << inclusive_scan_time << " ms" << std::endl;

    matrixTileMappedToWarpIndexData_.resize(numIndexData);

    timeCalculator.startClock();
    getIndexPerWarp_1<<<numBlocks, numThreadsPerBlock>>>(numWarps,
                                                         numWarpX,
                                                         numTileM,
                                                         numTileN,
                                                         nnz_,
                                                         rowIndex_.data(),
                                                         colIndex_.data(),
                                                         updateIndexDataPerWarpOperator_1(matrixTileMappedToWarpIndex_.data(),
                                                                                          matrixTileMappedToWarpIndexData_.data()));
    timeCalculator.endClock();
    float getTileIndexDataPerWarp_time = timeCalculator.getTime();
    std::cout << "  getTileIndexDataPerWarp_time : " << getTileIndexDataPerWarp_time << " ms" << std::endl;

    printf(" @@@Method 1 time : %f\n",
           getNumIndexPerWarp_1_time + inclusive_scan_time + getTileIndexDataPerWarp_time);


    //////////////////////////////// 2 error
    {
        dev::vector<UIN> numIndexOfPerWarp_2(numWarps);
        dev::fill_n(numIndexOfPerWarp_2.data(), numIndexOfPerWarp_2.size(), 0);
        timeCalculator.startClock();
        getIndexPerWarp_2<<<numBlocks, numThreadsPerBlock>>>(numWarps,
                                                             numWarpX,
                                                             numTileM,
                                                             numTileN,
                                                             nnz_,
                                                             rowIndex_.data(),
                                                             colIndex_.data(),
                                                             updateNumOfIndexOperator_2(numIndexOfPerWarp_2.data()));

        timeCalculator.endClock();
        float getIndexPerWarp_2_time = timeCalculator.getTime();
        std::cout << "  getIndexPerWarp_2_time : " << getIndexPerWarp_2_time << " ms" << std::endl;

        printf("  check rightNum and numIndexOfPerWarp_2\n");
        const int indexNum = 0;
//    printf("    rightNum[%d] = %d, num2[%d] = %d\n", indexNum, rightNum[indexNum], indexNum, num2[indexNum]);
//        if (!checkData(rightNum, numIndexOfPerWarp_2)) {
//        exit(1);
//        }

        timeCalculator.startClock();

        dev::vector<UIN> matrixTileMappedToWarpIndex_2(numWarps + 1);
        dev::fill_n(matrixTileMappedToWarpIndex_2.data(), 1, 0);
        dev::inclusive_scan(numIndexOfPerWarp_2.data(),
                            numIndexOfPerWarp_2.data() + numIndexOfPerWarp_2.size(),
                            matrixTileMappedToWarpIndex_2.data() + 1);
        const UIN numIndexData_2 = matrixTileMappedToWarpIndex_2.back_data();
        timeCalculator.endClock();
        float inclusive_scan_2_time = timeCalculator.getTime();
        std::cout << "  inclusive_scan_2_time : " << inclusive_scan_2_time << " ms" << std::endl;

        dev::vector<UIN> matrixTileMappedToWarpIndexData_2(numIndexData_2);
        timeCalculator.startClock();
        getIndexPerWarp_2<<<numBlocks, numThreadsPerBlock>>>(numWarps,
                                                             numWarpX,
                                                             numTileM,
                                                             numTileN,
                                                             nnz_,
                                                             rowIndex_.data(),
                                                             colIndex_.data(),
                                                             updateIndexDataPerWarpOperator_2(
                                                                 matrixTileMappedToWarpIndex_2.data(),
                                                                 matrixTileMappedToWarpIndexData_2.data()));

        timeCalculator.endClock();
        float getIndexPerWarp_2_2_time = timeCalculator.getTime();
        std::cout << "  getIndexPerWarp_2_2_time : " << getIndexPerWarp_2_2_time << " ms" << std::endl;

        printf(" @@@Method 2 time : %f\n",
               getIndexPerWarp_2_time + inclusive_scan_2_time + getIndexPerWarp_2_2_time);

//        d2d(matrixTileMappedToWarpIndex_, matrixTileMappedToWarpIndex_2);
//        d2d(matrixTileMappedToWarpIndexData_, matrixTileMappedToWarpIndexData_2);
    }

    //////////////////////////////// 3 OK
    {
        dim3 gridForGetIndex;
        gridForGetIndex.x = (numWarps + NUMBER_OF_THREADS_PER_BLOCK - 1) / NUMBER_OF_THREADS_PER_BLOCK;
        gridForGetIndex.y = (nnz_ + SHARED_MEMORY_SIZE - 1) / SHARED_MEMORY_SIZE;
        dev::vector<UIN> scatteredNumOfIndexPerWarp_3(nnz_ * gridForGetIndex.y);
        dev::fill_n(scatteredNumOfIndexPerWarp_3.data(), scatteredNumOfIndexPerWarp_3.size(), 0);
        timeCalculator.startClock();
        getIndexPerWarp_3<<<gridForGetIndex, NUMBER_OF_THREADS_PER_BLOCK>>>(numWarpX,
                                                                            nnz_,
                                                                            rowIndex_.data(),
                                                                            colIndex_.data(),
                                                                            updateScatteredNumOfIndexOperator_3(
                                                                                scatteredNumOfIndexPerWarp_3.data()));
        timeCalculator.endClock();
        float updateScatteredNumOfIndexOperator_time = timeCalculator.getTime();
        std::cout << "    updateScatteredNumOfIndexOperator_time : " << updateScatteredNumOfIndexOperator_time << " ms"
                  << std::endl;

        timeCalculator.startClock();

        dev::vector<UIN> indexForScatteredNumOfIndex(scatteredNumOfIndexPerWarp_3.size() + 1);
        dev::fill_n(indexForScatteredNumOfIndex.data(), 1, 0);
        dev::inclusive_scan(scatteredNumOfIndexPerWarp_3.data(),
                            scatteredNumOfIndexPerWarp_3.data() + scatteredNumOfIndexPerWarp_3.size(),
                            indexForScatteredNumOfIndex.data() + 1);
        const UIN scatteredNumIndexData = indexForScatteredNumOfIndex.back_data();

        timeCalculator.endClock();
        float inclusive_scan_scattered_time = timeCalculator.getTime();
        std::cout << "    inclusive_scan_scattered_time : " << inclusive_scan_scattered_time
                  << " ms" << std::endl;

        dev::vector<UIN> scatteredMatrixTileMappedToWarpIndexData(scatteredNumIndexData);
        timeCalculator.startClock();

        getIndexPerWarp_3<<<gridForGetIndex, NUMBER_OF_THREADS_PER_BLOCK>>>(numWarpX,
                                                                            nnz_,
                                                                            rowIndex_.data(),
                                                                            colIndex_.data(),
                                                                            updateScatteredIndexDataPerWarpOperator_3(
                                                                                indexForScatteredNumOfIndex.data(),
                                                                                scatteredMatrixTileMappedToWarpIndexData.data()));
        timeCalculator.endClock();
        float updateScatteredIndexDataPerWarpOperator_time = timeCalculator.getTime();
        std::cout << "    updateScatteredIndexDataPerWarpOperator_time : "
                  << updateScatteredIndexDataPerWarpOperator_time
                  << " ms" << std::endl;

        dev::vector<UIN> numIndexPerWarp_3_sum(numWarps);
        dev::fill_n(numIndexPerWarp_3_sum.data(), numIndexPerWarp_3_sum.size(), 0);
        timeCalculator.startClock();
        mergeScatteredNumOfIndex<<<gridForGetIndex.x, NUMBER_OF_THREADS_PER_BLOCK>>>(numWarps, gridForGetIndex.y,
                                                                                     scatteredNumOfIndexPerWarp_3.data(),
                                                                                     numIndexPerWarp_3_sum.data());
        timeCalculator.endClock();
        float mergeNumOfIndexPerWarp_time = timeCalculator.getTime();
        std::cout << "    mergeNumOfIndexPerWarp_time : " << mergeNumOfIndexPerWarp_time << " ms" << std::endl;

        printf("check rightNum and numIndexPerWarp_3_sum\n");
        if (!checkData(rightNum, numIndexPerWarp_3_sum)) {
            exit(1);
        }
        timeCalculator.startClock();

        dev::vector<UIN> matrixTileMappedToWarpIndex_3(numWarps + 1);
        dev::fill_n(matrixTileMappedToWarpIndex_3.data(), 1, 0);
        dev::inclusive_scan(numIndexPerWarp_3_sum.data(),
                            numIndexPerWarp_3_sum.data() + numIndexPerWarp_3_sum.size(),
                            matrixTileMappedToWarpIndex_3.data() + 1);
        const UIN numIndexData_3 = matrixTileMappedToWarpIndex_3.back_data();

        dev::vector<UIN> matrixTileMappedToWarpIndexData_3(numIndexData_3);
        timeCalculator.endClock();
        float inclusive_scan_3_time = timeCalculator.getTime();
        std::cout << "    inclusive_scan_3_time : " << inclusive_scan_3_time
                  << " ms" << std::endl;

        timeCalculator.startClock();
        sortScatteredIndexData<<<gridForGetIndex.x, NUMBER_OF_THREADS_PER_BLOCK>>>(numWarps,
                                                                                   gridForGetIndex.y,
                                                                                   matrixTileMappedToWarpIndex_3.data(),
                                                                                   indexForScatteredNumOfIndex.data(),
                                                                                   scatteredMatrixTileMappedToWarpIndexData.data(),
                                                                                   matrixTileMappedToWarpIndexData_3.data());
        timeCalculator.endClock();
        float sortScatteredIndexData_time = timeCalculator.getTime();
        std::cout << "    sortScatteredIndexData_time : " << sortScatteredIndexData_time << " ms" << std::endl;

        printf(" @@@Method 3 time : %f\n",
               updateScatteredNumOfIndexOperator_time + inclusive_scan_scattered_time
                   + updateScatteredIndexDataPerWarpOperator_time + mergeNumOfIndexPerWarp_time + inclusive_scan_3_time
                   + sortScatteredIndexData_time);

//        d2d(matrixTileMappedToWarpIndex_, matrixTileMappedToWarpIndex_3);
//        d2d(matrixTileMappedToWarpIndexData_, matrixTileMappedToWarpIndexData_3);
    }
    ////////////////////////////////

//    std::cout << hipGetErrorString(hipDeviceSynchronize()) << std::endl;

//    // check
//    std::vector<UIN> rowIndex;
//    d2h(rowIndex, rowIndex_);
//    std::vector<UIN> colIndex;
//    d2h(colIndex, colIndex_);
//
//    std::set<std::pair<size_t, size_t>> rowColSet;
//    for (int idx = 0; idx < nnz_; ++idx) { // 检查是否有相同行列值
//        std::pair<size_t, size_t> rowColPair(rowIndex[idx], colIndex[idx]);
//        if (rowColSet.find(rowColPair) != rowColSet.end()) {
//            std::cout << " 有相同行列值1111???!!!!???!!! "
//                      << "idx = " << idx << ", "
//                      << rowIndex[idx] << " "
//                      << colIndex[idx]
//                      << std::endl;
//            exit(1);
//        }
//        rowColSet.insert(rowColPair);
//    }
//
//    std::vector<UIN> matrixTileIndexData;
//    d2h(matrixTileIndexData, matrixTileIndexData_);
//    for (int idx = 0; idx < matrixTileIndexData_.size(); ++idx) { // 检查是否出现不一样的值
//        std::pair<size_t, size_t> rowColPair(rowIndex[matrixTileIndexData[idx]], colIndex[matrixTileIndexData[idx]]);
//        if (rowColSet.find(rowColPair) == rowColSet.end()) {
//            std::cout << " 出现不一样的值333???!!!!???!!! " << rowIndex[matrixTileIndexData[idx]]
//                      << " " << colIndex[matrixTileIndexData[idx]]
//                      << std::endl;
//            exit(1);
//        }
//    }

}

template<typename T>
void dev::SparseMatrix<T>::closeTensorCoreMode() {
    if (!tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = false;
    row_ = rowBeforeChange_;
    col_ = colBeforeChange_;

    matrixTileMappedToWarpIndex_.clear();
    matrixTileMappedToWarpIndexData_.clear();
}

template
class dev::Matrix<int>;
template
class dev::Matrix<float>;
template
class dev::Matrix<double>;
template
class dev::SparseMatrix<int>;
template
class dev::SparseMatrix<float>;
template
class dev::SparseMatrix<double>;