#include <hip/hip_runtime.h>

#include "devMatrix.cuh"
#include "devMatrixKernel.cuh"
#include "parallelAlgorithm.cuh"

#include <set>

namespace dev {
template<typename T>
size_t Matrix<T>::rowOfValueIndex(size_t idx) const {
    if (idx == 0) {
        return 0;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        return idx / leadingDimension_;
    } else {
        return idx % leadingDimension_;
    }
}

template<typename T>
size_t Matrix<T>::colOfValueIndex(size_t idx) const {
    if (idx == 0) {
        return 0;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        return idx % leadingDimension_;
    } else {
        return idx / leadingDimension_;
    }
}

template<typename T>
void Matrix<T>::openTensorCoreMode(MatrixMultiplicationOrder multiplicationOrder) {
    if (tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = true;
    rowBeforeChange_ = row_;
    colBeforeChange_ = col_;

    size_t rowComplement = 0;
    size_t colComplement = 0;
    if (multiplicationOrder == MatrixMultiplicationOrder::left_multiplication) {
        rowComplement = rowBeforeChange_ % WMMA_M == 0 ? 0 : WMMA_M - rowBeforeChange_ % WMMA_M;
        colComplement = colBeforeChange_ % WMMA_K == 0 ? 0 : WMMA_K - colBeforeChange_ % WMMA_K;
    } else {
        rowComplement = rowBeforeChange_ % WMMA_K == 0 ? 0 : WMMA_K - rowBeforeChange_ % WMMA_K;
        colComplement = colBeforeChange_ % WMMA_N == 0 ? 0 : WMMA_N - colBeforeChange_ % WMMA_N;
    }

    row_ = rowBeforeChange_ + rowComplement;
    col_ = colBeforeChange_ + colComplement;
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        leadingDimension_ = col_;
    } else {
        leadingDimension_ = row_;
    }

    if (storageOrder_ == MatrixStorageOrder::row_major) {
        for (size_t rowIter = 0; rowIter < rowBeforeChange_; ++rowIter) {
//            values_.insert(values_.begin() + rowIter * leadingDimension_ + colBeforeChange_, colComplement, 0);
        }
//        values_.insert(values_.end(), rowComplement * col_, 0);
    } else {
        for (size_t colIter = 0; colIter < colBeforeChange_; ++colIter) {
//            values_.insert(values_.begin() + colIter * leadingDimension_ + rowBeforeChange_, rowComplement, 0);
        }
//        values_.insert(values_.end(), colComplement * row_, 0);
    }
}

template<typename T>
void Matrix<T>::closeTensorCoreMode() {
    if (!tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = false;

    const size_t rowComplement = row_ < rowBeforeChange_ ? rowBeforeChange_ - row_ : row_ - rowBeforeChange_;
    const size_t colComplement = col_ < colBeforeChange_ ? colBeforeChange_ - col_ : col_ - colBeforeChange_;

    row_ = rowBeforeChange_;
    col_ = colBeforeChange_;
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        leadingDimension_ = col_;
    } else {
        leadingDimension_ = row_;
    }
    if (storageOrder_ == MatrixStorageOrder::row_major) {
        for (size_t rowIter = 0; rowIter < rowBeforeChange_; ++rowIter) {
            const auto curRowBeginIter = values_.begin() + rowIter * leadingDimension_ + colBeforeChange_;
//            values_.erase(curRowBeginIter, curRowBeginIter + colComplement);
        }
    } else {
        for (size_t colIter = 0; colIter < colBeforeChange_; ++colIter) {
            const auto curColBeginIter = values_.begin() + colIter * leadingDimension_ + rowBeforeChange_;
//            values_.erase(curColBeginIter, curColBeginIter + rowComplement);
        }
    }
    values_.resize(row_ * col_);
}

template<typename T>
void SparseMatrix<T>::openTensorCoreModeForSampled(TensorCoreConfig tensorCoreConfig) {
    if (tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = true;
    rowBeforeChange_ = row_;
    colBeforeChange_ = col_;

    const int rowComplement = rowBeforeChange_ % WMMA_M == 0 ? 0 : WMMA_M - rowBeforeChange_ % WMMA_M;
    const int colComplement = colBeforeChange_ % WMMA_N == 0 ? 0 : WMMA_N - colBeforeChange_ % WMMA_N;
    row_ = rowBeforeChange_ + rowComplement;
    col_ = colBeforeChange_ + colComplement;

    const UIN numTileM = row_ / WMMA_M;
    const UIN numTileN = col_ / WMMA_N;

    const UIN numWarpX = tensorCoreConfig.numWarpX();
    const UIN numWarpY = tensorCoreConfig.numWarpY();
    const UIN numWarps = numWarpX * numWarpY;

    dev::vector<UIN> numIndexPerWarp(numWarps);
    const UIN numThreadsPerBlock = 1024;
    const UIN numBlocks = (numWarps + numThreadsPerBlock - 1) / numThreadsPerBlock;
    getNumIndexPerWarp<<<numBlocks, numThreadsPerBlock>>>(numWarps,
                                                          numWarpX,
                                                          numTileM,
                                                          numTileN,
                                                          nnz_,
                                                          rowIndex_.data(),
                                                          colIndex_.data(),
                                                          numIndexPerWarp.data());
    matrixTileIndex_.resize(numWarps + 1);
    dev::fill_n(matrixTileIndex_.data(), 1, 0);
    dev::inclusive_scan(numIndexPerWarp.data(),
                        numIndexPerWarp.data() + numIndexPerWarp.size(),
                        matrixTileIndex_.data() + 1);
    const UIN numIndexData = matrixTileIndex_.back_data();
    matrixTileIndexData_.resize(numIndexData);
    getTileIndexDataPerWarp<<<numBlocks, numThreadsPerBlock>>>(numWarps,
                                                               numWarpX,
                                                               numTileM,
                                                               numTileN,
                                                               nnz_,
                                                               rowIndex_.data(),
                                                               colIndex_.data(),
                                                               matrixTileIndex_.data(),
                                                               matrixTileIndexData_.data());

//    // check
//    std::vector<UIN> rowIndex;
//    d2h(rowIndex, rowIndex_);
//    std::vector<UIN> colIndex;
//    d2h(colIndex, colIndex_);
//
//    std::set<std::pair<size_t, size_t>> rowColSet;
//    for (int idx = 0; idx < nnz_; ++idx) { // 检查是否有相同行列值
//        std::pair<size_t, size_t> rowColPair(rowIndex[idx], colIndex[idx]);
//        if (rowColSet.find(rowColPair) != rowColSet.end()) {
//            std::cout << " 有相同行列值1111???!!!!???!!! "
//                      << "idx = " << idx << ", "
//                      << rowIndex[idx] << " "
//                      << colIndex[idx]
//                      << std::endl;
//            exit(1);
//        }
//        rowColSet.insert(rowColPair);
//    }
//
//    std::vector<UIN> matrixTileIndexData;
//    d2h(matrixTileIndexData, matrixTileIndexData_);
//    for (int idx = 0; idx < matrixTileIndexData_.size(); ++idx) { // 检查是否出现不一样的值
//        std::pair<size_t, size_t> rowColPair(rowIndex[matrixTileIndexData[idx]], colIndex[matrixTileIndexData[idx]]);
//        if (rowColSet.find(rowColPair) == rowColSet.end()) {
//            std::cout << " 出现不一样的值333???!!!!???!!! " << rowIndex[matrixTileIndexData[idx]]
//                      << " " << colIndex[matrixTileIndexData[idx]]
//                      << std::endl;
//            exit(1);
//        }
//    }

}

template<typename T>
void SparseMatrix<T>::closeTensorCoreMode() {
    if (!tensorCoreMode_) {
        return;
    }
    tensorCoreMode_ = false;
    row_ = rowBeforeChange_;
    col_ = colBeforeChange_;

    matrixTileIndex_.clear();
    matrixTileIndexData_.clear();
}

template
class Matrix<int>;
template
class Matrix<float>;
template
class Matrix<double>;
template
class SparseMatrix<int>;
template
class SparseMatrix<float>;
template
class SparseMatrix<double>;
} // namespace dev