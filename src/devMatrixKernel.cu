#include "hip/hip_runtime.h"
#include <cstdio>

#include "devMatrixKernel.cuh"

template<typename T>
__global__ void getValuesFromDenseData(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                       const UIN *rowIndex, const UIN *colIndex,
                                       const T *denseData, T *output) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

}

template __global__ void getValuesFromDenseData<int>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                     const UIN *rowIndex, const UIN *colIndex,
                                                     const int *denseData, int *output);
template __global__ void getValuesFromDenseData<float>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                       const UIN *rowIndex, const UIN *colIndex,
                                                       const float *denseData, float *output);
template __global__ void getValuesFromDenseData<double>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                        const UIN *rowIndex, const UIN *colIndex,
                                                        const double *denseData, double *output);

__global__ void getNumIndexPerWarp_1(const UIN size, const UIN numWarpX,
                                     const UIN numTileM, const UIN numTileN,
                                     const UIN nnz,
                                     const UIN *rowIndex,
                                     const UIN *colIndex,
                                     UIN *numIndexPerWarp) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= size) {
        return;
    }

    const int curWarpX = tid % numWarpX;
    const int curWarpY = tid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (tid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (tid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (tid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (tid % numWarpX + 1) * WMMA_N;

    UIN num = 0;
    for (int idx = 0; idx < nnz; ++idx) {
        const UIN curRow = rowIndex[idx];
        const UIN curCol = colIndex[idx];
        if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
            curCol >= colBeginOfTile && curCol < colEndOfTile) {
            ++num;
        }
    }

    numIndexPerWarp[tid] = num;
}

__global__ void getTileIndexDataPerWarp(const UIN size, const UIN numWarpX,
                                        const UIN numTileM, const UIN numTileN,
                                        const UIN nnz,
                                        const UIN *rowIndex,
                                        const UIN *colIndex,
                                        const UIN *matrixTileMappedToWarpIndex,
                                        UIN *matrixTileMappedToWarpIndexData) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= size) {
        return;
    }
    const int curWarpX = tid % numWarpX;
    const int curWarpY = tid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (tid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (tid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (tid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (tid % numWarpX + 1) * WMMA_N;

    const UIN beginIdx = matrixTileMappedToWarpIndex[tid];

    UIN count = 0;
    for (int idx = 0; idx < nnz; ++idx) {
        const UIN curRow = rowIndex[idx];
        const UIN curCol = colIndex[idx];
        if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
            curCol >= colBeginOfTile && curCol < colEndOfTile) {
            matrixTileMappedToWarpIndexData[beginIdx + count] = idx;
            ++count;
        }
    }

}

__global__ void getNumIndexPerWarp_2(const UIN size, const UIN numWarpX,
                                     const UIN numTileM, const UIN numTileN,
                                     const UIN nnz,
                                     const UIN *rowIndex,
                                     const UIN *colIndex,
                                     UIN *numIndexPerWarp) {
    const int globalBid = blockDim.x * blockIdx.x;
    const int globalTid = globalBid + threadIdx.x;
//    if (globalTid >= size) {
//        return;
//    }

    const int curWarpX = globalTid % numWarpX;
    const int curWarpY = globalTid / numWarpX;
//    if (curWarpX > numTileN || curWarpY > numTileM) {
//        return;
//    }

    const UIN rowBeginOfTile = (globalTid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (globalTid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (globalTid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (globalTid % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[SharedMemorySize];
    __shared__ UIN colIndexShared[SharedMemorySize];

    const int sharedBeginIdx = threadIdx.x * NumberOfOperationsOnSharedByOneThread;
    const int sharedEndIdx = sharedBeginIdx + NumberOfOperationsOnSharedByOneThread;
//    printf(" sharedBeginIdx = %d, sharedEndIdx = %d\n", sharedBeginIdx, sharedEndIdx);

    UIN num = 0;
    for (int loop = 0; loop < nnz; loop += SharedMemorySize) {

#pragma unroll
        for (int mtxIdx = loop + sharedBeginIdx, sharedIdx = sharedBeginIdx;
             mtxIdx < nnz && sharedIdx < sharedEndIdx;
             ++sharedIdx, ++mtxIdx) {
            rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
            colIndexShared[sharedIdx] = colIndex[mtxIdx];
        }
        __syncthreads();

        const UIN sharedLoopEnd = nnz - loop;

#pragma unroll
        for (int sharedIdx = 0; sharedIdx < SharedMemorySize && sharedIdx < sharedLoopEnd; ++sharedIdx) {
            const UIN curRow = rowIndexShared[sharedIdx];
            const UIN curCol = colIndexShared[sharedIdx];
            if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
                curCol >= colBeginOfTile && curCol < colEndOfTile) {
                ++num;
            }
        }
    }
    numIndexPerWarp[globalTid] = num;
}

__global__ void getTileIndexDataPerWarp_2(const UIN size, const UIN numWarpX,
                                          const UIN numTileM, const UIN numTileN,
                                          const UIN nnz,
                                          const UIN *rowIndex,
                                          const UIN *colIndex,
                                          const UIN *matrixTileMappedToWarpIndex,
                                          UIN *matrixTileMappedToWarpIndexData) {
    const int globalBid = blockDim.x * blockIdx.x;
    const int globalTid = globalBid + threadIdx.x;
//    if (globalTid >= size) {
//        return;
//    }

    const int curWarpX = globalTid % numWarpX;
    const int curWarpY = globalTid / numWarpX;
//    if (curWarpX > numTileN || curWarpY > numTileM) {
//        return;
//    }

    const UIN rowBeginOfTile = (globalTid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (globalTid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (globalTid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (globalTid % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[SharedMemorySize];
    __shared__ UIN colIndexShared[SharedMemorySize];

    const int beginIdxInShared = threadIdx.x * NumberOfOperationsOnSharedByOneThread;
    const int endIdxInShared = beginIdxInShared + NumberOfOperationsOnSharedByOneThread;

    const UIN beginIdx = matrixTileMappedToWarpIndex[globalTid];

    int count = 0;
    for (int loop = 0; loop < nnz; loop += SharedMemorySize) {

        for (int mtxIdx = loop + beginIdxInShared, sharedIdx = beginIdxInShared;
             mtxIdx < nnz && sharedIdx < endIdxInShared;
             ++sharedIdx, ++mtxIdx) {
            rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
            colIndexShared[sharedIdx] = colIndex[mtxIdx];
        }
        __syncthreads();

        const UIN sharedLoopEnd = nnz - loop;

#pragma unroll
        for (int sharedIdx = 0; sharedIdx < SharedMemorySize && sharedIdx < sharedLoopEnd; ++sharedIdx) {
            const UIN curRow = rowIndexShared[sharedIdx];
            const UIN curCol = colIndexShared[sharedIdx];
            if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
                curCol >= colBeginOfTile && curCol < colEndOfTile) {
                matrixTileMappedToWarpIndexData[beginIdx + count] = loop + sharedIdx;
                ++count;
            }
        }
    }
}

__global__ void getNumIndexPerWarp_3(const UIN numWarpX,
                                     const UIN nnz,
                                     const UIN *rowIndex,
                                     const UIN *colIndex,
                                     UIN *numIndexPerWarp) {

    const UIN warpIdInSDDMM = blockIdx.x * blockDim.x + threadIdx.x;
    const UIN blockIdY = blockIdx.y;

    const UIN rowBeginOfTile = (warpIdInSDDMM / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (warpIdInSDDMM / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (warpIdInSDDMM % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (warpIdInSDDMM % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[SharedMemorySize];
    __shared__ UIN colIndexShared[SharedMemorySize];

    const int sharedBeginIdx = threadIdx.x * NumberOfOperationsOnSharedByOneThread;
    const int sharedEndIdx = sharedBeginIdx + NumberOfOperationsOnSharedByOneThread;
//    printf(" sharedBeginIdx = %d, sharedEndIdx = %d\n", sharedBeginIdx, sharedEndIdx);

    const UIN sparseMatrixDataInThisBlock = (blockIdx.y * SharedMemorySize);

#pragma unroll
    for (int mtxIdx = sparseMatrixDataInThisBlock + sharedBeginIdx, sharedIdx = sharedBeginIdx;
         mtxIdx < nnz && sharedIdx < sharedEndIdx;
         ++sharedIdx, ++mtxIdx) {
        rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
        colIndexShared[sharedIdx] = colIndex[mtxIdx];
    }
    __syncthreads();

    const UIN sharedLoopEnd = nnz - sparseMatrixDataInThisBlock;

    UIN num = 0;
#pragma unroll
    for (int sharedIdx = 0; sharedIdx < SharedMemorySize && sharedIdx < sharedLoopEnd; ++sharedIdx) {
        const UIN curRow = rowIndexShared[sharedIdx];
        const UIN curCol = colIndexShared[sharedIdx];
        if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
            curCol >= colBeginOfTile && curCol < colEndOfTile) {
            ++num;
        }
    }
    numIndexPerWarp[warpIdInSDDMM * gridDim.y + threadIdx.x] = num;

}

__global__ void getTileIndexDataPerWarp_3(const UIN size, const UIN numWarpX,
                                          const UIN numTileM, const UIN numTileN,
                                          const UIN nnz,
                                          const UIN *rowIndex,
                                          const UIN *colIndex,
                                          const UIN *matrixTileMappedToWarpIndex,
                                          UIN *matrixTileMappedToWarpIndexData) {
    const int globalBid = blockDim.x * blockIdx.x;
    const int globalTid = globalBid + threadIdx.x;
//    if (globalTid >= size) {
//        return;
//    }

    const int curWarpX = globalTid % numWarpX;
    const int curWarpY = globalTid / numWarpX;
//    if (curWarpX > numTileN || curWarpY > numTileM) {
//        return;
//    }

    const UIN rowBeginOfTile = (globalTid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (globalTid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (globalTid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (globalTid % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[SharedMemorySize];
    __shared__ UIN colIndexShared[SharedMemorySize];

    const int beginIdxInShared = threadIdx.x * NumberOfOperationsOnSharedByOneThread;
    const int endIdxInShared = beginIdxInShared + NumberOfOperationsOnSharedByOneThread;

    const UIN beginIdx = matrixTileMappedToWarpIndex[globalTid];

    int count = 0;
    for (int loop = 0; loop < nnz; loop += SharedMemorySize) {

        for (int mtxIdx = loop + beginIdxInShared, sharedIdx = beginIdxInShared;
             mtxIdx < nnz && sharedIdx < endIdxInShared;
             ++sharedIdx, ++mtxIdx) {
            rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
            colIndexShared[sharedIdx] = colIndex[mtxIdx];
        }
        __syncthreads();

        const UIN sharedLoopEnd = nnz - loop;

#pragma unroll
        for (int sharedIdx = 0; sharedIdx < SharedMemorySize && sharedIdx < sharedLoopEnd; ++sharedIdx) {
            const UIN curRow = rowIndexShared[sharedIdx];
            const UIN curCol = colIndexShared[sharedIdx];
            if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
                curCol >= colBeginOfTile && curCol < colEndOfTile) {
                matrixTileMappedToWarpIndexData[beginIdx + count] = loop + sharedIdx;
                ++count;
            }
        }
    }
}