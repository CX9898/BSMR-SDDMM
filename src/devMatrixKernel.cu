#include "hip/hip_runtime.h"
#include "devMatrixKernel.cuh"

template<typename T>
__global__ void getValuesFromDenseData(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                       const UIN *rowIndex, const UIN *colIndex,
                                       const T *denseData, T *output) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;

}

template __global__ void getValuesFromDenseData<int>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                     const UIN *rowIndex, const UIN *colIndex,
                                                     const int *denseData, int *output);
template __global__ void getValuesFromDenseData<float>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                       const UIN *rowIndex, const UIN *colIndex,
                                                       const float *denseData, float *output);
template __global__ void getValuesFromDenseData<double>(const UIN row, const UIN col, const UIN nnz, const UIN ld,
                                                        const UIN *rowIndex, const UIN *colIndex,
                                                        const double *denseData, double *output);

__global__ void getNumIndexPerWarp(const UIN size, const UIN numWarpX,
                                   const UIN numTileM, const UIN numTileN,
                                   const UIN nnz,
                                   const UIN *rowIndex,
                                   const UIN *colIndex,
                                   UIN *numIndexPerWarp) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= size) {
        return;
    }

    const int curWarpX = tid % numWarpX;
    const int curWarpY = tid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (tid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (tid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (tid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (tid % numWarpX + 1) * WMMA_N;

    UIN num = 0;
    for (int idx = 0; idx < nnz; ++idx) {
        const UIN curRow = rowIndex[idx];
        const UIN curCol = colIndex[idx];
        if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
            curCol >= colBeginOfTile && curCol < colEndOfTile) {
            ++num;
        }
    }

    numIndexPerWarp[tid] = num;
}

__global__ void getTileIndexDataPerWarp(const UIN size, const UIN numWarpX,
                                        const UIN numTileM, const UIN numTileN,
                                        const UIN nnz,
                                        const UIN *rowIndex,
                                        const UIN *colIndex,
                                        const UIN *matrixTileMappedToWarpIndex,
                                        UIN *matrixTileMappedToWarpIndexData) {
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid >= size) {
        return;
    }
    const int curWarpX = tid % numWarpX;
    const int curWarpY = tid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (tid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (tid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (tid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (tid % numWarpX + 1) * WMMA_N;

    const UIN beginIdx = matrixTileMappedToWarpIndex[tid];

    UIN count = 0;
    for (int idx = 0; idx < nnz; ++idx) {
        const UIN curRow = rowIndex[idx];
        const UIN curCol = colIndex[idx];
        if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
            curCol >= colBeginOfTile && curCol < colEndOfTile) {
            matrixTileMappedToWarpIndexData[beginIdx + count] = idx;
            ++count;
        }
    }

}

__global__ void getNumIndexPerWarp_2(const UIN size, const UIN numWarpX,
                                     const UIN numTileM, const UIN numTileN,
                                     const UIN nnz,
                                     const UIN *rowIndex,
                                     const UIN *colIndex,
                                     UIN *numIndexPerWarp) {
    const int globalBid = blockDim.x * blockIdx.x;
    const int globalTid = globalBid + threadIdx.x;
    if (globalTid >= size) {
        return;
    }

    const int curWarpX = globalTid % numWarpX;
    const int curWarpY = globalTid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (globalTid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (globalTid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (globalTid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (globalTid % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[numOfMemoryReadInOneLoop];
    __shared__ UIN colIndexShared[numOfMemoryReadInOneLoop];

    const int beginIdxInShared = threadIdx.x * numOfMemoryReadInOneLoopOfOneThread;
    const int endIdxInShared = beginIdxInShared + numOfMemoryReadInOneLoopOfOneThread;

    UIN num = 0;
    for (int loop = 0; loop < nnz; loop += numOfMemoryReadInOneLoop) {

        for (int mtxIdx = loop + beginIdxInShared, sharedIdx = beginIdxInShared;
             mtxIdx < nnz && sharedIdx < endIdxInShared;
             ++sharedIdx, ++mtxIdx) {
            rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
            colIndexShared[sharedIdx] = colIndex[mtxIdx];
        }
        __syncthreads();

        for (int sharedIdx = 0; sharedIdx < numOfMemoryReadInOneLoop; ++sharedIdx) {
            const UIN curRow = rowIndexShared[sharedIdx];
            const UIN curCol = colIndexShared[sharedIdx];
            if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
                curCol >= colBeginOfTile && curCol < colEndOfTile) {
                ++num;
            }
        }
    }
    numIndexPerWarp[globalTid] = num;
}

__global__ void getTileIndexDataPerWarp_2(const UIN size, const UIN numWarpX,
                                          const UIN numTileM, const UIN numTileN,
                                          const UIN nnz,
                                          const UIN *rowIndex,
                                          const UIN *colIndex,
                                          const UIN *matrixTileMappedToWarpIndex,
                                          UIN *matrixTileMappedToWarpIndexData) {
    const int globalBid = blockDim.x * blockIdx.x;
    const int globalTid = globalBid + threadIdx.x;
    if (globalTid >= size) {
        return;
    }

    const int curWarpX = globalTid % numWarpX;
    const int curWarpY = globalTid / numWarpX;
    if (curWarpX > numTileN || curWarpY > numTileM) {
        return;
    }

    const UIN rowBeginOfTile = (globalTid / numWarpX) * WMMA_M;
    const UIN rowEndOfTile = (globalTid / numWarpX + 1) * WMMA_M;
    const UIN colBeginOfTile = (globalTid % numWarpX) * WMMA_N;
    const UIN colEndOfTile = (globalTid % numWarpX + 1) * WMMA_N;

    __shared__ UIN rowIndexShared[numOfMemoryReadInOneLoop];
    __shared__ UIN colIndexShared[numOfMemoryReadInOneLoop];

    const int beginIdxInShared = threadIdx.x * numOfMemoryReadInOneLoopOfOneThread;
    const int endIdxInShared = beginIdxInShared + numOfMemoryReadInOneLoopOfOneThread;

    const UIN beginIdx = matrixTileMappedToWarpIndex[globalTid];

    int count = 0;
    for (int loop = 0; loop < nnz; loop += numOfMemoryReadInOneLoop) {

        for (int mtxIdx = loop + beginIdxInShared, sharedIdx = beginIdxInShared;
             mtxIdx < nnz && sharedIdx < endIdxInShared;
             ++sharedIdx, ++mtxIdx) {
            rowIndexShared[sharedIdx] = rowIndex[mtxIdx];
            colIndexShared[sharedIdx] = colIndex[mtxIdx];
        }
        __syncthreads();

        for (int sharedIdx = 0; sharedIdx < numOfMemoryReadInOneLoop; ++sharedIdx) {
            const UIN curRow = rowIndexShared[sharedIdx];
            const UIN curCol = colIndexShared[sharedIdx];
            if (curRow >= rowBeginOfTile && curRow < rowEndOfTile &&
                curCol >= colBeginOfTile && curCol < colEndOfTile) {
                matrixTileMappedToWarpIndexData[beginIdx + count] = loop + sharedIdx;
                ++count;
            }
        }
    }
}