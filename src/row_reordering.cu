#include "hip/hip_runtime.h"
#include <cmath>
#include <omp.h>
#include <numeric>
#include <algorithm>

#include "reordering.hpp"
#include "parallelAlgorithm.cuh"

#define COL_BLOCK_SIZE 32

void encoding(const sparseDataType::CSR<float> &matrix, std::vector<std::vector<UIN>> &encodings) {
    const int colBlock = std::ceil(static_cast<float>(matrix.col_) / COL_BLOCK_SIZE);
    encodings.resize(matrix.row_);
#pragma omp parallel for dynamic
    for (int row = 0; row < matrix.row_; ++row) {
        encodings[row].resize(colBlock);
        for (int idx = matrix.rowOffsets_[row]; idx < matrix.rowOffsets_[row + 1]; ++idx) {
            const int col = matrix.colIndices_[idx];
            ++encodings[row][col / COL_BLOCK_SIZE];
        }
    }
}

void calculateDispersion(const UIN col,
                         const std::vector<std::vector<UIN>> &encodings,
                         std::vector<UIN> &dispersions) {
#pragma omp parallel for dynamic
    for (int row = 0; row < encodings.size(); ++row) {
        UIN numOfNonZeroColBlocks = 0;
        UIN zeroFillings = 0;
        for (int colBlockIdx = 0; colBlockIdx < encodings[row].size(); ++colBlockIdx) {
            const UIN numOfNonZeroCols = encodings[row][colBlockIdx];
            if (numOfNonZeroCols != 0) {
                ++numOfNonZeroColBlocks;
                zeroFillings += block_col_size - numOfNonZeroCols;
            }
        }
        dispersions[row] = COL_BLOCK_SIZE * numOfNonZeroColBlocks + zeroFillings;
    }
}

// return similarity between two encodings
float clusterComparison(const std::vector<UIN> &encoding_rep, const std::vector<UIN> &encoding_cmp) {
    UIN sum_of_squares_rep = 0;
    UIN sum_of_squares_cmp = 0;
    for (int idx = 0; idx < encoding_rep.size(); ++idx) {
        sum_of_squares_rep += encoding_rep[idx] * encoding_rep[idx];
        sum_of_squares_cmp += encoding_cmp[idx] * encoding_cmp[idx];
    }
    if (sum_of_squares_rep == 0 && sum_of_squares_cmp == 0) {
        return 1.0f;
    } else if ((sum_of_squares_rep == 0 || sum_of_squares_cmp == 0)) {
        return 0.0f;
    }
    float norm_rep = sqrt((float) sum_of_squares_rep);
    float norm_cmp = sqrt((float) sum_of_squares_cmp);
    float min_sum = 0.0f;
    float max_sum = 0.0f;
    for (int idx = 0; idx < encoding_rep.size(); ++idx) {
        float sim_rep = (float) encoding_rep[idx] / norm_rep;
        float sim_cmp = (float) encoding_cmp[idx] / norm_cmp;
        min_sum += fminf(sim_rep, sim_cmp);
        max_sum += fmaxf(sim_rep, sim_cmp);
    }
    return min_sum / max_sum;
}

void clustering(const std::vector<std::vector<UIN>> &encodings,
                const std::vector<UIN> &rows, const UIN startIndexOfNonZeroRow, std::vector<int> &clusterIds) {

//    UIN num = 0;
    for (int idxRow = startIndexOfNonZeroRow; idxRow < encodings.size() - 1; ++idxRow) {
        if (idxRow > startIndexOfNonZeroRow && clusterIds[idxRow] != -1) {
            continue;
        }
        clusterIds[idxRow] = idxRow;
//#pragma omp parallel for dynamic
        for (int cmpIdx = idxRow + 1; cmpIdx < encodings.size(); ++cmpIdx) {
            if (clusterIds[cmpIdx] != -1) {
                continue;
            }
            const float similarity =
                clusterComparison(encodings[rows[startIndexOfNonZeroRow]], encodings[rows[cmpIdx]]);
            if (similarity > row_similarity_threshold_alpha) {
                clusterIds[rows[cmpIdx]] = clusterIds[rows[idxRow]];
//                ++num;
            }
        }
    }
//    printf("!!! num = %d\n", num);
}

void row_reordering(const sparseDataType::CSR<float> &matrix, struct ReorderedMatrix &reorderedMatrix) {
    std::vector<std::vector<UIN>> encodings;
    encoding(matrix, encodings);

    std::vector<UIN> dispersions(matrix.row_);
    calculateDispersion(matrix.col_, encodings, dispersions);

    std::vector<UIN> ascendingRow(matrix.row_); // Store the original row id
    std::iota(ascendingRow.begin(), ascendingRow.end(), 0); // ascending = {0, 1, 2, 3, ... rows-1}
    std::stable_sort(ascendingRow.begin(),
                     ascendingRow.end(),
                     [&dispersions](size_t i, size_t j) { return dispersions[i] < dispersions[j]; });

    std::vector<int> clusterIds(matrix.row_, -1);
    UIN startIndexOfNonZeroRow = 0;
    while (startIndexOfNonZeroRow < matrix.row_ && dispersions[ascendingRow[startIndexOfNonZeroRow]] == 0) {
        clusterIds[ascendingRow[startIndexOfNonZeroRow]] = 0;
        ++startIndexOfNonZeroRow;
    }

//    printf("!!! 1103 numCols = %d\n", matrix.rowOffsets_[1104] - matrix.rowOffsets_[1103]);
//    printf("!!! 1178 numCols = %d\n", matrix.rowOffsets_[1179] - matrix.rowOffsets_[1178]);
//    for (int i = 0; i < encodings[1103].size(); ++i) {
//        if (encodings[1103][i] != 0) {
//            printf("!!! 1103 encodings[1103][%d] = %d\n", i, encodings[1103][i]);
//        }
//    }
//    for (int i = 0; i < encodings[1178].size(); ++i) {
//        if (encodings[1178][i] != 0) {
//            printf("!!! 1103 encodings[1103][%d] = %d\n", i, encodings[1178][i]);
//        }
//    }

    clustering(encodings, ascendingRow, startIndexOfNonZeroRow, clusterIds);

    reorderedMatrix.reorderedRowIndices_.resize(matrix.row_);
    std::iota(reorderedMatrix.reorderedRowIndices_.begin(),
              reorderedMatrix.reorderedRowIndices_.end(),
              0); // rowIndices = {0, 1, 2, 3, ... rows-1}
    std::stable_sort(reorderedMatrix.reorderedRowIndices_.begin(),
                     reorderedMatrix.reorderedRowIndices_.end(),
                     [&clusterIds](int i, int j) { return clusterIds[i] < clusterIds[j]; });

    // Remove zero rows
    {
        startIndexOfNonZeroRow = 0;
        while (startIndexOfNonZeroRow < matrix.row_
            && matrix.rowOffsets_[reorderedMatrix.reorderedRowIndices_[startIndexOfNonZeroRow] + 1]
                - matrix.rowOffsets_[reorderedMatrix.reorderedRowIndices_[startIndexOfNonZeroRow]] == 0) {
            ++startIndexOfNonZeroRow;
        }
        reorderedMatrix.reorderedRowIndices_.erase(reorderedMatrix.reorderedRowIndices_.begin(),
                                                   reorderedMatrix.reorderedRowIndices_.begin()
                                                       + startIndexOfNonZeroRow);
    }
}