#include "CudaTimeCalculator.cuh"
#include "BSMR.hpp"
#include "checkData.hpp"
#include "host.hpp"
#include "sddmm.hpp"
#include "sddmmKernel.cuh"

// Reordering method
void sddmm(const Options &options,
           const Matrix<float> &matrixA,
           const Matrix<float> &matrixB,
           sparseMatrix::CSR<float> &matrixP,
           Logger &logger) {
    // Reordering
    BSMR bsmr(matrixP, options.similarityThresholdAlpha(), options.columnNonZeroThresholdBeta());
    logger.zcx_preprocessing_time_ = bsmr.reorderingTime();
    logger.numRowPanels_ = bsmr.numRowPanels();

    // Device data
    RPHM rphm(matrixP, bsmr);

    for (int ITER = 0; ITER < logger.numITER_; ++ITER) {
        float sddmm_time = 0.0f;

        // sddmm comp by gpu
        sddmm_gpu(matrixA, matrixB, rphm, matrixP, sddmm_time);

        logger.zcx_sddmm_time_ += sddmm_time;
    }

    evaluationReordering(matrixP, bsmr, logger);

    // Error check
    // check_rphm(matrixP, bsmr, rphm, options.columnNonZeroThresholdBeta());
    // checkSddmm(matrixA, matrixB, matrixS, matrixP);
}

bool checkSddmm(const Matrix<float> &matrixA,
                const Matrix<float> &matrixB,
                const sparseMatrix::CSR<float> &matrixS,
                const sparseMatrix::CSR<float> &matrixP) {
    // sddmm comp by cpu
    sparseMatrix::CSR<MATRIX_C_TYPE> matrixP_cpu_res(matrixS);
    sddmm_cpu(matrixA, matrixB, matrixS, matrixP_cpu_res);

    // Error check
    printf("check cpu sddmm and BSMR sddmm: \n");
    size_t numError = 0;
    if (!checkData(matrixP_cpu_res.values(), matrixP.values(), numError)) {
        printf("[checkData : NO PASS Error rate : %2.2f%%]\n",
               static_cast<float>(numError) / static_cast<float>(matrixP.values().size()) * 100);
        return false;
    }

    return true;
}

void sddmmBatch(int seq_len,
                int emb_dim,
                int nnz,
                int numBatches,
                const float *dQuery,
                const float *dKey,
                const UIN *d_offsets,
                const UIN *d_columns,
                float *dAttn) {
    const int M = seq_len;
    const int K = emb_dim;

    std::vector<UIN> offsets(M + 1);
    std::vector<UIN> columns(nnz);
    hipMemcpy(offsets.data(), d_offsets, offsets.size() * sizeof(UIN), hipMemcpyDeviceToHost);
    hipMemcpy(columns.data(), d_columns, columns.size() * sizeof(UIN), hipMemcpyDeviceToHost);

    sparseMatrix::CSR<float> matrixP(M, M, nnz, offsets, columns);
    // Reordering
    BSMR bsmr(matrixP, 0.3, 4);

    // Device data
    RPHM rphm(matrixP, bsmr);
    float time = 0.0f;
    sddmm_gpu_batch(numBatches, M, M, K, nnz, dQuery, dKey, rphm, dAttn, time);
}

void sddmmBatch(int seq_len,
                int emb_dim,
                int nnz,
                int numBatches,
                const float *dQuery,
                const float *dKey,
                const int *d_offsets,
                const int *d_columns,
                float *dAttn) {
    dev::vector<UIN> converted_offsets(seq_len + 1);
    hipMemcpy(converted_offsets.data(), d_offsets, converted_offsets.size() * sizeof(int), hipMemcpyDeviceToDevice);
    dev::vector<UIN> converted_columns(nnz);
    hipMemcpy(converted_columns.data(), d_columns, converted_columns.size() * sizeof(int), hipMemcpyDeviceToDevice);

    sddmmBatch(seq_len,
               emb_dim,
               nnz,
               numBatches,
               dQuery,
               dKey,
               converted_offsets.data(),
               converted_columns.data(),
               dAttn);
}
