#include "CudaTimeCalculator.cuh"
#include "RPHM.hpp"
#include "checkData.hpp"
#include "host.hpp"
#include "sddmm.hpp"
#include "sddmmKernel.cuh"

// Reordering method
void sddmm(const Options &options,
           const Matrix<float> &matrixA,
           const Matrix<float> &matrixB,
           sparseMatrix::CSR<float> &matrixP,
           Logger &logger) {
    // Reordering
    RPHM rphm(matrixP, options.similarityThresholdAlpha(), options.columnNonZeroThresholdBeta());

    logger.zcx_other_time_ = rphm.time();

    for (int ITER = 0; ITER < logger.numITER_; ++ITER) {
        float sddmm_time = 0.0f;

        // sddmm comp by gpu
        sddmm_gpu(matrixA, matrixB, rphm, matrixP, sddmm_time);

        logger.zcx_sddmm_time_ += sddmm_time;
    }

    // Error check
    check_rphm(matrixP, rphm, logger);
    //    checkSddmm(matrixA, matrixB, matrixS, matrixP);
}

bool checkSddmm(const Matrix<float> &matrixA,
                const Matrix<float> &matrixB,
                const sparseMatrix::CSR<float> &matrixS,
                const sparseMatrix::CSR<float> &matrixP) {
    // sddmm comp by cpu
    sparseMatrix::CSR<MATRIX_C_TYPE> matrixP_cpu_res(matrixS);
    sddmm_cpu(matrixA, matrixB, matrixS, matrixP_cpu_res);

    // Error check
    printf("check cpu sddmm and BSMR sddmm: \n");
    size_t numError = 0;
    if (!checkData(matrixP_cpu_res.values(), matrixP.values(), numError)) {
        printf("[checkData : NO PASS Error rate : %2.2f%%]\n",
               static_cast<float>(numError) / static_cast<float>(matrixP.values().size()) * 100);
        return false;
    }

    return true;
}

void sddmmBatch(int seq_len,
                int emb_dim,
                int nnz,
                int numBatches,
                const float *dQuery,
                const float *dKey,
                const UIN *d_offsets,
                const UIN *d_columns,
                float *dAttn) {
    const int M = seq_len;
    const int K = emb_dim;

    std::vector<UIN> offsets(M + 1);
    std::vector<UIN> columns(nnz);
    hipMemcpy(offsets.data(), d_offsets, offsets.size() * sizeof(UIN), hipMemcpyDeviceToHost);
    hipMemcpy(columns.data(), d_columns, columns.size() * sizeof(UIN), hipMemcpyDeviceToHost);

    sparseMatrix::CSR<float> matrixP(M, M, nnz, offsets, columns);
    RPHM rphm(matrixP);

    float time = 0.0f;
    sddmm_gpu_batch(numBatches, M, M, K, nnz, dQuery, dKey, rphm, dAttn, time);
}

void sddmmBatch(int seq_len,
                int emb_dim,
                int nnz,
                int numBatches,
                const float *dQuery,
                const float *dKey,
                const int *d_offsets,
                const int *d_columns,
                float *dAttn) {
    dev::vector<UIN> converted_offsets(seq_len + 1);
    hipMemcpy(converted_offsets.data(), d_offsets, converted_offsets.size() * sizeof(int), hipMemcpyDeviceToDevice);
    dev::vector<UIN> converted_columns(nnz);
    hipMemcpy(converted_columns.data(), d_columns, converted_columns.size() * sizeof(int), hipMemcpyDeviceToDevice);

    sddmmBatch(seq_len,
               emb_dim,
               nnz,
               numBatches,
               dQuery,
               dKey,
               converted_offsets.data(),
               converted_columns.data(),
               dAttn);
}
