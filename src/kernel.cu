#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "kernel.cuh"
#include "TensorCoreConfig.cuh"
#include "reordering.hpp"

namespace kernel {

using namespace nvcuda;

__global__ void convertFp32ToFp16(const UIN n, const float *in, half *out) {
    UIN idx = static_cast<UIN> (blockDim.x * blockIdx.x + threadIdx.x);
    if (idx < n) {
        out[idx] = in[idx];
    }
}

template<typename T>
__global__ void convertDataType(const UIN n, const float *in, T *out) {
    const UIN idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<T>(in[idx]);
//        printf("in[%d] = %f, static_cast<float>out[%d] = %f\n", idx, in[idx], idx, static_cast<float>(out[idx]));
    }
}

template __global__ void convertDataType<int>(const UIN n, const float *in, int *out);
template __global__ void convertDataType<float>(const UIN n, const float *in, float *out);
template __global__ void convertDataType<double>(const UIN n, const float *in, double *out);
template __global__ void convertDataType<half>(const UIN n, const float *in, half *out);

const float SPARSITY_BOUND = 1.0f;

template<typename T>
__device__ float calculateMatrixTileSparsity(const int tileM,
                                             const int tileN,
                                             const UIN ld,
                                             const MatrixStorageOrder storageOrder,
                                             const T *matrixPtr) {
    UIN nnzCount = 0;
#pragma unroll
    for (UIN rowIter = 0; rowIter < tileM; ++rowIter) {
#pragma unroll
        for (UIN colIter = 0; colIter < tileN; ++colIter) {
            if (storageOrder == MatrixStorageOrder::row_major) {
                nnzCount += *(matrixPtr + rowIter * ld + colIter) == 0 ? 0 : 1;
            } else {
                nnzCount += *(matrixPtr + colIter * ld + rowIter) == 0 ? 0 : 1;
            }
        }
    }
    const int numValues = tileM * tileN;
    return static_cast<float>(numValues - nnzCount) / static_cast<float>(numValues);
}

__device__ void matrixTileMultiplicationUseCudaCode(int pRowId, int pColId,
                                                    const UIN M, const UIN N, const UIN K,
                                                    const half *matrixA,
                                                    const half *matrixB,
                                                    const float *matrixS,
                                                    float *matrixP) {

}

__device__ void matrixTileMultiplicationUseTensorCore(int pRowId, int pColId,
                                                      const UIN M, const UIN N, const UIN K,
                                                      const half *matrixA,
                                                      const half *matrixB,
                                                      const float *matrixS,
                                                      float *matrixP) {

    // Leading dimensions. Packed with no transpositions.
    const int lda = K;
    const int ldb = N;
    const int ldp = N;
    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;
    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

//#pragma unroll
//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//        const int sIdx = pRowId * ldc + pColId + idx;
//
//        cFrag.x[idx] *= matrixS[sIdx];
//    }

    wmma::store_matrix_sync(pOffsetPtr, cFrag, ldp, wmma::mem_row_major);
}

__device__ void matrixTileMultiplicationUseTensorCore_coo(TensorCoreConfig tensorCoreConfig,
                                                          const UIN pRowId,
                                                          const UIN pColId,
                                                          const UIN M,
                                                          const UIN N,
                                                          const UIN K,
                                                          const UIN nnz,
                                                          const half *matrixA,
                                                          const half *matrixB,
                                                          const UIN *matrixSRowIndex,
                                                          const UIN *matrixSColIndex,
                                                          const UIN *matrixTileIndex,
                                                          const float *matrixS,
                                                          float *matrixP) {

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int warpId = tensorCoreConfig.globalWarpId();
    const int laneId = tensorCoreConfig.laneId();

    for (int matrixPIdx = matrixTileIndex[warpId];
         matrixPIdx < matrixTileIndex[warpId + 1]; ++matrixPIdx) {
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

//        if (matrixPIdx == 8410 && warpId == 780 && laneId_ == 0) {
//            printf(" warpId = %d\n"
//                   " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, curValue = %f"
//                   " laneId_ = %d findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                   warpId,
//                   static_cast<int>(pRowId),
//                   static_cast<int>(pColId),
//                   static_cast<int>(curRow),
//                   static_cast<int>(curCol),
//                   static_cast<float>(matrixS[matrixPIdx]),
//                   laneId_,
//                   findLaneId,
//                   findIdx,
//                   findIdx,
//                   static_cast<float>(cFrag.x[findIdx]));
//            printf("frag : ");
//            for (int idx = 0; idx < 8; ++idx) {
//                printf("%f ", static_cast<float>(cFrag.x[idx]));
//            }
//            printf("\n");
//        }
        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
//            printf(
//                " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                static_cast<int>(pRowId),
//                static_cast<int>(pColId),
//                static_cast<int>(curRow),
//                static_cast<int>(curCol),
//                findLaneId,
//                findIdx,
//                findIdx,
//                static_cast<float>(cFrag.x[findIdx]));
        }
    }
}

__global__ void sddmm_gpu(const UIN M, const UIN N, const UIN K,
                          const half *matrixA, const half *matrixB,
                          const float *matrixS,
                          float *matrixP) {
    const UIN tidX = (blockDim.x * blockIdx.x + threadIdx.x);
    const UIN tidY = (blockDim.y * blockIdx.y + threadIdx.y);

    const UIN warpM = tidX / WARP_SIZE;
    const UIN warpN = tidY;

//    const int landIdM = tidX % WARP_SIZE;
//    const int landIdN = tidY % WARP_SIZE;

    // Compute dense matrix multiplication using Tensor core

    const UIN pRowId = warpM * WMMA_M;
    const UIN pColId = warpN * WMMA_N;
//    const UIN pRowId = warpN * WMMA_N;
//    const UIN pColId = warpM * WMMA_M;

    if (pRowId >= M || pColId >= N) {
        return;
    }
    matrixTileMultiplicationUseTensorCore(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    const int ldp = N;
//    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
//    const float sparsity = calculateMatrixTileSparsity(WMMA_M, WMMA_N, ldp, MatrixStorageOrder::row_major, pOffsetPtr);
//    if (sparsity < 0) {
//        matrixTileMultiplicationUseCudaCode(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    } else {
//        matrixTileMultiplicationUseTensorCore(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    }

}

__global__ void sddmm_gpu_coo_1(TensorCoreConfig tensorCoreConfig,
                                const UIN M, const UIN N, const UIN K, const UIN nnz,
                                const half *matrixA, const half *matrixB,
                                const UIN *matrixSRowIndex,
                                const UIN *matrixSColIndex,
                                const UIN *matrixTileIndex,
                                const float *matrixS,
                                float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int warpId = tensorCoreConfig.globalWarpId();
    const int numData = matrixTileIndex[warpId + 1] - matrixTileIndex[warpId];
    if (numData <= 0) {
        return;
    }

    // Compute dense matrix multiplication using Tensor core
    matrixTileMultiplicationUseTensorCore_coo(tensorCoreConfig,
                                              pRowId,
                                              pColId,
                                              M,
                                              N,
                                              K,
                                              nnz,
                                              matrixA,
                                              matrixB,
                                              matrixSRowIndex,
                                              matrixSColIndex,
                                              matrixTileIndex,
                                              matrixS,
                                              matrixP);
//    const int ldp = N;
//    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
//    const float sparsity = (WMMA_M * WMMA_N - numData) / WMMA_M * WMMA_N;
//    if (sparsity < 0) {
//        matrixTileMultiplicationUseCudaCode(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    } else {
//        matrixTileMultiplicationUseTensorCore_coo(tensorCoreConfig,
//                                                  pRowId,
//                                                  pColId,
//                                                  M,
//                                                  N,
//                                                  K,
//                                                  nnz,
//                                                  matrixA,
//                                                  matrixB,
//                                                  matrixSRowIndex,
//                                                  matrixSColIndex,
//                                                  matrixTileMappedToWarpIndex,
//                                                  matrixS,
//                                                  matrixP);
//    }

}

__global__ void sddmm_gpu_coo_2(TensorCoreConfig tensorCoreConfig,
                                const UIN M, const UIN N, const UIN K, const UIN nnz,
                                const half *matrixA, const half *matrixB,
                                const UIN *matrixSRowIndex,
                                const UIN *matrixSColIndex,
                                const float *matrixS,
                                const UIN *matrixSTileMappedToWarpIndex,
                                const UIN *matrixSTileMappedToWarpIndexData,
                                float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = matrixSTileMappedToWarpIndexData[tileIndexDataIdx];
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

//        if (matrixPIdx == 8820 && globalWarpId == 661 && laneId == 0) {
//            printf("matrixPIdx == %d "
//                   " globalWarpId = %d "
//                   " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, curValue = %f"
//                   " laneId = %d findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                   matrixPIdx,
//                   globalWarpId,
//                   static_cast<int>(pRowId),
//                   static_cast<int>(pColId),
//                   static_cast<int>(curRow),
//                   static_cast<int>(curCol),
//                   static_cast<float>(matrixS[matrixPIdx]),
//                   laneId,
//                   fragmentInformation.laneId_,
//                   fragmentInformation.index_,
//                   fragmentInformation.index_,
//                   static_cast<float>(cFrag.x[fragmentInformation.laneId_]));
//        }
        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
//            printf(
//                " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                static_cast<int>(pRowId),
//                static_cast<int>(pColId),
//                static_cast<int>(curRow),
//                static_cast<int>(curCol),
//                findLaneId,
//                findIdx,
//                findIdx,
//                static_cast<float>(cFrag.x[findIdx]));
        }
    }

}

__global__ void bank_conflicts_test(UIN N, UIN K, const int *matrixB, const int *matrixA) {

    int pRowId = 0;
    int pColId = 0;

    int localWarpId = threadIdx.x / WARP_SIZE;
    int laneId = threadIdx.x % WARP_SIZE;

    __shared__ int aTile[MATRIX_TILE_A_SIZE_PER_BLOCK];

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K * NUM_OF_Y_PER_BLOCK) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
        const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

        for (int iter = 0; iter < 8; ++iter) {
            int beginIdxOfSharedMemory = localWarpId * NUMBER_OF_MEMORY_ACCESSES_MATRIX_TILE_A_PER_WARP;
            aTile[beginIdxOfSharedMemory + laneId + iter * WARP_SIZE] =
                matrixA[beginIdxOfSharedMemory + laneId + iter * WARP_SIZE];
        }
    }

}

//__device__ void loadMatrixAToSharedMemorySync(UIN laconst half *matrixA, half *sharedMemory) {
//
//#pragma unroll
//    for (int iter = 0; iter < 8; ++iter) {
//        const UIN startIdxOfSharedMemoryOfMtxA = localWarpId * NUMBER_OF_MATRIX_A_TILE_MEMORY_ACCESSES_PER_WARP;
//        const UIN startIdxOfGlobalMemoryOfMtxA = pRowIdForBlock * lda + K;
//        const UIN iterationSpan = laneId + iter * WARP_SIZE;
//        sharedMemory[startIdxOfSharedMemoryOfMtxA + iterationSpan] =
//            matrixA[startIdxOfGlobalMemoryOfMtxA + iterationSpan];
//    }
//    __syncthreads();
//}

// 在核函数中加入共享内存: 整块64×64的矩阵块A和块B按连续的顺序载入共享内存.
// 未完全实现.
// 问题: 共享内存中一次储存64×64个矩阵数据, 但是超过这个大小的矩阵无法载入, 会出现错误
// 放弃原因: 在 `wmma::load_matrix_sync` 中, 出现了bank conflict, 无法解决
__global__ void sddmm_gpu_coo_4_matrixA_row_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN globalWarpId = tensorCoreConfig.globalWarpId();

    __shared__ half aTileSharedMem[MATRIX_TILE_A_SIZE_PER_BLOCK];
    __shared__ half bTileSharedMem[MATRIX_TILE_B_SIZE_PER_BLOCK];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    const UIN pRowIdForBlock = tensorCoreConfig.blockStarRow();
    const UIN pColIdForBlock = tensorCoreConfig.blockStarCol();

    const UIN localWarpId = tensorCoreConfig.localWarpId();
    const UIN laneId = tensorCoreConfig.laneId();

    const UIN localWarpX = tensorCoreConfig.localWarpX();
    const UIN localWarpY = tensorCoreConfig.localWarpY();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];

    const int numDataInThisWarp = tileIndexEnd - tileIndexBegin;

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K * NUM_OF_WARP_X_PER_BLOCK) {

        // load matrix tile A to shared memory
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {

//            const UIN localARowIdForThisIteration = ;
//            const UIN localAColIdForThisIteration = ;
//            const UIN localBRowIdForThisIteration = ;
//            const UIN localBColIdForThisIteration = ;
//
//            const UIN indexForThisIterationA = laneId + iter * WARP_SIZE;
//            aTileSharedMem[startIdxOfSharedMemoryOfMtxA + indexForThisIterationA] =
//                matrixA[startIdxOfGlobalMemoryOfMtxA + indexForThisIterationA];
//
//            const UIN indexForThisIterationB = laneId + iter * WARP_SIZE;
//            bTileSharedMem[startIdxOfSharedMemoryOfMtxB + indexForThisIterationB] =
//                matrixB[startIdxOfGlobalMemoryOfMtxB + indexForThisIterationB];

//            if (globalWarpId == 0 && localWarpId == 5 && laneId == 0 && kIter == 0) {
//                printf("startIdxOfSharedMemoryOfMtxA + indexForThisIterationA = %d",
//                       startIdxOfSharedMemoryOfMtxA + indexForThisIterationA);
//            }

//            if (static_cast<int>( matrixA[startIdxOfGlobalMemoryOfMtxA + indexForThisIterationA]) == 1024
//                && blockIdx.y == 0 && globalWarpId == 29) {
//                printf(
//                    "globalWarpId = %d, localWarpId = %d, laneId = %d, kIter = %d, startIdxOfSharedMemoryOfMtxA = %d, startIdxOfGlobalMemoryOfMtxA = %d, indexForThisIterationA = %d, matrixA = %d\n",
//                    globalWarpId,
//                    localWarpId,
//                    laneId,
//                    kIter,
//                    startIdxOfSharedMemoryOfMtxA,
//                    startIdxOfGlobalMemoryOfMtxA,
//                    indexForThisIterationA,
//                    static_cast<int>(matrixA[startIdxOfGlobalMemoryOfMtxA + indexForThisIterationA]));
//            }
        }
        __syncthreads();

//        if (blockIdx.x == 0 && blockIdx.y == 0 && localWarpId == 0 && laneId == 0 && kIter == 0) {
//            for (int i = 0; i < MATRIX_TILE_A_SIZE_PER_BLOCK; ++i) {
//                printf("[%d] = %d\n", i, static_cast<int>(aTileSharedMem[i]));
//            }
//        }

        if (numDataInThisWarp > 0) {
            for (int sharedMemIter = 0; sharedMemIter < NUMBER_OF_MATRIX_TILE_K_IN_SHARED_MEMORY; ++sharedMemIter) {
                const UIN localKIterInSharedMem = sharedMemIter * WMMA_K;
                const auto aOffsetPtr = aTileSharedMem
                    + (localWarpY * WMMA_M * MATRIX_TILE_A_LEADING_DIMENSION)
                    + localKIterInSharedMem;
                const auto bOffsetPtr = bTileSharedMem
                    + (localKIterInSharedMem * MATRIX_TILE_B_LEADING_DIMENSION)
                    + localWarpX * WMMA_N;

                wmma::load_matrix_sync(aFrag, aOffsetPtr, MATRIX_TILE_A_LEADING_DIMENSION);
                wmma::load_matrix_sync(bFrag, bOffsetPtr, MATRIX_TILE_B_LEADING_DIMENSION);

                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }
        __syncthreads();
    }

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

// 在核函数中加入共享内存: 整块64×64的矩阵块A和块B按照16×16的块的顺序载入共享内存
__global__ void sddmm_gpu_coo_5_matrixA_row_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN globalWarpId = tensorCoreConfig.globalWarpId();

    __shared__ half aTileSMEM[MATRIX_TILE_A_SIZE_PER_BLOCK];
    __shared__ half bTileSMEM[MATRIX_TILE_B_SIZE_PER_BLOCK];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    const UIN localWarpId = tensorCoreConfig.localWarpId();
    const UIN laneId = tensorCoreConfig.laneId();

    const UIN localWarpX = tensorCoreConfig.localWarpX();
    const UIN localWarpY = tensorCoreConfig.localWarpY();

    const UIN startIndexOfMatrixS = matrixSTileMappedToWarpIndex[globalWarpId];
    const UIN endIndexOfMatrixS = matrixSTileMappedToWarpIndex[globalWarpId + 1];

    const UIN numDataInThisWarp = endIndexOfMatrixS - startIndexOfMatrixS;

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    const UIN startIndexOfSharedMemoryOfMatrixA = localWarpId * NUMBER_OF_MEMORY_ACCESSES_MATRIX_TILE_A_PER_WARP;
    const UIN startIndexOfSharedMemoryOfMatrixB = localWarpId * NUMBER_OF_MEMORY_ACCESSES_MATRIX_TILE_B_PER_WARP;

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += ITERATION_STEP_OF_K) {
        // Load matrix tile A to shared memory
#pragma unroll
        for (int iter = 0; iter < 8; ++iter) {

            const UIN localRowIdInThisIteration = 2 * iter + laneId / WMMA_K;
            const UIN localColIdInThisIteration = laneId % WMMA_K;

            const UIN aRowId = pRowId + localRowIdInThisIteration;
            const UIN aColId = kIter + localWarpX * WMMA_K + localColIdInThisIteration;

            const UIN bRowId = kIter + localWarpY * WMMA_K + localRowIdInThisIteration;
            const UIN bColId = pColId + localColIdInThisIteration;

            const UIN indexOfSharedMemoryInThisIteration = iter * WARP_SIZE + laneId;

            aTileSMEM[startIndexOfSharedMemoryOfMatrixA + indexOfSharedMemoryInThisIteration] =
                (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<half>(0);

            bTileSMEM[startIndexOfSharedMemoryOfMatrixB + indexOfSharedMemoryInThisIteration] =
                (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<half>(0);
        }
        __syncthreads();

        // Only warps where data exists need to be computed
        if (numDataInThisWarp > 0) {
            for (int sharedMemIter = 0; sharedMemIter < NUMBER_OF_MATRIX_TILE_K_IN_SHARED_MEMORY; ++sharedMemIter) {
                const auto aOffsetPtr = aTileSMEM
                    + (localWarpY * NUM_OF_WARP_X_PER_BLOCK + sharedMemIter) * MATRIX_TILE_A_SIZE;
                const auto bOffsetPtr = bTileSMEM
                    + (sharedMemIter * NUM_OF_WARP_X_PER_BLOCK + localWarpX) * MATRIX_TILE_B_SIZE;

                wmma::load_matrix_sync(aFrag, aOffsetPtr, WMMA_K);
                wmma::load_matrix_sync(bFrag, bOffsetPtr, WMMA_M);

                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }
        }
        __syncthreads();
    }

    for (UIN matrixPIdx = startIndexOfMatrixS; matrixPIdx < endIndexOfMatrixS; ++matrixPIdx) {
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

__global__ void sddmm_gpu_coo_3_matrixA_row_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN globalWarpId = tensorCoreConfig.globalWarpId();

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];

    __shared__ half aTile[WMMA_M * NUM_OF_Y_PER_BLOCK * WMMA_K * NUM_OF_WARP_X_PER_BLOCK];
    __shared__ half bTile[WMMA_K * NUM_OF_Y_PER_BLOCK * WMMA_N * NUM_OF_WARP_X_PER_BLOCK];

    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

__global__ void sddmm_gpu_coo_3_matrixA_row_matrixB_col(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = K;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId + bColId * ldb;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}


// TODO：Finish the following kernels. Error occurs when compiling the code.
__global__ void sddmm_gpu_coo_3_matrixA_col_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = M;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::col_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId + aColId * lda;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

// TODO: Finish the following kernels. Error occurs when compiling the code.
__global__ void sddmm_gpu_coo_3_matrixA_col_matrixB_col(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.warpStarRow();
    const UIN pColId = tensorCoreConfig.warpStarCol();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = M;
    const UIN ldb = K;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::col_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId + aColId * lda;
            const auto bOffsetPtr = matrixB + bRowId + bColId * ldb;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

// blockDim: [64, 1, 1]
__global__ void sddmm_gpu_csr_matrix_row_matrix_row(const UIN M,
                                                    const UIN N,
                                                    const UIN K,
                                                    const half *matrixA,
                                                    const half *matrixB,
                                                    const UIN numNonZeroRow,
                                                    const UIN *reorderedMatrixRowIndices,
                                                    const UIN *reorderedMatrixColIndicesOffset,
                                                    const UIN *reorderedMatrixColIndicesInEachRowPanel,
                                                    const UIN *reorderedMatrixPanelOffsets,
                                                    float *matrixP) {
    __shared__ half aTileSMEM[256];
    __shared__ half bTileSMEM[256];

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    const UIN laneId = threadIdx.x % WARP_SIZE;
    const UIN warpId = threadIdx.x % WARP_SIZE;

    const UIN rowPanelId = blockIdx.x;

    const UIN lda = K;
    const UIN ldb = N;

    const UIN numPanels = reorderedMatrixPanelOffsets[rowPanelId + 1] - reorderedMatrixPanelOffsets[rowPanelId];
    for (int colTileIdx = 0; colTileIdx < numPanels; colTileIdx += 2) {
        const UIN startIndexOfColTile =
            reorderedMatrixColIndicesOffset[rowPanelId] + col_tile_size * colTileIdx;
        const UIN endIndexOfColTile = reorderedMatrixColIndicesOffset[rowPanelId + 1];

        // Loop over K
        for (int kIter = 0; kIter < K; kIter += WMMA_K) {
            // Load matrix A into shared memory, each thread loads 4 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 4; ++iter) {
                const UIN idxOfReorderedMatrixRowIndices =
                    (rowPanelId * row_panel_size) + (warpId * 8) + (laneId / 16) + (iter * 2);
                const UIN aRowId = idxOfReorderedMatrixRowIndices < numNonZeroRow ?
                    reorderedMatrixRowIndices[idxOfReorderedMatrixRowIndices] : M;
                const UIN aColId = kIter + laneId;

                aTileSMEM[warpId * 128 + iter * 32 + laneId] =
                    (aRowId < M && aColId < K) ? matrixA[aRowId * lda + aColId] : static_cast<half>(0);
            }

            // Load matrix B data into shared memory, each thread loads 8 elements, conflict-free access
#pragma unroll
            for (int iter = 0; iter < 8; ++iter) {
                const UIN bRowId = warpId * 8 + iter;
                const UIN idxOfReorderedMatrixColIndicesInEachRowPanel = startIndexOfColTile + laneId;
                const UIN bColId = idxOfReorderedMatrixColIndicesInEachRowPanel < endIndexOfColTile ?
                    reorderedMatrixColIndicesInEachRowPanel[idxOfReorderedMatrixColIndicesInEachRowPanel] : N;

                bTileSMEM[warpId * 256 + laneId * 32] =
                    (bRowId < K && bColId < N) ? matrixB[bRowId * ldb + bColId] : static_cast<half>(0);
            }

            __syncthreads();

            // Compute the matrix multiplication
            {
                wmma::load_matrix_sync(aFrag, aTileSMEM, WMMA_N);
                wmma::load_matrix_sync(bFrag, bTileSMEM + warpId * WMMA_N, WMMA_N * 2);

                wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
            }

            __syncthreads();
        }

        // Store the result

    }
}

} // namespace kernel

void calculateKernelSettings(const UIN size, UIN &numBlocks, UIN &numThreads) {
    const UIN maxThreadsPerBlock = 1024;
    numThreads = size < maxThreadsPerBlock ? size : maxThreadsPerBlock;
    numBlocks = (size + numThreads - 1) / numThreads;
}

void sddmm_gpu_coo_3(TensorCoreConfig tensorCoreConfig,
                     const UIN M, const UIN N, const UIN K,
                     const half *matrixA, const MatrixStorageOrder matrixAStorageOrder,
                     const half *matrixB, const MatrixStorageOrder matrixBStorageOrder,
                     const UIN *matrixSRowIndex,
                     const UIN *matrixSColIndex,
                     const float *matrixS,
                     const UIN *matrixSTileMappedToWarpIndex,
                     float *matrixP) {
    if (matrixAStorageOrder == MatrixStorageOrder::row_major && matrixBStorageOrder == MatrixStorageOrder::row_major) {
        kernel::sddmm_gpu_coo_5_matrixA_row_matrixB_row<<<tensorCoreConfig.gridDim(), tensorCoreConfig.blockDim()>>>(tensorCoreConfig,
            M,
            N,
            K,
            matrixA,
            matrixB,
            matrixSRowIndex,
            matrixSColIndex,
            matrixS,
            matrixSTileMappedToWarpIndex,
            matrixP);
    } else if (matrixAStorageOrder == MatrixStorageOrder::row_major
        && matrixBStorageOrder == MatrixStorageOrder::col_major) {
        kernel::sddmm_gpu_coo_3_matrixA_row_matrixB_col<<<tensorCoreConfig.gridDim(), tensorCoreConfig.blockDim()>>>(tensorCoreConfig,
            M,
            N,
            K,
            matrixA,
            matrixB,
            matrixSRowIndex,
            matrixSColIndex,
            matrixS,
            matrixSTileMappedToWarpIndex,
            matrixP);
    } else if (matrixAStorageOrder == MatrixStorageOrder::col_major
        && matrixBStorageOrder == MatrixStorageOrder::row_major) {
        kernel::sddmm_gpu_coo_3_matrixA_col_matrixB_row<<<tensorCoreConfig.gridDim(), tensorCoreConfig.blockDim()>>>(tensorCoreConfig,
            M,
            N,
            K,
            matrixB,
            matrixA,
            matrixSColIndex,
            matrixSRowIndex,
            matrixS,
            matrixSTileMappedToWarpIndex,
            matrixP);
    } else {
        kernel::sddmm_gpu_coo_3_matrixA_col_matrixB_col<<<tensorCoreConfig.gridDim(), tensorCoreConfig.blockDim()>>>(tensorCoreConfig,
            M,
            N,
            K,
            matrixB,
            matrixA,
            matrixSColIndex,
            matrixSRowIndex,
            matrixS,
            matrixSTileMappedToWarpIndex,
            matrixP);
    }

}