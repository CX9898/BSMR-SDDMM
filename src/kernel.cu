#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "kernel.cuh"
#include "wmmaSetting.hpp"

const int WARP_SIZE = 32;

using namespace nvcuda::wmma;

template<typename T>
__global__ void test(int n, T *a) {
    for (int i = 0; i < n; ++i) {
        printf("%f ", a[i]);
    }
}

template __global__ void test<float>(int n, float *a);
template __global__ void test<half>(int n, half *a);

__global__ void convertFp32ToFp16(const int n, const float *in, half *out) {
    int idx = (int) (blockDim.x * blockIdx.x + threadIdx.x);
    if (idx < n) {
        out[idx] = in[idx];
//        printf("in : %f, out : %f ", in[idx], out[idx]);
    }
}

__global__ void comp_sddmm_gpu(const int M, const int N, const int K,
                               const half *matrixA, const half *matrixB,
                               const float *matrixS,
                               float *matrixP) {
    const int warpM = (int) (blockDim.x * blockIdx.x + threadIdx.x) / WARP_SIZE;
    const int warpN = (int) (blockDim.x * blockIdx.x + threadIdx.x);

    // Compute dense matrix multiplication using Tensor core

    const int pRowId = warpM * WMMA_M;
    const int pColId = warpN * WMMA_N;

    if (pRowId >= M || pColId >= N) {
        return;
    }

    fragment<matrix_a, WMMA_M, WMMA_N, WMMA_K, half, row_major> aFrag;
    fragment<matrix_b, WMMA_M, WMMA_N, WMMA_K, half, row_major> bFrag;

    fragment<accumulator, WMMA_M, WMMA_N, WMMA_K, float> cFrag;
    fill_fragment(cFrag, 0.0f);

    // Leading dimensions. Packed with no transpositions.
    const int lda = K;
    const int ldb = N;
    const int ldp = N;

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            load_matrix_sync(aFrag, aOffsetPtr, lda);
            load_matrix_sync(bFrag, bOffsetPtr, ldb);

            mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//        const int sIdx = pRowId * ldc + pColId + idx;
//
//        cFrag.x[idx] *= matrixS[sIdx];
//    }
//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//
//        printf("%f ", aFrag.x[idx]);
//    }

    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
    store_matrix_sync(pOffsetPtr, cFrag, ldp, mem_row_major);
}
