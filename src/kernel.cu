#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "kernel.cuh"
#include "wmmaSetting.hpp"
#include "Matrix.hpp"

const float SPARSITY_BOUND = 0.5f;

template<typename T>
__device__ int sparsityComparator(const size_t WMMA_M,
                                  const size_t WMMA_N,
                                  const size_t ld,
                                  const MatrixStorageOrder storageOrder,
                                  const T *matrixPtr) {
    size_t nnzCount = 0;
#pragma unroll
    for (size_t rowIter = 0; rowIter < WMMA_M; ++rowIter) {
#pragma unroll
        for (size_t colIter = 0; colIter < WMMA_N; ++colIter) {
            if (storageOrder == MatrixStorageOrder::row_major) {
                nnzCount += *(matrixPtr + rowIter * ld + colIter) == 0 ? 0 : 1;
            } else {
                nnzCount += *(matrixPtr + colIter * ld + rowIter) == 0 ? 0 : 1;
            }
        }
    }
    const int numValues = WMMA_M * WMMA_N;
    const float sparsity = static_cast<float>(numValues - nnzCount) / numValues;
    return sparsity >= SPARSITY_BOUND ? 1 : 0;
}

template<typename T>
__global__ void printData(size_t n, T *a) {
    for (size_t i = 0; i < n; ++i) {
        printf("%f ", static_cast<float>(a[i]));
    }
}

template __global__ void printData<float>(size_t n, float *a);
template __global__ void printData<half>(size_t n, half *a);

__global__ void convertFp32ToFp16(const size_t n, const float *in, half *out) {
    size_t idx = static_cast<size_t> (blockDim.x * blockIdx.x + threadIdx.x);
    if (idx < n) {
        out[idx] = in[idx];
    }
}

__global__ void comp_sddmm_gpu(const size_t M, const size_t N, const size_t K,
                               const half *matrixA, const half *matrixB,
                               const float *matrixS,
                               float *matrixP) {
    const size_t tidX = (blockDim.x * blockIdx.x + threadIdx.x);
    const size_t tidY = (blockDim.y * blockIdx.y + threadIdx.y);

    const size_t warpM = (blockDim.x * blockIdx.x + threadIdx.x) / WARP_SIZE;
    const size_t warpN = (blockDim.y * blockIdx.y + threadIdx.y);

    // Compute dense matrix multiplication using Tensor core

    const size_t pRowId = warpM * WMMA_M;
    const size_t pColId = warpN * WMMA_N;

    if (pRowId >= M || pColId >= N) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const int lda = K;
    const int ldb = N;
    const int ldp = N;
    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;

//    if (sparsityComparator(WMMA_M, WMMA_N, ldp, MatrixStorageOrder::row_major, pOffsetPtr)) {
//
//    }

//    printf("pRowId : %d, pColId : %d\n", pRowId,pColId);
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, nvcuda::wmma::row_major>
        aFrag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, nvcuda::wmma::row_major>
        bFrag;

    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;
    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;
//        if (tidX == 0) {
//            printf(" cur kIter = %d\n", kIter);
//            printf(" cur aRowId = %d, aColId = %d, bRowId = %d, bColId = %d\n", aRowId, aColId, bRowId, bColId);
//        }
        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            nvcuda::wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            nvcuda::wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            nvcuda::wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

//#pragma unroll
//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//        const int sIdx = pRowId * ldc + pColId + idx;
//
//        cFrag.x[idx] *= matrixS[sIdx];
//    }
    if (tidX == 0) {
        printf("\n cFrag.num_elements : %d\n", cFrag.num_elements);
        for (int idx = 0; idx < cFrag.num_elements; ++idx) {
            printf(" %f ", static_cast<float>(cFrag.x[idx]));
        }

        nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> sFrag;
        const int lds = N;
        const auto ptrS = matrixS + pRowId * lds + pColId;
        nvcuda::wmma::load_matrix_sync(sFrag,ptrS,lds,nvcuda::wmma::mem_row_major);

        printf("\n sFrag.num_elements : %d\n", sFrag.num_elements);
        for (int idx = 0; idx < sFrag.num_elements; ++idx) {
            printf(" %f ", static_cast<float>(sFrag.x[idx]));
        }
    }

    nvcuda::wmma::store_matrix_sync(pOffsetPtr, cFrag, ldp, nvcuda::wmma::mem_row_major);
}
