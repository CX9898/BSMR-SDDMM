#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "kernel.cuh"
#include "TensorCoreConfig.cuh"

using namespace nvcuda;

template<typename T>
__global__ void printData(UIN n, T *a) {
    for (UIN i = 0; i < n; ++i) {
        printf("%f ", static_cast<float>(a[i]));
    }
}

template __global__ void printData<float>(UIN n, float *a);

template __global__ void printData<half>(UIN n, half *a);

__global__ void convertFp32ToFp16(const UIN n, const float *in, half *out) {
    UIN idx = static_cast<UIN> (blockDim.x * blockIdx.x + threadIdx.x);
    if (idx < n) {
        out[idx] = in[idx];
    }
}

template<typename T>
__global__ void convertDataType(const UIN n, const float *in, T *out) {
    const UIN idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        out[idx] = static_cast<T>(in[idx]);
    }
}

template __global__ void convertDataType<int>(const UIN n, const float *in, int *out);
template __global__ void convertDataType<float>(const UIN n, const float *in, float *out);
template __global__ void convertDataType<double>(const UIN n, const float *in, double *out);
template __global__ void convertDataType<half>(const UIN n, const float *in, half *out);

const float SPARSITY_BOUND = 1.0f;

template<typename T>
__device__ float calculateMatrixTileSparsity(const int tileM,
                                             const int tileN,
                                             const UIN ld,
                                             const MatrixStorageOrder storageOrder,
                                             const T *matrixPtr) {
    UIN nnzCount = 0;
#pragma unroll
    for (UIN rowIter = 0; rowIter < tileM; ++rowIter) {
#pragma unroll
        for (UIN colIter = 0; colIter < tileN; ++colIter) {
            if (storageOrder == MatrixStorageOrder::row_major) {
                nnzCount += *(matrixPtr + rowIter * ld + colIter) == 0 ? 0 : 1;
            } else {
                nnzCount += *(matrixPtr + colIter * ld + rowIter) == 0 ? 0 : 1;
            }
        }
    }
    const int numValues = tileM * tileN;
    return static_cast<float>(numValues - nnzCount) / static_cast<float>(numValues);
}

__device__ void matrixTileMultiplicationUseCudaCode(int pRowId, int pColId,
                                                    const UIN M, const UIN N, const UIN K,
                                                    const half *matrixA,
                                                    const half *matrixB,
                                                    const float *matrixS,
                                                    float *matrixP) {

}

__device__ void matrixTileMultiplicationUseTensorCore(int pRowId, int pColId,
                                                      const UIN M, const UIN N, const UIN K,
                                                      const half *matrixA,
                                                      const half *matrixB,
                                                      const float *matrixS,
                                                      float *matrixP) {

    // Leading dimensions. Packed with no transpositions.
    const int lda = K;
    const int ldb = N;
    const int ldp = N;
    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;
    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

//#pragma unroll
//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//        const int sIdx = pRowId * ldc + pColId + idx;
//
//        cFrag.x[idx] *= matrixS[sIdx];
//    }

    wmma::store_matrix_sync(pOffsetPtr, cFrag, ldp, wmma::mem_row_major);
}

__device__ void matrixTileMultiplicationUseTensorCore_coo(TensorCoreConfig tensorCoreConfig,
                                                          const UIN pRowId,
                                                          const UIN pColId,
                                                          const UIN M,
                                                          const UIN N,
                                                          const UIN K,
                                                          const UIN nnz,
                                                          const half *matrixA,
                                                          const half *matrixB,
                                                          const UIN *matrixSRowIndex,
                                                          const UIN *matrixSColIndex,
                                                          const UIN *matrixTileIndex,
                                                          const float *matrixS,
                                                          float *matrixP) {

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int warpId = tensorCoreConfig.globalWarpId();
    const int laneId = tensorCoreConfig.laneId();

    for (int matrixPIdx = matrixTileIndex[warpId];
         matrixPIdx < matrixTileIndex[warpId + 1]; ++matrixPIdx) {
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

//        if (matrixPIdx == 8410 && warpId == 780 && laneId_ == 0) {
//            printf(" warpId = %d\n"
//                   " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, curValue = %f"
//                   " laneId_ = %d findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                   warpId,
//                   static_cast<int>(pRowId),
//                   static_cast<int>(pColId),
//                   static_cast<int>(curRow),
//                   static_cast<int>(curCol),
//                   static_cast<float>(matrixS[matrixPIdx]),
//                   laneId_,
//                   findLaneId,
//                   findIdx,
//                   findIdx,
//                   static_cast<float>(cFrag.x[findIdx]));
//            printf("frag : ");
//            for (int idx = 0; idx < 8; ++idx) {
//                printf("%f ", static_cast<float>(cFrag.x[idx]));
//            }
//            printf("\n");
//        }
        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
//            printf(
//                " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                static_cast<int>(pRowId),
//                static_cast<int>(pColId),
//                static_cast<int>(curRow),
//                static_cast<int>(curCol),
//                findLaneId,
//                findIdx,
//                findIdx,
//                static_cast<float>(cFrag.x[findIdx]));
        }
    }
}

__global__ void sddmm_gpu(const UIN M, const UIN N, const UIN K,
                          const half *matrixA, const half *matrixB,
                          const float *matrixS,
                          float *matrixP) {
    const UIN tidX = (blockDim.x * blockIdx.x + threadIdx.x);
    const UIN tidY = (blockDim.y * blockIdx.y + threadIdx.y);

    const UIN warpM = tidX / WARP_SIZE;
    const UIN warpN = tidY;

//    const int landIdM = tidX % WARP_SIZE;
//    const int landIdN = tidY % WARP_SIZE;

    // Compute dense matrix multiplication using Tensor core

    const UIN pRowId = warpM * WMMA_M;
    const UIN pColId = warpN * WMMA_N;
//    const UIN pRowId = warpN * WMMA_N;
//    const UIN pColId = warpM * WMMA_M;

    if (pRowId >= M || pColId >= N) {
        return;
    }
    matrixTileMultiplicationUseTensorCore(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    const int ldp = N;
//    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
//    const float sparsity = calculateMatrixTileSparsity(WMMA_M, WMMA_N, ldp, MatrixStorageOrder::row_major, pOffsetPtr);
//    if (sparsity < 0) {
//        matrixTileMultiplicationUseCudaCode(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    } else {
//        matrixTileMultiplicationUseTensorCore(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    }

}

__global__ void sddmm_gpu_coo_1(TensorCoreConfig tensorCoreConfig,
                                const UIN M, const UIN N, const UIN K, const UIN nnz,
                                const half *matrixA, const half *matrixB,
                                const UIN *matrixSRowIndex,
                                const UIN *matrixSColIndex,
                                const UIN *matrixTileIndex,
                                const float *matrixS,
                                float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int warpId = tensorCoreConfig.globalWarpId();
    const int numData = matrixTileIndex[warpId + 1] - matrixTileIndex[warpId];
    if (numData <= 0) {
        return;
    }

    // Compute dense matrix multiplication using Tensor core
    matrixTileMultiplicationUseTensorCore_coo(tensorCoreConfig,
                                              pRowId,
                                              pColId,
                                              M,
                                              N,
                                              K,
                                              nnz,
                                              matrixA,
                                              matrixB,
                                              matrixSRowIndex,
                                              matrixSColIndex,
                                              matrixTileIndex,
                                              matrixS,
                                              matrixP);
//    const int ldp = N;
//    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
//    const float sparsity = (WMMA_M * WMMA_N - numData) / WMMA_M * WMMA_N;
//    if (sparsity < 0) {
//        matrixTileMultiplicationUseCudaCode(pRowId, pColId, M, N, K, matrixA, matrixB, matrixS, matrixP);
//    } else {
//        matrixTileMultiplicationUseTensorCore_coo(tensorCoreConfig,
//                                                  pRowId,
//                                                  pColId,
//                                                  M,
//                                                  N,
//                                                  K,
//                                                  nnz,
//                                                  matrixA,
//                                                  matrixB,
//                                                  matrixSRowIndex,
//                                                  matrixSColIndex,
//                                                  matrixTileMappedToWarpIndex,
//                                                  matrixS,
//                                                  matrixP);
//    }

}

__global__ void sddmm_gpu_coo_2(TensorCoreConfig tensorCoreConfig,
                                const UIN M, const UIN N, const UIN K, const UIN nnz,
                                const half *matrixA, const half *matrixB,
                                const UIN *matrixSRowIndex,
                                const UIN *matrixSColIndex,
                                const float *matrixS,
                                const UIN *matrixSTileMappedToWarpIndex,
                                const UIN *matrixSTileMappedToWarpIndexData,
                                float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = matrixSTileMappedToWarpIndexData[tileIndexDataIdx];
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

//        if (matrixPIdx == 8820 && globalWarpId == 661 && laneId == 0) {
//            printf("matrixPIdx == %d "
//                   " globalWarpId = %d "
//                   " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, curValue = %f"
//                   " laneId = %d findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                   matrixPIdx,
//                   globalWarpId,
//                   static_cast<int>(pRowId),
//                   static_cast<int>(pColId),
//                   static_cast<int>(curRow),
//                   static_cast<int>(curCol),
//                   static_cast<float>(matrixS[matrixPIdx]),
//                   laneId,
//                   fragmentInformation.laneId_,
//                   fragmentInformation.index_,
//                   fragmentInformation.index_,
//                   static_cast<float>(cFrag.x[fragmentInformation.laneId_]));
//        }
        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
//            printf(
//                " pRowId = %d, pColId = %d, curRow = %d, curCol = %d, findLaneId = %d, findIdx = %d, cFrag.x[%d] = %f\n",
//                static_cast<int>(pRowId),
//                static_cast<int>(pColId),
//                static_cast<int>(curRow),
//                static_cast<int>(curCol),
//                findLaneId,
//                findIdx,
//                findIdx,
//                static_cast<float>(cFrag.x[findIdx]));
        }
    }

}

__device__ void sddmm_gpu_coo_3_tensorCore(TensorCoreConfig tensorCoreConfig,
                                           const UIN M,
                                           const UIN N,
                                           const UIN K,
                                           const half *matrixA,
                                           const half *matrixB,
                                           const UIN *matrixSRowIndex,
                                           const UIN *matrixSColIndex,
                                           const float *matrixS,
                                           const UIN *matrixSTileMappedToWarpIndex,
                                           float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN globalWarpId = tensorCoreConfig.globalWarpId();

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];

    __shared__ half aTile[WMMA_M * NUM_OF_Y_PER_BLOCK * WMMA_K * NUM_OF_WARP_X_PER_BLOCK];
    __shared__ half bTile[WMMA_K * NUM_OF_Y_PER_BLOCK * WMMA_N * NUM_OF_WARP_X_PER_BLOCK];

    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

__device__ void sddmm_gpu_coo_3_matrixA_row_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN globalWarpId = tensorCoreConfig.globalWarpId();

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];

    __shared__ half aTile[WMMA_M * NUM_OF_Y_PER_BLOCK * WMMA_K * NUM_OF_WARP_X_PER_BLOCK];
    __shared__ half bTile[WMMA_K * NUM_OF_Y_PER_BLOCK * WMMA_N * NUM_OF_WARP_X_PER_BLOCK];

    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

__device__ void sddmm_gpu_coo_3_matrixA_row_matrixB_col(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = K;
    const UIN ldb = K;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::row_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId + bColId * ldb;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}


// TODO：Finish the following kernels. Error occurs when compiling the code.
__device__ void sddmm_gpu_coo_3_matrixA_col_matrixB_row(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = M;
    const UIN ldb = N;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::col_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::row_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId + aColId * lda;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

// TODO: Finish the following kernels. Error occurs when compiling the code.
__device__ void sddmm_gpu_coo_3_matrixA_col_matrixB_col(TensorCoreConfig tensorCoreConfig,
                                                        const UIN M,
                                                        const UIN N,
                                                        const UIN K,
                                                        const half *matrixA,
                                                        const half *matrixB,
                                                        const UIN *matrixSRowIndex,
                                                        const UIN *matrixSColIndex,
                                                        const float *matrixS,
                                                        const UIN *matrixSTileMappedToWarpIndex,
                                                        float *matrixP) {
    tensorCoreConfig.initByKernel(blockIdx, blockDim, threadIdx);

    const UIN pRowId = tensorCoreConfig.rowBeginOfTile();
    const UIN pColId = tensorCoreConfig.colBeginOfTile();

    if (pRowId >= M || pColId >= N) {
        return;
    }

    const int globalWarpId = tensorCoreConfig.globalWarpId();

    const int tileIndexBegin = matrixSTileMappedToWarpIndex[globalWarpId];
    const int tileIndexEnd = matrixSTileMappedToWarpIndex[globalWarpId + 1];
    const int numData = tileIndexEnd - tileIndexBegin;
    if (numData <= 0) {
        return;
    }

    // Leading dimensions. Packed with no transpositions.
    const UIN lda = M;
    const UIN ldb = K;

    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, MATRIX_A_TYPE, wmma::col_major> aFrag;
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, MATRIX_B_TYPE, wmma::col_major> bFrag;

    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, MATRIX_C_TYPE> cFrag;

    fill_fragment(cFrag, 0.0f);

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;

        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId + aColId * lda;
            const auto bOffsetPtr = matrixB + bRowId + bColId * ldb;

            wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

    const int laneId = tensorCoreConfig.laneId();

    for (int tileIndexDataIdx = tileIndexBegin; tileIndexDataIdx < tileIndexEnd; ++tileIndexDataIdx) {
        const UIN matrixPIdx = tileIndexDataIdx;
        const UIN curRow = matrixSRowIndex[matrixPIdx];
        const UIN curCol = matrixSColIndex[matrixPIdx];

        FragmentInformation fragmentInformation;
        tensorCoreConfig.positionCalculator(pRowId, pColId, curRow, curCol, fragmentInformation);

        if (laneId == fragmentInformation.laneId_) {
            matrixP[matrixPIdx] = cFrag.x[fragmentInformation.index_];
        }
    }
}

__global__ void sddmm_gpu_coo_3(TensorCoreConfig tensorCoreConfig,
                                const UIN M, const UIN N, const UIN K,
                                const half *matrixA, const MatrixStorageOrder matrixAStorageOrder,
                                const half *matrixB, const MatrixStorageOrder matrixBStorageOrder,
                                const UIN *matrixSRowIndex,
                                const UIN *matrixSColIndex,
                                const float *matrixS,
                                const UIN *matrixSTileMappedToWarpIndex,
                                float *matrixP) {
    if (matrixAStorageOrder == MatrixStorageOrder::row_major && matrixBStorageOrder == MatrixStorageOrder::row_major) {
        sddmm_gpu_coo_3_matrixA_row_matrixB_row(tensorCoreConfig,
                                                M,
                                                N,
                                                K,
                                                matrixA,
                                                matrixB,
                                                matrixSRowIndex,
                                                matrixSColIndex,
                                                matrixS,
                                                matrixSTileMappedToWarpIndex,
                                                matrixP);
    } else if (matrixAStorageOrder == MatrixStorageOrder::row_major
        && matrixBStorageOrder == MatrixStorageOrder::col_major) {
        sddmm_gpu_coo_3_matrixA_row_matrixB_col(tensorCoreConfig,
                                                M,
                                                N,
                                                K,
                                                matrixA,
                                                matrixB,
                                                matrixSRowIndex,
                                                matrixSColIndex,
                                                matrixS,
                                                matrixSTileMappedToWarpIndex,
                                                matrixP);
    } else if (matrixAStorageOrder == MatrixStorageOrder::col_major
        && matrixBStorageOrder == MatrixStorageOrder::row_major) {
        sddmm_gpu_coo_3_matrixA_col_matrixB_row(tensorCoreConfig,
                                                M,
                                                N,
                                                K,
                                                matrixB,
                                                matrixA,
                                                matrixSColIndex,
                                                matrixSRowIndex,
                                                matrixS,
                                                matrixSTileMappedToWarpIndex,
                                                matrixP);
    } else {
        sddmm_gpu_coo_3_matrixA_col_matrixB_col(tensorCoreConfig,
                                                M,
                                                N,
                                                K,
                                                matrixB,
                                                matrixA,
                                                matrixSColIndex,
                                                matrixSRowIndex,
                                                matrixS,
                                                matrixSTileMappedToWarpIndex,
                                                matrixP);
    }
}