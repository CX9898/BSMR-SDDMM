#include "hip/hip_runtime.h"
#include <cstdio>

#include <mma.h>

#include "kernel.cuh"
#include "wmmaSetting.hpp"
#include "Matrix.hpp"

const float SPARSITY_BOUND = 0.5f;

template<typename T>
__device__ int sparsityComparator(const int WMMA_M,
                                  const int WMMA_N,
                                  const int ld,
                                  const MatrixStorageOrder storageOrder,
                                  const T *matrixPtr) {
    int nnzCount = 0;
#pragma unroll
    for (int rowIter = 0; rowIter < WMMA_M; ++rowIter) {
#pragma unroll
        for (int colIter = 0; colIter < WMMA_N; ++colIter) {
            if (storageOrder == MatrixStorageOrder::row_major) {
                nnzCount += *(matrixPtr + rowIter * ld + colIter) == 0 ? 0 : 1;
            } else {
                nnzCount += *(matrixPtr + colIter * ld + rowIter) == 0 ? 0 : 1;
            }
        }
    }
    const int numValues = WMMA_M * WMMA_N;
    const float sparsity = static_cast<float>(numValues - nnzCount) / numValues;
    return sparsity >= SPARSITY_BOUND ? 1 : 0;
}

template<typename T>
__global__ void printData(int n, T *a) {
    for (int i = 0; i < n; ++i) {
        printf("%f ", static_cast<float>(a[i]));
    }
}

template __global__ void printData<float>(int n, float *a);
template __global__ void printData<half>(int n, half *a);

__global__ void convertFp32ToFp16(const int n, const float *in, half *out) {
    int idx = (int) (blockDim.x * blockIdx.x + threadIdx.x);
    if (idx < n) {
        out[idx] = in[idx];
    }
}

__global__ void comp_sddmm_gpu(const int M, const int N, const int K,
                               const half *matrixA, const half *matrixB,
                               const float *matrixS,
                               float *matrixP) {
    const int tidX = (blockDim.x * blockIdx.x + threadIdx.x);
    const int tidY = (blockDim.y * blockIdx.y + threadIdx.y);

    const int warpM = (int) (blockDim.x * blockIdx.x + threadIdx.x) / WARP_SIZE;
    const int warpN = (int) (blockDim.y * blockIdx.y + threadIdx.y);

    // Compute dense matrix multiplication using Tensor core

    const int pRowId = warpM * WMMA_M;
    const int pColId = warpN * WMMA_N;

    if (pRowId >= M || pColId >= N) {
        return;
    }
//    printf("pRowId : %d, pColId : %d\n", pRowId,pColId);
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> aFrag;
    nvcuda::wmma::fragment<nvcuda::wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, nvcuda::wmma::row_major> bFrag;

    nvcuda::wmma::fragment<nvcuda::wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> cFrag;
    fill_fragment(cFrag, 0.0f);

    // Leading dimensions. Packed with no transpositions.
    const int lda = K;
    const int ldb = N;
    const int ldp = N;

    // Loop over k
    for (int kIter = 0; kIter < K; kIter += WMMA_K) {
        const int aRowId = pRowId;
        const int aColId = kIter;

        const int bRowId = kIter;
        const int bColId = pColId;
        if (tidX == 0) {
            printf(" cur kIter = %d\n", kIter);
            printf(" cur aRowId = %d, aColId = %d, bRowId = %d, bColId = %d\n", aRowId, aColId, bRowId, bColId);
        }
        // Bounds checking
        if (aRowId < M && aColId < K && bRowId < K && bColId < N) {
            const auto aOffsetPtr = matrixA + aRowId * lda + aColId;
            const auto bOffsetPtr = matrixB + bRowId * ldb + bColId;

            nvcuda::wmma::load_matrix_sync(aFrag, aOffsetPtr, lda);
            nvcuda::wmma::load_matrix_sync(bFrag, bOffsetPtr, ldb);

            nvcuda::wmma::mma_sync(cFrag, aFrag, bFrag, cFrag);
        }
    }

//#pragma unroll
//    for (int idx = 0; idx < cFrag.num_elements; ++idx) {
//        const int sIdx = pRowId * ldc + pColId + idx;
//
//        cFrag.x[idx] *= matrixS[sIdx];
//    }
//    if (tidX == 4) {
//        printf("\n aFrag.num_elements : %d\n", aFrag.num_elements);
//        for (int idx = 0; idx < aFrag.num_elements; ++idx) {
//            printf(" %f ", static_cast<float>(aFrag.x[idx]));
//        }
//    }

    const auto pOffsetPtr = matrixP + pRowId * ldp + pColId;
    nvcuda::wmma::store_matrix_sync(pOffsetPtr, cFrag, ldp, nvcuda::wmma::mem_row_major);
}
