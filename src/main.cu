#include <iostream>
#include <string>

#include "Matrix.hpp"
#include "sddmm.h"
#include "kernel.cuh"
#include "util.h"
#include "wmmaSetting.hpp"
#include "cudaErrorCheck.cuh"
#include "cudaUtil.cuh"
#include "CudaTimeCalculator.cuh"
#include "host.hpp"

const std::string folderPath("../dataset/");
const std::string fileName = ("nips.mtx");
//const std::string fileName = ("test1.mtx");
const std::string filePath = folderPath + fileName;

int main() {
    SparseMatrix<float> matrixS;
    matrixS.initializeFromMatrixMarketFile(filePath);

    const int K = 256;
    const int M = matrixS.row();
    const int N = matrixS.col();
    const int MATRIX_A_SIZE = M * K;
    const int MATRIX_B_SIZE = K * N;

//    std::cout << "matrixS : " << std::endl;
//    matrixS.printfValue();

    Matrix<float> matrixS2D;
    matrixS2D.initializeFromSparseMatrix(matrixS);
//    matrixS2D.changeStorageOrder();

    Matrix<float> matrixA(M, K, MATRIX_A_SIZE, MatrixStorageOrder::row_major, K);
    initial(matrixA.setValues(), M, K);
    Matrix<float> matrixB(K, N, MATRIX_B_SIZE, MatrixStorageOrder::row_major, N);
    initial(matrixB.setValues(), N, K);
//    matrixA.changeStorageOrder();
    matrixB.changeStorageOrder();

    SparseMatrix<float> matrixP_cpu_res(matrixS.row(), matrixS.col(), matrixS.nnz(),
                                        matrixS.rowIndex(), matrixS.colIndex());
    sddmm_cpu_coo(matrixA, matrixB, matrixS, matrixP_cpu_res);
//    std::cout << "matrixP.values() : " << std::endl;
//    matrixP_host.printfValue();

    float *valuesA_d;
    half *valuesAfp16_d;
    float *valuesB_d;
    half *valuesBfp16_d;
    float *valuesS_d;
    float *valuesP_d;

    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesA_d), matrixA.size() * sizeof(float)));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesAfp16_d), matrixA.size() * sizeof(half)));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesB_d), matrixB.size() * sizeof(float)));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesBfp16_d), matrixB.size() * sizeof(half)));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesS_d), matrixS2D.size() * sizeof(float)));
    cudaErrCheck(hipMalloc(reinterpret_cast<void **>(&valuesP_d), matrixS2D.size() * sizeof(float)));

    dev::H2D(valuesA_d, matrixA.values().data(), matrixA.size());
    dev::H2D(valuesB_d, matrixB.values().data(), matrixA.size());

    const int numThreadPerBlock = 1024;
    convertFp32ToFp16<<< (matrixA.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
        matrixA.size(), valuesA_d, valuesAfp16_d);
    convertFp32ToFp16<<< (matrixB.size() + numThreadPerBlock - 1) / numThreadPerBlock, numThreadPerBlock>>>(
        matrixB.size(), valuesB_d, valuesBfp16_d);

    dim3 grid;
    dim3 block;
    block.x = 128;
    block.y = 4;
    grid.x = (matrixS2D.row() + (WMMA_M * block.x / 32 - 1)) / (WMMA_M * block.x / 32);
    grid.y = (matrixS2D.col() + WMMA_N * block.y - 1) / (WMMA_N * block.y);

    CudaTimeCalculator timeCalculator;
    timeCalculator.startClock();
    comp_sddmm_gpu<<<grid, block>>>(M, N, K, valuesAfp16_d, valuesBfp16_d, valuesS_d, valuesP_d);
    timeCalculator.endClock();
    std::cout << "Func compSddmm time : " << timeCalculator.getTime() << "ms" << std::endl;

    SparseMatrix<float> matrixP_gpu_res(matrixS.row(), matrixS.col(), matrixS.nnz(),
                                        matrixS.rowIndex(), matrixS.colIndex());

    dev::D2H(matrixP_gpu_res.setValues().data(), valuesP_d, matrixP_gpu_res.values().size());


    std::cout << "matrixP_gpu_res : " << std::endl;
    matrixP_gpu_res.printfValue();

    isratnisa::Matrix isratnisaMatrixS;
    isratnisaMatrixS.copyFromMatrix(matrixS);

    float *valuesP_isratnisa = nullptr;
    preprocessing(isratnisaMatrixS, matrixA.values(), matrixB.values(), valuesP_isratnisa);

    return 0;
}